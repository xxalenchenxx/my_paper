#include "hip/hip_runtime.h"
#ifndef COMMON
#define COMMON
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <iomanip>  // 需要包含這個頭文件以使用 setprecision
#include <hip/hip_runtime.h>
#include "device_atomic_functions.h"
#endif

#include <vector>
#include <queue>
using namespace std;
#include "headers.h"
#define INFINITE 1000000000
// #define DEBUGx
#define DEBUG

// Updated Q_struct definition
typedef struct q_struct {
    uint64_t traverse_S;
    int nodeID;
} Q_struct;

#define CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(err); \
        } \
    } while (0)


#pragma region DefineLabel
// #define DEBUG
// #define CheckDistAns
// #define CheckCC_Ans
#pragma endregion //DefineLabel

#pragma region globalVar
int tempSourceID        = 1;
int CheckedNodeCount    = 0;

double forward_Time= 0;
double backward_Time= 0;
double total_time= 0;
double multi_forward_Time= 0;
double multi_backward_Time= 0;
double multi_total_time= 0;

double time_start                       = 0;
double time_end                         = 0;
double time1                            = 0;
double time2                            = 0;

double multi_time_start                 = 0;
double multi_time_end                   = 0;
double multi_time1                      = 0;
double multi_time2                      = 0;

double mymethod_start                 = 0;
double mymethod_end                   = 0;
double mymethod_time1                      = 0;
double mymethod_time2                      = 0;


#pragma endregion //globalVar


inline void resetQueue(struct qQueue* _Q){
    _Q->front   = 0;
    _Q->rear    = -1;
    //Q->size如果不變，就不須memcpy
}

//原版brandes
void check_ans(std::vector<float> ans, std::vector<float> my_ans);
void check_ans_int( int* ans, int* my_ans, CSR csr);
void compute_graph_degree( struct CSR& csr);
void brandes_ORIGIN_for_Seq( CSR& csr, int V, vector<float> &BC);
void brandes_with_predecessors(CSR& csr, int V, float* BC);
void computeBC_D1folding(struct CSR* _csr, float* _BCs);
void compute_D1_AP_BC(struct CSR* _csr, float* _BCs);
void brandes_with_predecessors_dynamic_check_ans(CSR csr, int V,int sourceID_test, vector<float> BC_ckeck);
//CC範例
void computeCC_ans(struct CSR* _csr, int* _CCs);
void compute_D1_CC(struct CSR* _csr, int* _CCs);
void compute_diameter(CSR* _csr);
void Seq_multi_source_brandes_ordered( CSR& csr, int max_multi, vector<float> &BC);
void Seq_multi_source_brandes(CSR& csr, int max_multi, vector<float> &BC);

//single source以及multi-source平行版本
void brandes_SS_par( CSR& csr, int V, float *BC);
void brandes_MS_par( CSR& csr, int max_multi, float* BC);
void brandes_MS_par_VnextQ( CSR& csr, int max_multi, float* BC);
void brandes_MS_Me_AP_D1( CSR& csr, int max_multi, float* BC);

//test 程式
void computeBC_shareBased_Successor_SS( CSR* _csr, float* _BCs);
void computeBC_shareBased_Successor_SS_edge_update( CSR* _csr, float* _BCs);
void computeBC_shareBased_Successor_SS_test( CSR* _csr, float* _BCs);
void computeBC_shareBased_Successor_MS( CSR* _csr, float* _BCs);

//DMF演算法
void computeBC_DMF_2018(struct CSR& csr,float* _BCs);
//DMF 延伸的演算法
void computeBC_DMFBased_Sequential_Vsquared(struct CSR& csr,float* _BCs);
void computeBC_DMFBased_Sequential_save_memory(struct CSR& csr,float* _BCs);

void printbinary(int data,int mappingcount){
    int count = mappingcount;
    while(count--){
        int byte = (data>>count) &1;
        printf("%d",byte);
        if(count%8==0){
            printf(" ");
        }
    }
    printf("]\n");
}


int main(int argc, char* argv[]){
    if (argc < 3) {
        cout << "Error: insufficient input arguments.\n";
        cout << "Usage: " << argv[0] << " <datasetPath> <max_multi>\n";
        return 1;
    }
    char* datasetPath = argv[1];
    int max_multi=stoi(argv[2]);
    printf("exeName = %s\n", argv[0]);
    printf("datasetPath = %s\n", datasetPath);
    struct Graph* graph = buildGraph(datasetPath);
    struct CSR* csr     = createCSR(graph);
    //答案專區
    vector<float> ans(csr->csrVSize,0.0);
    int *ans_CC= (int*)calloc(csr->csrVSize, sizeof(int));
    int *my_CC= (int*)calloc(csr->csrVSize, sizeof(int));
    float *ans_para= (float*)calloc(csr->csrVSize, sizeof(float));
    float *ans_para2= (float*)calloc(csr->csrVSize, sizeof(float));
    vector<float> my_BC(csr->csrVSize,0.0);
    vector<float> ans_para_vec(csr->csrVSize,0.0);
    vector<float> ans_para_vec2(csr->csrVSize,0.0);
    //brandes start
    printf("csrVSize   : %d\n",csr->csrVSize);
    printf("startNodeID: %d\n",csr->startNodeID);
    printf("endNodeID  : %d\n",csr->endNodeID);
    printf("startAtZero: %d\n",csr->startAtZero);
    // int max_multi=32;
    // compute_diameter(csr);

    time1 = seconds();
    // computeCC_shareBased_oneTraverse(csr,my_CC);
    // cout<<"max_degree: "<<csr->maxDegree<<endl;

    // brandes_ORIGIN_for_Seq(*csr,csr->csrVSize,ans);

    // brandes_SS_par(*csr,csr->csrVSize,ans_para);
    // brandes_MS_par(*csr , max_multi , ans_para);
    // // brandes_MS_par(*csr , max_multi , ans_para);

    // brandes_with_predecessors(*csr,csr->csrVSize,ans_para);
    time2 = seconds();
    printf("done 1\n");


    multi_time1 = seconds();
    // computeCC_ans(csr,ans_CC);

    // computeBC_DMFBased_Sequential_Vsquared(*csr,ans_para2);
    // computeBC_DMFBased_Sequential_save_memory(*csr,ans_para2);
    // computeBC_DMF_2018(*csr,ans_para2);
    // computeBC_D1folding(csr,ans_para2);
    compute_D1_AP_BC(csr,ans_para2);
    // computeCC_ans(csr,ans_CC);
    // compute_D1_CC(csr,my_CC);

    multi_time2 = seconds();
    printf("done 2\n");

    // computeBC_shareBased(csr,my_BC);
    // Seq_multi_source_brandes( *csr , max_multi , my_BC );

    // mymethod_time1 = seconds();
    // computeBC_shareBased_Successor_SS(csr,ans_para2);
    // computeBC_shareBased_Successor_SS_edge_update(csr,ans_para2);
    // mymethod_time2 = seconds();
    // printf("done 3\n");


    //檢查答案BC
    // for(int i=0;i<csr->csrVSize;i++){
    //     ans_para_vec[i]=ans_para[i];
    //     ans_para_vec2[i]=ans_para2[i];
    // }
    // // check_ans(ans_para_vec,ans_para_vec2);
    // check_ans(ans,ans_para_vec2);
    // check_ans_int(ans_CC,my_CC,*csr);

    //答案檢查CC
    // bool flag=true;
    // for(int node = csr->startNodeID;node<=csr->endNodeID;node++){
    //     if(ans_CC[node]!=my_CC[node]){
    //         printf("[ERROR] ans[%d]:%d\tmy[%d]:%d\n",node,ans_CC[node],node,my_CC[node]);
    //         flag=false;
    //     }
    // }
    // if(flag){
    //     cout<<"[CORRECT] CC!!!\n";
    // }

    // compute_graph_degree(*csr);

    // #ifdef DEBUG
    //     for(auto i=0;i<csr->csrVSize;i++){
    //         printf("BC[%d]: %f\n",i,ans[i]);
    //     }
    // #endif
    //brandes end
    // showCSR(csr);
    // printf("\n=================================single_source run time=================================\n");
    // printf("[Execution Time] forward_Time  = %.6f, %.6f \n", forward_Time, forward_Time / total_time);
    // printf("[Execution Time] backward_Time = %.6f, %.6f \n", backward_Time, backward_Time / total_time);
    // printf("[Execution Time] total_time    = %.6f, %.6f \n", total_time, (forward_Time+backward_Time )/ total_time);
    // printf("\n=================================multi_source run time=================================\n");
    // printf("[Execution Time] forward_Time  = %.6f, %.6f \n", multi_forward_Time, multi_forward_Time / multi_total_time);
    // printf("[Execution Time] backward_Time = %.6f, %.6f \n", multi_backward_Time, multi_backward_Time / multi_total_time);
    // printf("[Execution Time] total_time    = %.6f, %.6f \n", multi_total_time, (multi_forward_Time + multi_backward_Time ) / multi_total_time);
    
    
    // printf("[Execution Time] No_pred_total_time        = %.6f secs\n", time2-time1);
    // printf("[Execution Time] pred_total_time           = %.6f secs\n", multi_time2-multi_time1);
    // printf("[Execution Time] my_total_time             = %.6f secs\n", mymethod_time2-mymethod_time1);
    // printf("[Execution Time] speedup ratio             = %.6f secs\n", (time2-time1)/(multi_time2-multi_time1));
    // printf("[Execution Time] speedup ratio           = %.6f secs\n", (time2-time1)/(mymethod_time2-mymethod_time1));
    // printf("[Execution Time] speedup2(Pred) ratio             = %.6f secs\n", (multi_time2-multi_time1)/(mymethod_time2-mymethod_time1));
    return 0;
}

void check_ans( std::vector<float> ans, std::vector<float> my_ans) {
    if (ans.size() != my_ans.size()) {
        std::cout << "[ERROR] Vectors have different sizes: ans.size()=" << ans.size()
                  << ", my_ans.size()=" << my_ans.size() << std::endl;
        return;
    }

    bool all_correct = true;
    float epsilon = 0.01;  // 定義誤差率為 1%

    for (size_t i = 0; i < ans.size(); i++) {
        // 計算絕對誤差
        float delta = std::fabs(ans[i] - my_ans[i]);
        // 計算允許的誤差範圍
        float error_rate = std::fabs(ans[i]) * epsilon;  // 基於 ans[i] 的相對誤差
        // float error_rate =0.0;  // 基於 ans[i] 的相對誤差

        if (delta > error_rate) {
            // 顯示完整的小數精度
            std::cout << std::fixed << std::setprecision(6);
            std::cout << "[ERROR] ans[" << i << "] = " << ans[i]
                      << ", my_ans[" << i << "] = " << my_ans[i]
                      << ", delta = " << delta << ", allowed error = " << error_rate
                      << std::endl;
            all_correct = false;
        }
    }

    if (all_correct) {
        std::cout << "[CORRECT] my_ans matches ans " << std::endl;
    }

    return;
}

void check_ans_int( int* ans, int* my_ans, CSR csr) {
   

    bool all_correct = true;
    float epsilon = 0.01;  // 定義誤差率為 1%

    for (size_t i = csr.startNodeID; i <= csr.endNodeID; i++) {

        if ( ans[i]!=my_ans[i] ) {
            // 顯示完整的小數精度
            std::cout << std::fixed << std::setprecision(6);
            std::cout << "[ERROR] ans[" << i << "] = " << ans[i]
                      << ", my_ans[" << i << "] = " << my_ans[i]
                      << std::endl;
            all_correct = false;
        }
    }

    if (all_correct) {
        std::cout << "[CORRECT] my_ans matches ans " << std::endl;
    }

    return;
}


void quicksort_nodeID_with_degree(int* _nodes, int* _nodeDegrees, int _left, int _right){
    if(_left > _right){
        return;
    }
    int smallerAgent = _left;
    int smallerAgentNode = -1;
    int equalAgent = _left;
    int equalAgentNode = -1;
    int largerAgent = _right;
    int largerAgentNode = -1;

    int pivotNode = _nodes[_right];
    // printf("pivot : degree[%d] = %d .... \n", pivotNode, _nodeDegrees[pivotNode]);
    int tempNode = 0;
    while(equalAgent <= largerAgent){
        #ifdef DEBUG
        // printf("\tsmallerAgent = %d, equalAgent = %d, largerAgent = %d\n", smallerAgent, equalAgent, largerAgent);
        #endif

        smallerAgentNode = _nodes[smallerAgent];
        equalAgentNode = _nodes[equalAgent];
        largerAgentNode = _nodes[largerAgent];
        
        #ifdef DEBUG
        // printf("\tDegree_s[%d] = %d, Degree_e[%d] = %d, Degree_l[%d] = %d\n", smallerAgentNode, _nodeDegrees[smallerAgentNode], equalAgentNode, _nodeDegrees[equalAgentNode], largerAgentNode, _nodeDegrees[largerAgentNode]);
        #endif

        if(_nodeDegrees[equalAgentNode] > _nodeDegrees[pivotNode]){ //equalAgentNode的degree < pivotNode的degree
            // swap smallerAgentNode and equalAgentNode
            tempNode = _nodes[smallerAgent];
            _nodes[smallerAgent] = _nodes[equalAgent];
            _nodes[equalAgent] = tempNode;

            smallerAgent ++;
            equalAgent ++;
        }
        else if(_nodeDegrees[equalAgentNode] < _nodeDegrees[pivotNode]){ //equalAgentNode的degree > pivotNode的degree
            // swap largerAgentNode and equalAgentNode
            tempNode = _nodes[largerAgent];
            _nodes[largerAgent] = _nodes[equalAgent];
            _nodes[equalAgent] = tempNode;

            largerAgent --;
        }
        else{ //equalAgentNode的degree == pivotNode的degree
            equalAgent ++;
        }

    }
    
    // exit(1);
    #ifdef DEBUG
        
    #endif

    // smallerAgent現在是pivot key的開頭
    // largerAgent現在是pivotKey的結尾
    quicksort_nodeID_with_degree(_nodes, _nodeDegrees, _left, smallerAgent - 1);
    quicksort_nodeID_with_degree(_nodes, _nodeDegrees, largerAgent + 1, _right);
}


void compute_graph_degree( struct CSR& csr){
    int *degree_List      = (int*) calloc(sizeof(int), csr.csrVSize);
    float avg_degree= (float)(csr.csrESize/csr.csrVSize);
    printf("avg_degree: %.2f\n",avg_degree);
    for(int i=csr.startNodeID;i<=csr.endNodeID;i++){
        degree_List[csr.csrNodesDegree[i]]++;
    }

    for(int i=0;i<csr.csrVSize;i++){
        if(i>10)
            break;
        if(degree_List[i] )
            printf("degree[%d]: %d\t%.2f%\n", i,degree_List[i], (float)degree_List[i]/csr.csrVSize*100);
    }

}


//************************************************ */
//                   循序_brandes SS原版
//************************************************ */
 #pragma region brandes //forward traverse

void brandes_ORIGIN_for_Seq(CSR& csr, int V, std::vector<float>& BC) {
    // Time measurement
    double time_phase1 = 0.0;
    double time_phase2 = 0.0;
    double start_time = 0.0;
    double end_time = 0.0;

    // Allocate memory for sigma, dist, delta, and the stack S using STL containers
    std::vector<std::vector<int>> S(V);       // S is a 2D stack
    std::vector<int> sigma(V, 0);            // Sigma array
    std::vector<int> dist(V, -1);            // Distance array
    std::vector<float> delta(V, 0.0);        // Delta array
    std::queue<int> f1;                      // Current frontier queue
    std::queue<int> f2;                      // Next frontier queue

    for (int s = csr.startNodeID; s <= csr.endNodeID; ++s) {
        start_time = seconds();
        
        // Initialize variables for each source node
        std::fill(sigma.begin(), sigma.end(), 0);
        std::fill(dist.begin(), dist.end(), -1);
        std::fill(delta.begin(), delta.end(), 0.0);
        S.assign(V, vector<int>());  // Reset Successors with empty vectors
        sigma[s] = 1;
        dist[s] = 0;

        // Initialize the frontier
        f1.push(s);

        int level = 0;

        // BFS forward phase
        while (!f1.empty()) {
            while (!f1.empty()) {
                int u = f1.front();
                f1.pop();
                S[level].push_back(u);  // Put node u into its level

                // Traverse the adjacent nodes in CSR format
                for (int i = csr.csrV[u]; i < csr.csrV[u + 1]; ++i) {
                    int w = csr.csrE[i];

                    // If w has not been visited, update distance and add to next frontier
                    if (dist[w] < 0) {
                        dist[w] = dist[u] + 1;
                        f2.push(w);
                    }

                    // When a shortest path is found
                    if (dist[w] == dist[u] + 1) {
                        sigma[w] += sigma[u];
                    }
                }
            }

            // Swap queues for the next level
            std::swap(f1, f2);
            level++;
        }

        end_time = seconds();
        time_phase1 += end_time - start_time;
        start_time = seconds();

        // Backward phase to compute BC values
        for (int d = level - 1; d >= 0; --d) {  // Start from the furthest level
            for (int w : S[d]) {
                for (int i = csr.csrV[w]; i < csr.csrV[w + 1]; ++i) {
                    int v = csr.csrE[i];
                    if (dist[v] == dist[w] - 1) {
                        delta[v] += (sigma[v] / (float)sigma[w]) * (1.0 + delta[w]);
                    }
                }
                if (w != s) {
                    BC[w] += delta[w];
                }
            }
        }

        end_time = seconds();
        time_phase2 += end_time - start_time;
    }
    // time2 += time1 + time_phase1 + time_phase2;
    printf("phase1 time: %0.6f\n", time_phase1);
    printf("phase2 time: %0.6f\n", time_phase2);
    for (int i = 0; i < V; i++) {
        S[i].clear();  // 清除內部 vector
    }
    S.clear();
    sigma.clear();
    dist.clear();
    delta.clear();

}

void brandes_ORIGIN_for_Seq_noSTL(const CSR& csr, int V, vector<float> &BC) {
    // time_start = seconds();

  

    // Allocate memory for sigma, dist, delta, and the stack S
    int**  S = (int**)malloc(V * sizeof(int*));      // S is a 2D array (stack)
    int*   sigma = (int*)malloc(V * sizeof(int));     // sigma is a 1D array
    int*   dist = (int*)malloc(V * sizeof(int));      // dist is a 1D array
    float* delta = (float*)malloc(V * sizeof(float)); // delta is a 1D array
    int*   S_size = (int*)malloc(V * sizeof(int));    // S_size records the size of each level
    int*   f1 = (int*)malloc((V ) * sizeof(int));
    int*   f2 = (int*)malloc((V ) * sizeof(int));
    int    f1_indicator;
    int    f2_indicator;

    for (int i = 0; i < V; i++) {
        S[i] = (int*)malloc(V *sizeof(int));       // Each level's stack size V, adjust as needed
        S_size[i] = 0;                              // Initialize the size of each level
    }

    for (int s = csr.startNodeID; s <= csr.endNodeID; ++s) {
        // Initialize variables for each source node
        // time1 = seconds();
        for (int i = 0; i < V; i++) {
            free(S[i]);
            S[i] = (int*)malloc(V*sizeof(int));   // Each level's stack size V, adjust as needed
            sigma[i] = 0;
            dist[i] = -1;
            delta[i] = 0.0;
            S_size[i] = 0; // Reset size of each level
        }
    
        sigma[s] = 1;
        dist[s] = 0;
        f1_indicator = 0;
        f2_indicator = 0;
        // Re-initialize current_queue
        f1[f1_indicator++] = s;
        

        int level =0;
        // BFS forward phase: frontier-based BFS with extra mallocs
        while (f1_indicator>0) { //!qIsEmpty(current_queue)
            // printf("level: %d\n",level);
            
            // Allocate new memory for next_queue in each iteration
            int* currentQueue;
            int* nextQueue;
            if(level% 2 == 0){
                currentQueue = f1;
                nextQueue = f2;
            }
            else{
                currentQueue = f2;
                nextQueue = f1;
            }
            
            for(int v=0; v<f1_indicator; v++) {
                int u = currentQueue[v];
                S[level][S_size[level]++] = u;  // Put node u into its level
                
                // Traverse the adjacent nodes in CSR format
                for (int i = csr.csrV[u]; i < csr.csrV[u + 1]; ++i) {
                    int w = csr.csrE[i];
                    
                    // If w has not been visited, update distance and add to next_queue
                    if (dist[w] < 0) {
                        dist[w] = dist[u] + 1;
                        nextQueue[f2_indicator++] = w;
                        
                    }

                    // When a shortest path is found
                    if (dist[w] == dist[u] + 1) {
                        sigma[w] += sigma[u];
                    }
                }
            }
            
            // Free current_queue and set it to next_queue for the next iteration
            f1_indicator = f2_indicator;
            f2_indicator = 0;
            level++;
            
            // printf("level: %d\n",level);
            // qShowAllElement(current_queue);
            
        }

        // time2 = seconds();
        // forward_Time += (time2 - time1);
        // time1 = seconds();

        // Backward phase to compute BC values
        for (int d = level - 1; d >= 0; --d) {  // Start from the furthest level
            
            for (int i = 0; i < S_size[d]; ++i) {
                int w = S[d][i];
                
                for (int j = csr.csrV[w]; j < csr.csrV[w + 1]; ++j) {
                    int v = csr.csrE[j];
                    if (dist[v] == dist[w] - 1) {
                        delta[v] += (sigma[v] / (float)sigma[w]) * (1.0 + delta[w]);
                    }
                }
                if (w != s) {
                    BC[w] += delta[w];
                }
            }
            
        }
        // time2 = seconds();
        // backward_Time += (time2 - time1);
    }

    // time_end = seconds();
    // total_time = (time_end - time_start);
    // Free memory for S and its levels
    for (int i = 0; i < V; i++) {
        free(S[i]);
    }
    free(S);
    free(sigma);
    free(dist);
    free(delta);
    free(S_size);
}

#pragma endregion


//************************************************ */
//                   循序程式 DMF-延伸
//************************************************ */

void computeBC_DMFBased_Sequential_Vsquared(struct CSR& csr,float* _BCs) {
    // Allocate memory for time
    double time_phase1 = 0.0f;
    double time_phase2 = 0.0f;
    double time_phase3 = 0.0f;
    double time_phase4 = 0.0f;
    double time_phase5 = 0.0f;
    double start_time  = 0.0f;
    double end_time    = 0.0f;

    // Allocate memory for vertex coverage
    bool *edge_covered = (bool*)calloc(sizeof(bool), csr.csrESize); //確認edge已被cover到
    // bool *total_VC_List      = (bool*) calloc(sizeof(bool), csr.csrVSize); //確認nodeID被標記為主動cover
    int *VC_List      = (int*) calloc(sizeof(int), csr.csrVSize); //nodeID被標記為cover的點集合
    int *nonVC_List      = (int*) calloc(sizeof(int), csr.csrVSize); //nodeID被標記為非VCcovered的點集合
    int  VC_List_size  = 0, nonVC_List_size = 0;

    //找出avg_degree
    int V=csr.csrVSize;
    float avg_degree= (float)(csr.csrESize/V);
    // printf("avg_degree: %.2f\n",avg_degree);


    start_time=seconds();

    //用degree做排序 大->小
    csr.orderedCsrV  = (int*)calloc(sizeof(int), (csr.csrVSize));
    for(int i=csr.startNodeID;i<=csr.endNodeID;i++){
            csr.orderedCsrV[i]=i;
    }
    quicksort_nodeID_with_degree(csr.orderedCsrV, csr.csrNodesDegree, csr.startNodeID, csr.endNodeID);
    
    //=================================
    //         找出VC點以及非VC點
    //=================================
    //degree大於avg_degree標為VC，並記錄該edge為covered
    int avg_degree_nodeStartIndex=0;
    for(int index=csr.startNodeID ; index<=csr.endNodeID ; index++){
        int sourceID = csr.orderedCsrV[index];
        if(csr.csrNodesDegree[sourceID]<= avg_degree){
            avg_degree_nodeStartIndex=index;
            break;
        }

        // total_VC_List[sourceID]=1;
        VC_List[VC_List_size++]=sourceID;
        //記錄該edge為coverd
        for(int neighborIndex = csr.csrV[sourceID] ; neighborIndex < csr.csrV[sourceID + 1] ; neighborIndex ++){
            int neighborNodeID = csr.csrE[neighborIndex];
            edge_covered[neighborIndex]=true;
            for(int sIndex = csr.csrV[neighborNodeID] ; sIndex < csr.csrV[neighborNodeID + 1] ; sIndex ++){
                if(csr.csrE[sIndex]==sourceID){
                    edge_covered[sIndex]=true;
                    break;
                }
            }
        }
    }

    //degree <= avg 的 node找VC，如果edge都有covered 則為 nonVC
    for(int index=avg_degree_nodeStartIndex ; index<=csr.endNodeID ; index++){
        int sourceID = csr.orderedCsrV[index];
        bool nonVC_flag = true;
        for(int neighborIndex = csr.csrV[sourceID] ; neighborIndex < csr.csrV[sourceID + 1] ; neighborIndex ++){
            int neighborNodeID = csr.csrE[neighborIndex];
            if(!edge_covered[neighborIndex]){
                edge_covered[neighborIndex]=true;
                // total_VC_List[sourceID]=1;
                
                if(nonVC_flag)
                    VC_List[VC_List_size++]=sourceID;
                
                nonVC_flag=false;
                for(int sIndex = csr.csrV[neighborNodeID] ; sIndex < csr.csrV[neighborNodeID + 1] ; sIndex ++){
                    if(csr.csrE[sIndex]==sourceID){
                        edge_covered[sIndex]=true;
                        break;
                    }
                }

            }
        }
        if(nonVC_flag){
            nonVC_List[nonVC_List_size++] = sourceID;
        }
    }

    free(edge_covered);

    //紀錄nonVC鄰居的長度
    int nonVC_Neighbor_size=0;
    bool *nonVC_Neighbor_boolList    = (bool*) calloc(sizeof(bool), csr.csrVSize); //nodeID為需要紀錄sigma以及delta的點，以及map依據。
    int *nonVC_NeighborID_List       = (int*)  calloc(sizeof(int), csr.csrVSize); //nodeID為需要紀錄sigma以及delta的點，以及map依據。
    int *nonVC_Neighbor_ID2Index     = (int*)  calloc(sizeof(int), csr.csrVSize); //nodeID map回 nonVC_NeighborID_List的Index
    memset(nonVC_Neighbor_ID2Index, -1 ,csr.csrVSize*sizeof(int));
    
    for(int nonVIndex=0;nonVIndex<nonVC_List_size;nonVIndex++){
        int nonVC_ID = nonVC_List[nonVIndex];
        for(int neighborIndex = csr.csrV[nonVC_ID] ; neighborIndex < csr.csrV[nonVC_ID + 1] ; neighborIndex ++){
            int neighborNodeID = csr.csrE[neighborIndex];
            nonVC_Neighbor_boolList[neighborNodeID]= true;
        }
    }

    for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
        if(nonVC_Neighbor_boolList[nodeID]){
            nonVC_NeighborID_List[nonVC_Neighbor_size]=nodeID;
            nonVC_Neighbor_ID2Index[nodeID]=nonVC_Neighbor_size;
            nonVC_Neighbor_size++;
        }
    }
    // printf("nonVC_Neighbor_size: %d\n",nonVC_Neighbor_size);

    int **nonVC_NeighborID_dist  = (int**) malloc(nonVC_Neighbor_size * sizeof(int*)); //nodeID為需要紀錄sigma以及delta的點，以及map依據。
    int **nonVC_NeighborID_sigma = (int**) malloc(nonVC_Neighbor_size * sizeof(int*)); //nodeID為需要紀錄sigma以及delta的點，以及map依據。
    for (int i = 0; i < nonVC_Neighbor_size; i++) {
        nonVC_NeighborID_dist [i]= (int*)malloc(V *sizeof(int)); // 每個ID的點都需要紀錄距離
        nonVC_NeighborID_sigma[i]= (int*)malloc(V *sizeof(int)); // 每個ID的點都需要紀錄路徑數量
    }
    // printf("malloc: nonVC_NeighborID_sigma and dist\n");
    #pragma region printvalue
    
    // for (int i= csr.startNodeID;i<=csr.endNodeID;i++) {
    //     printf("total_VC_List[%d]: (%d)\n",i,total_VC_List[i]);
    // }

    // for (int i= 0;i<VC_List_size;i++) {
    //     printf("VC_List[%d]: (%d)\n",i,VC_List[i]);
    // }

    // for (int i= 0;i<nonVC_List_size;i++) {
    //     printf("nonVC_List[%d]: (%d)\n",i,nonVC_List[i]);
    // }

    // for (int i= 0;i<nonVC_Neighbor_size;i++) {
    //     printf("nonVC_NeighborID_List[%d]: (%d)\n",i,nonVC_NeighborID_List[i]);
    // }

    // for (int i= csr.startNodeID;i<=csr.endNodeID;i++) {
    //     printf("nonVC_Neighbor_ID2Index[%d]: (%d)\n",i,nonVC_Neighbor_ID2Index[i]);
    // }

    // printf("VC_List_size   : (%d) (%.2f)\n",VC_List_size,(float)VC_List_size/V);
    // printf("nonVC_List_size: (%d) (%.2f)\n",nonVC_List_size,(float)nonVC_List_size/V);
    // printf("nonVC_Neighbor_size   : (%d) (%.2f)\n",nonVC_Neighbor_size,(float)nonVC_Neighbor_size/V);
    #pragma endregion

    end_time=seconds();
    time_phase1 = end_time-start_time;
    
    //=============================
    //VC List點先做BFS brandes的算法
    //=============================
    // printf("Do VC list brandes\n");
    // Allocate memory for sigma, dist, delta, and the stack S
    int*   S = (int*)malloc(V * sizeof(int));      // S is a 2D array (stack)
    int*   sigma = (int*)malloc(V * sizeof(int));     // sigma is a 1D array
    int*   dist = (int*)malloc(V * sizeof(int));      // dist is a 1D array
    float* delta = (float*)malloc(V * sizeof(float)); // delta is a 1D array
    // int*   S_size = (int*)malloc(V * sizeof(int));    // S_size records the size of each level
    int*   f1 = (int*)malloc((V) * sizeof(int));
    int*   f2 = (int*)malloc((V) * sizeof(int));
    int    f1_indicator=0;
    int    f2_indicator=0;
    int    S_indicator =0;

    

    for (auto index = 0; index<VC_List_size ; index++ ) {
        int SourceID = VC_List[index];
        
        start_time=seconds();
        //forward
        for (int i = 0; i < V; i++) {
            sigma[i] = 0;
            dist[i] = -1;
            delta[i] = 0.0f;
        }
    

        sigma[SourceID] = 1;
        dist[SourceID]  = 0;
        f1_indicator    = 0;
        f2_indicator    = 0;
        S_indicator     = 0;
        // Re-initialize current_queue
        f1[f1_indicator++] = SourceID;

        int level =0;
        // BFS forward phase: frontier-based BFS with extra mallocs
        while (f1_indicator>0) { 
            // printf("level: %d\n",level);
            
            // Allocate new memory for next_queue in each iteration
            int* currentQueue;
            int* nextQueue;
            if(level% 2 == 0){
                currentQueue = f1;
                nextQueue = f2;
            }
            else{
                currentQueue = f2;
                nextQueue = f1;
            }
            
            for(int v=0; v<f1_indicator; v++) {
                int u = currentQueue[v];
                S[S_indicator++] = u;  // Put node u into its level
                
                // Traverse the adjacent nodes in CSR format
                for (int i = csr.csrV[u]; i < csr.csrV[u + 1]; ++i) {
                    int w = csr.csrE[i]; //w為u的鄰居
                    
                    // If w has not been visited, update distance and add to next_queue
                    if (dist[w] < 0) {
                        dist[w] = dist[u] + 1;
                        nextQueue[f2_indicator++] = w;
                    }

                    // When a shortest path is found
                    if (dist[w] == dist[u] + 1) {
                        sigma[w] += sigma[u];
                    }
                }
            }
            // Free current_queue and set it to next_queue for the next iteration
            f1_indicator = f2_indicator;
            f2_indicator = 0;
            level++;
        }

        end_time=seconds();
        time_phase2 += end_time-start_time;
        start_time=seconds();
        //backward
        for (int d = S_indicator - 1; d > 0; --d) {  // Start from the furthest level
            int w = S[d];
            
            for (int j = csr.csrV[w]; j < csr.csrV[w + 1]; ++j) {
                int v = csr.csrE[j];
                if (dist[v] == dist[w] - 1) {
                    delta[v] += (sigma[v] / (float)sigma[w]) * (1.0 + delta[w]);
                }
            }
            _BCs[w] += delta[w]; 
        }

        //檢查SourceID是否為該nonVC的鄰居需要紀錄?
        if(nonVC_Neighbor_ID2Index[SourceID]!=-1) { 
            //copy dist & sigma to nonVC_NeighborID_List
            for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
                nonVC_NeighborID_dist [nonVC_Neighbor_ID2Index[SourceID]][nodeID] =  dist[nodeID];
                nonVC_NeighborID_sigma[nonVC_Neighbor_ID2Index[SourceID]][nodeID] = sigma[nodeID];
            }
        }

        end_time=seconds();
        time_phase3 += end_time-start_time;

        #pragma region print
        
        // printf("========Source: %d=============\n",SourceID);
        // for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
        //     printf("delta[%d]: %.2f\n",nodeID,delta[nodeID]);
        // }
        
        #pragma endregion

    }

    free(f1);
    free(f2);
    free(VC_List);
    free(csr.orderedCsrV);
    // printf("Done VC list brandes\n");
    #pragma region print nonVC_NeighborID_sigma
    // for(int SourceID = 0 ; SourceID<  nonVC_Neighbor_size; SourceID++){
    //     printf("=======%d=======\n",nonVC_NeighborID_List[SourceID]);
    //     for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
    //         printf("nonVC_NeighborID_sigma[%d][%d]: %d\n",nonVC_NeighborID_List[SourceID],nodeID,nonVC_NeighborID_sigma [SourceID][nodeID]);
    //     }
    // }
    
    // for(int SourceID = 0 ; SourceID<  nonVC_Neighbor_size; SourceID++){
    //     printf("=======%d=======\n",nonVC_NeighborID_List[SourceID]);
    //     for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
    //         printf("nonVC_NeighborID_dist[%d][%d]: %d\n",nonVC_NeighborID_List[SourceID],nodeID,nonVC_NeighborID_dist [SourceID][nodeID]);
    //     }
    // }
    #pragma endregion
    
    //=================================
    //nonVC List點使用DMF的算法完成BC算法
    //=================================
    // printf("Do nonVC list brandes\n");
    #pragma region nonVC_NeighborID_sigma
    
    for (auto index = 0; index<nonVC_List_size ; index++ ) {
        int SourceID = nonVC_List[index];
        start_time=seconds();

        for (int i = 0; i < V; i++) {
            sigma[i]  =    0;
            dist[i]   =   -1;
            delta[i]  = 0.0f;
            S[i]      =    i;
        }
        
        //DMF-forward
        int max_depth=0;
        for(int v_ID = csr.startNodeID; v_ID <= csr.endNodeID; ++v_ID){
            if(v_ID==SourceID){
                dist[SourceID] =0;
                sigma[SourceID]=1;
            }else{
                //Source_path[s][v_ID]    : 每個S的鄰居到v_ID的距離最小就累加。
                //Source_distance[s][v_ID]: 每個S的鄰居到v_ID的距離最小。
                int min_distance=INT32_MAX;
                for (int i = csr.csrV[SourceID]; i < csr.csrV[SourceID+1]; ++i) { //每個S的鄰居到v_ID的距離
                    #ifdef DEBUG
                        if(nonVC_Neighbor_ID2Index[csr.csrE[i]]==-1){
                            printf("This Sourcenode %d is not covered by its neighbor %d\n",SourceID,csr.csrE[i]);
                        }
                    #endif
                    min_distance=min(min_distance, nonVC_NeighborID_dist[nonVC_Neighbor_ID2Index[csr.csrE[i]]][v_ID]);
                    
                }
                dist[v_ID]=min_distance+1;
                // S[ dist[v_ID] ][ S_size[dist[v_ID]]++ ]=v_ID;
                max_depth=max(max_depth,dist[v_ID]);
                
                for (int i = csr.csrV[SourceID]; i < csr.csrV[SourceID+1]; ++i) { //每個S的鄰居到v_ID的距離
                    int current_source = csr.csrE[i];
                    if( min_distance == nonVC_NeighborID_dist[nonVC_Neighbor_ID2Index[current_source]][v_ID]){ //current_source距離v_ID是最短
                        sigma[v_ID]+=nonVC_NeighborID_sigma[nonVC_Neighbor_ID2Index[current_source]][v_ID];
                    }
                }

            }
        }
        //利用nonVC到其他點的距離排列，得出backward的順序
        quicksort_nodeID_with_degree(S, dist, csr.startNodeID, csr.endNodeID);
        // for (int i= csr.startNodeID;i<=csr.endNodeID;i++) {
        //     printf("S[%d]   : (%d)\n",i,S[i]);
           
        // }
        // for (int i= csr.startNodeID;i<=csr.endNodeID;i++) {
        //     printf("dist[%d]: (%d)\n",i,dist[i]);
        // }

        end_time=seconds();
        time_phase4 += end_time-start_time;
        start_time=seconds();


        // printf("============Source: %d=============\n",SourceID);
        // for (int i= csr.startNodeID;i<=csr.endNodeID;i++) {
        //     printf("sigma[%d]: (%d)\n",i,sigma[i]);
        // }

        //backward
        for (int d = csr.startNodeID; d <= csr.endNodeID; ++d) {
            int w = S[d];
            for (int i = csr.csrV[w]; i < csr.csrV[w + 1]; ++i) {
                int v = csr.csrE[i];
                if (dist[v] == dist[w] - 1 ) {
                   delta[v] += (sigma[v] / (float)sigma[w]) * (1.0 + delta[w]);
                }
            }
            if(w!=SourceID)
                _BCs[w] += delta[w]; 
        }

        end_time=seconds();
        time_phase5 += end_time-start_time;

    }
    
    #pragma endregion 
    // printf("Done nonVC list brandes\n");

    // 釋放 nonVC_NeighborID_dist 內部的記憶體
    for (int i = 0; i < nonVC_Neighbor_size; i++) {
        free(nonVC_NeighborID_dist[i]); 
    }
    free(nonVC_NeighborID_dist);

    // 釋放 nonVC_NeighborID_sigma 內部的記憶體
    for (int i = 0; i < nonVC_Neighbor_size; i++) {
        free(nonVC_NeighborID_sigma[i]); 
    }
    free(nonVC_NeighborID_sigma);
    
    free(S);
    free(sigma);
    free(dist);
    free(delta);

    printf("phase1 time: %0.6f\n", time_phase1);
    printf("phase2 time: %0.6f\n", time_phase2);
    printf("phase3 time: %0.6f\n", time_phase3);
    printf("phase4 time: %0.6f\n", time_phase4);
    printf("phase5 time: %0.6f\n", time_phase5);
    printf("VC_List_size: %d\n", VC_List_size );
    printf("nonVC_List_size: %d\n", nonVC_List_size);
    printf("nonVC_Neighbor_size: %d\n", nonVC_Neighbor_size);
}

//比較演算法只做D2 沒有做D1效果更好
void computeBC_DMF_2018(struct CSR& csr,float* _BCs) {
    // Allocate memory for time
    double time_phase1 = 0.0f;
    double time_phase2 = 0.0f;
    double time_phase3 = 0.0f;
    double time_phase4 = 0.0f;
    double time_phase5 = 0.0f;
    double start_time  = 0.0f;
    double end_time    = 0.0f;

    int DMFnode=0;

    // Allocate memory for vertex coverage
    int V=csr.csrVSize;
    bool *Vertex_computed = (bool*)calloc(sizeof(bool), csr.csrESize); //確認node已被計算過BC值

    //找出avg_degree
    int avg_degree= (int)ceil(csr.csrESize/V);
    printf("avg_degree: %d\n",avg_degree);

    // Allocate memory for sigma, dist, delta, and the stack S
    int*   S     = (int*)malloc(V * sizeof(int));      // S is a 2D array (stack)
    int*   Source_sigma = (int*)malloc(V * sizeof(int));     // sigma is a 1D array
    int*   Source_dist  = (int*)malloc(V * sizeof(int));      // dist is a 1D array
    int**   sigma = (int**)malloc((avg_degree) * sizeof(int*));     // sigma is a 2D array
    int**   dist  = (int**)malloc((avg_degree) * sizeof(int*));     // sigma is a 2D array
    for (int i = 0; i < avg_degree; i++) {
        dist [i] = (int*)malloc(V *sizeof(int)); // 每個ID的點都需要紀錄距離
        sigma[i] = (int*)malloc(V *sizeof(int)); // 每個ID的點都需要紀錄路徑數量
    }
    

    float* delta = (float*)malloc(V * sizeof(float)); // delta is a 1D array
    int*   f1 = (int*)malloc(V * sizeof(int));
    int*   f2 = (int*)malloc(V * sizeof(int));
    int    f1_indicator=0;
    int    f2_indicator=0;
    int    S_indicator =0;

    //用degree做排序 大->小
    csr.orderedCsrV  = (int*)calloc(sizeof(int), V);
    memset(csr.orderedCsrV, -1, sizeof(int) * V);
    for(int i=csr.startNodeID;i<=csr.endNodeID;i++){
            csr.orderedCsrV[i]=i;
    }
    quicksort_nodeID_with_degree(csr.orderedCsrV, csr.csrNodesDegree, csr.startNodeID, csr.endNodeID);

    int othernodeStart=0;
    for(int i=csr.endNodeID;i>=csr.startNodeID;i--){
        int SourceID = csr.orderedCsrV[i];

        if(csr.csrNodesDegree[SourceID]==1) continue;

        //如果SourceID的degree大於avg_degree，不適合DMF定理
        if(csr.csrNodesDegree[SourceID] > 2){ //DMF只做到degree 2
            othernodeStart=i;
            break;
        }

        
        //檢查SourceID的鄰居是否有被計算過了? 有的話就不做DMF，直接做BFS。
        bool N_flag =false;
        for(int neighborIndex = csr.csrV[SourceID] ; neighborIndex < csr.csrV[SourceID + 1] ; neighborIndex ++){
            if(Vertex_computed[csr.csrE[neighborIndex]]){
               N_flag=true;
               break; 
            }
        }
        
        //做BFS先跳過
        //檢查SourceID是否有被計算過了?
        if(N_flag || Vertex_computed[SourceID]) continue;
        
        //這個Source
        Vertex_computed[SourceID]=true;
        DMFnode++;

        //計算SourceID鄰居的dist以及sigma (forward)
        for(int NeighborSource_index = csr.csrV[SourceID]; NeighborSource_index < csr.csrV[SourceID + 1] ; ++NeighborSource_index) {
            int NeighborSourceID = csr.csrE[NeighborSource_index];
            Vertex_computed[NeighborSourceID]=true;
            // printf("NeighborSourceID: %d\n",NeighborSourceID);

            start_time=seconds();

            //forward
            int NOneighbor= (NeighborSource_index-csr.csrV[SourceID]); //從0開始記錄鄰居的dist和sigma
            for (int i = 0; i < V; i++) {
                sigma[NOneighbor][i] =  0;
                dist[NOneighbor][i]  = -1;
                delta[i] = 0.0f;
            }


            sigma[NOneighbor][NeighborSourceID] = 1;
            dist[NOneighbor][NeighborSourceID]  = 0;
            f1_indicator    = 0;
            f2_indicator    = 0;
            S_indicator     = 0;
            // Re-initialize current_queue
            f1[f1_indicator++] = NeighborSourceID;
            int level = 0;
            while (f1_indicator>0) { 
            // printf("level: %d\n",level);

                // Allocate new memory for next_queue in each iteration
                int* currentQueue;
                int* nextQueue;
                if(level% 2 == 0){
                    currentQueue = f1;
                    nextQueue = f2;
                }
                else{
                    currentQueue = f2;
                    nextQueue = f1;
                }

                for(int v=0; v<f1_indicator; v++) {
                    int u = currentQueue[v];
                    S[S_indicator++] = u;  // Put node u into its level

                    // Traverse the adjacent nodes in CSR format
                    for (int i = csr.csrV[u]; i < csr.csrV[u + 1]; ++i) {
                        int w = csr.csrE[i]; //w為u的鄰居

                        // If w has not been visited, update distance and add to next_queue
                        if (dist[NOneighbor][w] < 0) {
                            dist[NOneighbor][w] = dist[NOneighbor][u] + 1;
                            nextQueue[f2_indicator++] = w;
                        }

                        // When a shortest path is found
                        if (dist[NOneighbor][w] == dist[NOneighbor][u] + 1) {
                            sigma[NOneighbor][w] += sigma[NOneighbor][u];
                        }
                    }
                }
                // Free current_queue and set it to next_queue for the next iteration
                f1_indicator = f2_indicator;
                f2_indicator = 0;
                level++;
            }

            end_time=seconds();
            time_phase1 += end_time-start_time;
            start_time=seconds();

            //backward
            for (int d = S_indicator - 1; d > 0; --d) {  // Start from the furthest level
                int w = S[d];
                // for(int v: predecessors[w]){
                //     delta[v] += (sigma[NOneighbor][v] / (float)sigma[NOneighbor][w]) * (1.0 + delta[w]);
                // }
                for (int j = csr.csrV[w]; j < csr.csrV[w + 1]; ++j) {
                    int v = csr.csrE[j];
                    if (dist[NOneighbor][v] == dist[NOneighbor][w] - 1) {
                        delta[v] += (sigma[NOneighbor][v] / (float)sigma[NOneighbor][w]) * (1.0 + delta[w]);
                    }
                }
                _BCs[w] += delta[w]; 
            }

            end_time=seconds();
            time_phase2 += end_time-start_time;
            //print dist and sigma
            #pragma region print
            // printf("==========S: %d==========\n",NeighborSourceID);
            // for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
            //     printf("dist[%d]: %d\n",nodeID,dist[NOneighbor][nodeID]);
            // }
            // for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
            //     printf("sigma[%d]: %d\n",nodeID,sigma[NOneighbor][nodeID]);
            // }
            // for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
            //     printf("delta[%d]: %.2f\n",nodeID,delta[nodeID]);
            // }
            #pragma endregion
        }
        

        start_time=seconds();

        //再計算出SourceID的forward
        int max_depth_my=0;
        for (int i = 0; i < V; i++) {
            Source_sigma[i]  =    0;
            Source_dist[i]   =   -1;
            delta[i]         = 0.0f;
            S[i]             =    i;
        }

        // SourceID forward bt DMF
        for(int v_ID = csr.startNodeID; v_ID <= csr.endNodeID; ++v_ID){
            if(v_ID==SourceID){
                Source_dist[SourceID] =0;
                Source_sigma[SourceID]=1;
            }else{
                //Source_path[s][v_ID]    : 每個S的鄰居到v_ID的距離最小就累加。
                //Source_distance[s][v_ID]: 每個S的鄰居到v_ID的距離最小。
                int min_distance=INT32_MAX;
                for (int current_source = 0; current_source < csr.csrNodesDegree[SourceID]; ++current_source) { //每個S的鄰居到v_ID的距離
                    min_distance=min(min_distance,dist[current_source][v_ID]);
                }
                Source_dist[v_ID]=min_distance+1;
                max_depth_my=max(max_depth_my,Source_dist[v_ID]);
                for (int current_source = 0; current_source < csr.csrNodesDegree[SourceID]; ++current_source) { //每個S的鄰居到v_ID的距離
                    if( min_distance == dist[current_source][v_ID]){ //current_source距離v_ID是最短
                        Source_sigma[v_ID]+=sigma[current_source][v_ID];
                    }
                }
            }

        }
        quicksort_nodeID_with_degree(S, Source_dist, csr.startNodeID, csr.endNodeID);
        
        end_time=seconds();
        time_phase3 += end_time-start_time;
        start_time=seconds();
        
        //SourceID的backward
        for (int d = csr.startNodeID; d <= csr.endNodeID; ++d) {
            int w = S[d];
            for (int i = csr.csrV[w]; i < csr.csrV[w + 1]; ++i) {
                int v = csr.csrE[i];
                if (Source_dist[v] == Source_dist[w] - 1 ) {
                   delta[v] += (Source_sigma[v] / (float)Source_sigma[w]) * (1.0 + delta[w]);
                }
            }
            if(w!=SourceID)
                _BCs[w] += delta[w]; 
        }

        end_time=seconds();
        time_phase4 += end_time-start_time;

        #pragma region print
        // printf("==========S: %d==========\n",SourceID);
        // for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
        //     printf("Source_dist[%d]: %d\n",nodeID,Source_dist[nodeID]);
        // }
        // for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
        //     printf("Source_sigma[%d]: %d\n",nodeID,Source_sigma[nodeID]);
        // }
        // for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
        //     printf("delta[%d]: %.2f\n",nodeID,delta[nodeID]);
        // }
        #pragma endregion
    
    }

    //確認所有node尚未被計算過BC值(要做BFS)
    int *notDoneIDList = (int*)calloc(sizeof(int), csr.csrVSize); 
    int lowDegreeBFSList_size=0;
    //找出尚未當Source的點
    for(int i=csr.startNodeID;i<=csr.endNodeID; ++i){
        if(!Vertex_computed[i]){
            notDoneIDList[lowDegreeBFSList_size++]=i;
        }
    }

    // printf("done notDoneIDList!!\n");

    //othernode 正常brandes
    for(int i=0 ; i<lowDegreeBFSList_size ; ++i){
        start_time=seconds();
        int SourceID = notDoneIDList[i];
        //initial
        for (int i = 0; i < V; i++) {
            Source_sigma[i] =  0;
            Source_dist[i]  = -1;
            delta[i] = 0.0f;
        }
        
        //forward
        Source_sigma[SourceID] = 1;
        Source_dist[SourceID]  = 0;
        f1_indicator    = 0;
        f2_indicator    = 0;
        S_indicator     = 0;
        // Re-initialize current_queue
        f1[f1_indicator++] = SourceID;
        int level = 0;

        while (f1_indicator>0){ 
        // printf("level: %d\n",level);
            // Allocate new memory for next_queue in each iteration
            int* currentQueue;
            int* nextQueue;
            if(level% 2 == 0){
                currentQueue = f1;
                nextQueue = f2;
            }
            else{
                currentQueue = f2;
                nextQueue = f1;
            }
            
            for(int v=0; v<f1_indicator; v++) {
                int u = currentQueue[v];
                S[S_indicator++] = u;  // Put node u into its level
                // Traverse the adjacent nodes in CSR format
                for (int i = csr.csrV[u]; i < csr.csrV[u + 1]; ++i) {
                    int w = csr.csrE[i]; //w為u的鄰居
                    // If w has not been visited, update distance and add to next_queue
                    if (Source_dist[w] < 0) {
                        Source_dist[w] = Source_dist[u] + 1;
                        nextQueue[f2_indicator++] = w;
                    }
                    // When a shortest path is found
                    if (Source_dist[w] == Source_dist[u] + 1) {
                        Source_sigma[w] += Source_sigma[u];
                    }
                }
            }
            // Free current_queue and set it to next_queue for the next iteration
            f1_indicator = f2_indicator;
            f2_indicator = 0;
            level++;
        }

        end_time=seconds();
        time_phase1 += end_time-start_time;
        start_time=seconds();

        //backward
        for (int d = S_indicator - 1; d > 0; --d) {  // Start from the furthest level
            int w = S[d];
            // for(int v: predecessors[w]){
            //     delta[v] += (Source_sigma[v] / (float)Source_sigma[w]) * (1.0 + delta[w]);
            // }
            for (int j = csr.csrV[w]; j < csr.csrV[w + 1]; ++j) {
                int v = csr.csrE[j];
                if (Source_dist[v] == Source_dist[w] - 1) {
                    delta[v] += (Source_sigma[v] / (float)Source_sigma[w]) * (1.0 + delta[w]);
                }
            }
            _BCs[w] += delta[w]; 
        }

        end_time=seconds();
        time_phase2 += end_time-start_time;

    }


    printf("phase1 time: %0.6f\n", time_phase1); //BFS forward
    printf("phase2 time: %0.6f\n", time_phase2); //BFS backward
    printf("phase3 time: %0.6f\n", time_phase3); //DMF forward
    printf("phase4 time: %0.6f\n", time_phase4); //DMF backward
    // printf("phase5 time: %0.6f\n", time_phase5);
    // printf("BFSNode_size: %d\n", csr.csrVSize - DMFnode -2);
    // printf("DMFnode_size: %d\n", DMFnode);

    
}

//目前最可行的方法!!
void computeBC_DMFBased_Sequential_save_memory(struct CSR& csr,float* _BCs) {
    // Allocate memory for time
    double time_phase1 = 0.0f;
    double time_phase2 = 0.0f;
    double time_phase3 = 0.0f;
    double time_phase4 = 0.0f;
    double time_phase5 = 0.0f;
    double start_time  = 0.0f;
    double end_time    = 0.0f;

    int DMFnode=0;

    // Allocate memory for vertex coverage
    int V=csr.csrVSize;
    bool *Vertex_computed = (bool*)calloc(sizeof(bool), csr.csrESize); //確認node已被計算過BC值

    //找出avg_degree
    int avg_degree= (int)ceil(csr.csrESize/V);
    printf("avg_degree: %d\n",avg_degree);

    // Allocate memory for sigma, dist, delta, and the stack S
    int*   S     = (int*)malloc(V * sizeof(int));      // S is a 2D array (stack)
    int*   Source_sigma = (int*)malloc(V * sizeof(int));     // sigma is a 1D array
    int*   Source_dist  = (int*)malloc(V * sizeof(int));      // dist is a 1D array
    int**   sigma = (int**)malloc((avg_degree) * sizeof(int*));     // sigma is a 2D array
    int**   dist  = (int**)malloc((avg_degree) * sizeof(int*));     // sigma is a 2D array
    vector<vector<int>> predecessors(V);   // Predecessor list
    for (int i = 0; i < avg_degree; i++) {
        dist [i] = (int*)malloc(V *sizeof(int)); // 每個ID的點都需要紀錄距離
        sigma[i] = (int*)malloc(V *sizeof(int)); // 每個ID的點都需要紀錄路徑數量
    }
    

    float* delta = (float*)malloc(V * sizeof(float)); // delta is a 1D array
    int*   f1 = (int*)malloc(V * sizeof(int));
    int*   f2 = (int*)malloc(V * sizeof(int));
    int    f1_indicator=0;
    int    f2_indicator=0;
    int    S_indicator =0;

    //用degree做排序 大->小
    csr.orderedCsrV  = (int*)calloc(sizeof(int), V);
    memset(csr.orderedCsrV, -1, sizeof(int) * V);
    for(int i=csr.startNodeID;i<=csr.endNodeID;i++){
            csr.orderedCsrV[i]=i;
    }
    quicksort_nodeID_with_degree(csr.orderedCsrV, csr.csrNodesDegree, csr.startNodeID, csr.endNodeID);

    int othernodeStart=0;
    for(int i=csr.endNodeID;i>=csr.startNodeID;i--){
        int SourceID = csr.orderedCsrV[i];

        //如果SourceID的degree大於avg_degree，不適合DMF定理
        if(csr.csrNodesDegree[SourceID] > avg_degree){
            othernodeStart=i;
            break;
        }

        
        //檢查SourceID的鄰居是否有被計算過了? 有的話就不做DMF，直接做BFS。
        bool N_flag =false;
        for(int neighborIndex = csr.csrV[SourceID] ; neighborIndex < csr.csrV[SourceID + 1] ; neighborIndex ++){
            if(Vertex_computed[csr.csrE[neighborIndex]]){
               N_flag=true;
               break; 
            }
        }
        
        //做BFS先跳過
        //檢查SourceID是否有被計算過了?
        if(N_flag || Vertex_computed[SourceID]) continue;
        
        //這個Source
        Vertex_computed[SourceID]=true;
        DMFnode++;

        //計算SourceID鄰居的dist以及sigma (forward)
        for(int NeighborSource_index = csr.csrV[SourceID]; NeighborSource_index < csr.csrV[SourceID + 1] ; ++NeighborSource_index) {
            int NeighborSourceID = csr.csrE[NeighborSource_index];
            Vertex_computed[NeighborSourceID]=true;
            // printf("NeighborSourceID: %d\n",NeighborSourceID);

            start_time=seconds();

            predecessors.assign(V, vector<int>());  // Reset Successors with empty vectors
            //forward
            int NOneighbor= (NeighborSource_index-csr.csrV[SourceID]); //從0開始記錄鄰居的dist和sigma
            for (int i = 0; i < V; i++) {
                sigma[NOneighbor][i] =  0;
                dist[NOneighbor][i]  = -1;
                delta[i] = 0.0f;
            }


            sigma[NOneighbor][NeighborSourceID] = 1;
            dist[NOneighbor][NeighborSourceID]  = 0;
            f1_indicator    = 0;
            f2_indicator    = 0;
            S_indicator     = 0;
            // Re-initialize current_queue
            f1[f1_indicator++] = NeighborSourceID;
            int level = 0;
            while (f1_indicator>0) { 
            // printf("level: %d\n",level);

                // Allocate new memory for next_queue in each iteration
                int* currentQueue;
                int* nextQueue;
                if(level% 2 == 0){
                    currentQueue = f1;
                    nextQueue = f2;
                }
                else{
                    currentQueue = f2;
                    nextQueue = f1;
                }

                for(int v=0; v<f1_indicator; v++) {
                    int u = currentQueue[v];
                    S[S_indicator++] = u;  // Put node u into its level

                    // Traverse the adjacent nodes in CSR format
                    for (int i = csr.csrV[u]; i < csr.csrV[u + 1]; ++i) {
                        int w = csr.csrE[i]; //w為u的鄰居

                        // If w has not been visited, update distance and add to next_queue
                        if (dist[NOneighbor][w] < 0) {
                            dist[NOneighbor][w] = dist[NOneighbor][u] + 1;
                            nextQueue[f2_indicator++] = w;
                        }

                        // When a shortest path is found
                        if (dist[NOneighbor][w] == dist[NOneighbor][u] + 1) {
                            sigma[NOneighbor][w] += sigma[NOneighbor][u];
                            predecessors[w].push_back(u);
                        }
                    }
                }
                // Free current_queue and set it to next_queue for the next iteration
                f1_indicator = f2_indicator;
                f2_indicator = 0;
                level++;
            }

            end_time=seconds();
            time_phase1 += end_time-start_time;
            start_time=seconds();

            //backward
            for (int d = S_indicator - 1; d > 0; --d) {  // Start from the furthest level
                int w = S[d];
                for(int v: predecessors[w]){
                    delta[v] += (sigma[NOneighbor][v] / (float)sigma[NOneighbor][w]) * (1.0 + delta[w]);
                }
                // for (int j = csr.csrV[w]; j < csr.csrV[w + 1]; ++j) {
                //     int v = csr.csrE[j];
                //     if (dist[NOneighbor][v] == dist[NOneighbor][w] - 1) {
                //         delta[v] += (sigma[NOneighbor][v] / (float)sigma[NOneighbor][w]) * (1.0 + delta[w]);
                //     }
                // }
                _BCs[w] += delta[w]; 
            }

            end_time=seconds();
            time_phase2 += end_time-start_time;
            //print dist and sigma
            #pragma region print
            // printf("==========S: %d==========\n",NeighborSourceID);
            // for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
            //     printf("dist[%d]: %d\n",nodeID,dist[NOneighbor][nodeID]);
            // }
            // for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
            //     printf("sigma[%d]: %d\n",nodeID,sigma[NOneighbor][nodeID]);
            // }
            // for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
            //     printf("delta[%d]: %.2f\n",nodeID,delta[nodeID]);
            // }
            #pragma endregion
        }
        

        start_time=seconds();

        //再計算出SourceID的forward
        int max_depth_my=0;
        for (int i = 0; i < V; i++) {
            Source_sigma[i]  =    0;
            Source_dist[i]   =   -1;
            delta[i]         = 0.0f;
            S[i]             =    i;
        }

        // SourceID forward bt DMF
        for(int v_ID = csr.startNodeID; v_ID <= csr.endNodeID; ++v_ID){
            if(v_ID==SourceID){
                Source_dist[SourceID] =0;
                Source_sigma[SourceID]=1;
            }else{
                //Source_path[s][v_ID]    : 每個S的鄰居到v_ID的距離最小就累加。
                //Source_distance[s][v_ID]: 每個S的鄰居到v_ID的距離最小。
                int min_distance=INT32_MAX;
                for (int current_source = 0; current_source < csr.csrNodesDegree[SourceID]; ++current_source) { //每個S的鄰居到v_ID的距離
                    min_distance=min(min_distance,dist[current_source][v_ID]);
                }
                Source_dist[v_ID]=min_distance+1;
                max_depth_my=max(max_depth_my,Source_dist[v_ID]);
                for (int current_source = 0; current_source < csr.csrNodesDegree[SourceID]; ++current_source) { //每個S的鄰居到v_ID的距離
                    if( min_distance == dist[current_source][v_ID]){ //current_source距離v_ID是最短
                        Source_sigma[v_ID]+=sigma[current_source][v_ID];
                    }
                }
            }

        }
        quicksort_nodeID_with_degree(S, Source_dist, csr.startNodeID, csr.endNodeID);
        
        end_time=seconds();
        time_phase3 += end_time-start_time;
        start_time=seconds();
        
        //SourceID的backward
        for (int d = csr.startNodeID; d <= csr.endNodeID; ++d) {
            int w = S[d];
            for (int i = csr.csrV[w]; i < csr.csrV[w + 1]; ++i) {
                int v = csr.csrE[i];
                if (Source_dist[v] == Source_dist[w] - 1 ) {
                   delta[v] += (Source_sigma[v] / (float)Source_sigma[w]) * (1.0 + delta[w]);
                }
            }
            if(w!=SourceID)
                _BCs[w] += delta[w]; 
        }

        end_time=seconds();
        time_phase4 += end_time-start_time;

        #pragma region print
        // printf("==========S: %d==========\n",SourceID);
        // for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
        //     printf("Source_dist[%d]: %d\n",nodeID,Source_dist[nodeID]);
        // }
        // for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
        //     printf("Source_sigma[%d]: %d\n",nodeID,Source_sigma[nodeID]);
        // }
        // for(int nodeID=csr.startNodeID ; nodeID<=csr.endNodeID ; nodeID++){
        //     printf("delta[%d]: %.2f\n",nodeID,delta[nodeID]);
        // }
        #pragma endregion
    
    }

    //確認所有node尚未被計算過BC值(要做BFS)
    int *notDoneIDList = (int*)calloc(sizeof(int), csr.csrVSize); 
    int lowDegreeBFSList_size=0;
    //找出尚未當Source的點
    for(int i=csr.startNodeID;i<=csr.endNodeID; ++i){
        if(!Vertex_computed[i]){
            notDoneIDList[lowDegreeBFSList_size++]=i;
        }
    }

    // printf("done notDoneIDList!!\n");

    //othernode 正常brandes
    for(int i=0 ; i<lowDegreeBFSList_size ; ++i){
        start_time=seconds();
        int SourceID = notDoneIDList[i];
        //initial
        predecessors.assign(V, vector<int>());  // Reset Successors with empty vectors
        for (int i = 0; i < V; i++) {
            Source_sigma[i] =  0;
            Source_dist[i]  = -1;
            delta[i] = 0.0f;
        }
        
        //forward
        Source_sigma[SourceID] = 1;
        Source_dist[SourceID]  = 0;
        f1_indicator    = 0;
        f2_indicator    = 0;
        S_indicator     = 0;
        // Re-initialize current_queue
        f1[f1_indicator++] = SourceID;
        int level = 0;

        while (f1_indicator>0){ 
        // printf("level: %d\n",level);
            // Allocate new memory for next_queue in each iteration
            int* currentQueue;
            int* nextQueue;
            if(level% 2 == 0){
                currentQueue = f1;
                nextQueue = f2;
            }
            else{
                currentQueue = f2;
                nextQueue = f1;
            }
            
            for(int v=0; v<f1_indicator; v++) {
                int u = currentQueue[v];
                S[S_indicator++] = u;  // Put node u into its level
                // Traverse the adjacent nodes in CSR format
                for (int i = csr.csrV[u]; i < csr.csrV[u + 1]; ++i) {
                    int w = csr.csrE[i]; //w為u的鄰居
                    // If w has not been visited, update distance and add to next_queue
                    if (Source_dist[w] < 0) {
                        Source_dist[w] = Source_dist[u] + 1;
                        nextQueue[f2_indicator++] = w;
                    }
                    // When a shortest path is found
                    if (Source_dist[w] == Source_dist[u] + 1) {
                        Source_sigma[w] += Source_sigma[u];
                        predecessors[w].push_back(u);
                    }
                }
            }
            // Free current_queue and set it to next_queue for the next iteration
            f1_indicator = f2_indicator;
            f2_indicator = 0;
            level++;
        }

        end_time=seconds();
        time_phase1 += end_time-start_time;
        start_time=seconds();

        //backward
        for (int d = S_indicator - 1; d > 0; --d) {  // Start from the furthest level
            int w = S[d];
            for(int v: predecessors[w]){
                delta[v] += (Source_sigma[v] / (float)Source_sigma[w]) * (1.0 + delta[w]);
            }
            // for (int j = csr.csrV[w]; j < csr.csrV[w + 1]; ++j) {
            //     int v = csr.csrE[j];
            //     if (Source_dist[v] == Source_dist[w] - 1) {
            //         delta[v] += (Source_sigma[v] / (float)Source_sigma[w]) * (1.0 + delta[w]);
            //     }
            // }
            _BCs[w] += delta[w]; 
        }

        end_time=seconds();
        time_phase2 += end_time-start_time;

    }


    printf("phase1 time: %0.6f\n", time_phase1); //BFS forward
    printf("phase2 time: %0.6f\n", time_phase2); //BFS backward
    printf("phase3 time: %0.6f\n", time_phase3); //DMF forward
    printf("phase4 time: %0.6f\n", time_phase4); //DMF backward
    // printf("phase5 time: %0.6f\n", time_phase5);
    printf("BFSNode_size: %d\n", csr.csrVSize - DMFnode -2);
    printf("DMFnode_size: %d\n", DMFnode);

    
}

//D1-folding BC
void computeBC_D1folding(struct CSR* _csr, float* _BCs){
    int V=_csr->csrVSize;
    //D1 folding
    D1Folding(_csr);

    // Allocate memory for sigma, dist, delta, and the stack S
    int*   S      = (int*)malloc(V * sizeof(int));      // S is a 2D array (stack)
    int*   sigma  = (int*)malloc(V * sizeof(int));     // sigma is a 1D array
    int*   dist   = (int*)malloc(V * sizeof(int));      // dist is a 1D array
    float* delta  = (float*)malloc(V * sizeof(float)); // delta is a 1D array
    int*   S_size = (int*)malloc(V * sizeof(int));    // S_size records the size of each level
    int*   f1     = (int*)malloc((V ) * sizeof(int));
    int*   f2     = (int*)malloc((V ) * sizeof(int));
    int    f1_indicator;
    int    f2_indicator;
    int    S_indicator =0;
    vector<vector<int>> predecessors(V);
    #pragma region SourceTraverse_With_ff_And_represent
    //In this block, we get the CC of each remaining node in the component
    int sourceID = -1;
    for(int notD1NodeIndex = 0 ; notD1NodeIndex < _csr->ordinaryNodeCount ; notD1NodeIndex ++){
        sourceID = _csr->notD1Node[notD1NodeIndex];
        #ifdef DEBUG
            // printf("sourceID:%d ff:%d representNode:%d\n",sourceID,_csr->ff[sourceID],_csr->representNode[sourceID]);
        #endif   
        //initial value
        predecessors.assign(V, vector<int>());  // Reset Successors with empty vectors
        for (int i = 0; i < V; i++) {
            sigma[i] =  0;
            dist[i]  = -1;
            //照該node的reach點數來初始，代表其他點看到這點至少看過reach-1個點在這個node之後。
            delta[i] = (float)_csr->representNode[i] - 1.0f;
        }
        

        sigma[sourceID] = 1;
        dist[sourceID]  = 0;
        f1_indicator    = 0;
        f2_indicator    = 0;
        S_indicator     = 0;
        // Re-initialize current_queue
        f1[f1_indicator++] = sourceID;
        int level = 0;
        //forward traverse
        while (f1_indicator>0){ 
        // printf("level: %d\n",level);
            // Allocate new memory for next_queue in each iteration
            int* currentQueue;
            int* nextQueue;
            if(level% 2 == 0){
                currentQueue = f1;
                nextQueue = f2;
            }
            else{
                currentQueue = f2;
                nextQueue = f1;
            }
            
            for(int v=0; v<f1_indicator; v++) {
                int u = currentQueue[v];
                S[S_indicator++] = u;  // Put node u into its level
                // Traverse the adjacent nodes in CSR format
                for (int i = _csr->csrV[u]; i < _csr->oriCsrV[u + 1]; ++i) {
                    int w = _csr->csrE[i]; //w為u的鄰居
                    // If w has not been visited, update distance and add to next_queue
                    if (dist[w] < 0) {
                        dist[w] = dist[u] + 1;
                        nextQueue[f2_indicator++] = w;
                    }
                    // When a shortest path is found
                    if (dist[w] == dist[u] + 1) {
                        sigma[w] += sigma[u];
                        predecessors[w].push_back(u);
                    }
                }
            }
            // Free current_queue and set it to next_queue for the next iteration
            f1_indicator = f2_indicator;
            f2_indicator = 0;
            level++;
        }

        //backward accumulation
        //backward
        for (int d = S_indicator - 1; d > 0; --d) {  // Start from the furthest level
            int w = S[d];
            for(int v: predecessors[w]){
                delta[v] += (sigma[v] / (float)sigma[w]) * (1.0 + delta[w]);
            }
            // for (int j = csr.csrV[w]; j < csr.csrV[w + 1]; ++j) {
            //     int v = csr.csrE[j];
            //     if (Source_dist[v] == Source_dist[w] - 1) {
            //         delta[v] += (Source_sigma[v] / (float)Source_sigma[w]) * (1.0 + delta[w]);
            //     }
            // }
            _BCs[w] += delta[w] * _csr->representNode[sourceID]; 
        }


    }
    #pragma endregion //SourceTraverse_With_ff_And_represent

    #pragma region d1Node_Dist_And_BC_Recovery
    // printf("_csr->totalNodeNumber = %2d\n", _csr->totalNodeNumber);
    int d1NodeID        = -1;
    int d1NodeParentID  = -1;
    for(int d1NodeIndex = _csr->degreeOneNodesQ->rear ; d1NodeIndex >= 0 ; d1NodeIndex --){
        d1NodeID        = _csr->degreeOneNodesQ->dataArr[d1NodeIndex];
        d1NodeParentID  = _csr->D1Parent[d1NodeID];
        _BCs[d1NodeID]  = (_csr->representNode[d1NodeID]-1) * (V-1-_csr->representNode[d1NodeID]);
        _BCs[d1NodeParentID]  += (V-_csr->representNode[d1NodeID]-2) * (_csr->representNode[d1NodeID]); //(|v|-1 - representNode[d1node] -1(程式預設多一個))
        // printf("d1NodeID = %2d, _CCs[%2d] = %2d, ParentID = %2d, _CCs[%2d] = %2d\n", d1NodeID, d1NodeID, _CCs[d1NodeID], d1NodeParentID, d1NodeParentID, _CCs[d1NodeParentID]);
    }
    #pragma endregion //d1Node_Dist_And_CC_Recovery

}

//D1-folding,AP BC
void compute_D1_AP_BC(struct CSR* _csr, float* _BCs){
    int V=_csr->csrVSize;
    
    // Allocate memory for sigma, dist, delta, and the stack S
    int*   S      = (int*)malloc(V * 2* sizeof(int));      // S is a 2D array (stack)
    int*   sigma  = (int*)malloc(V * 2* sizeof(int));     // sigma is a 1D array
    int*   dist   = (int*)malloc(V * 2* sizeof(int));      // dist is a 1D array
    float* delta  = (float*)malloc(V * 2* sizeof(float)); // delta is a 1D array
    int*   S_size = (int*)malloc(V * 2* sizeof(int));    // S_size records the size of each level
    int*   f1     = (int*)malloc(V * 2 * sizeof(int));
    int*   f2     = (int*)malloc(V * 2 * sizeof(int));
    int    f1_indicator;
    int    f2_indicator;
    int    S_indicator =0;
    
    //D1 Folding
    D1Folding(_csr);

    //AP Process
    AP_detection(_csr);
    AP_Copy_And_Split(_csr); //先計算ap本尊的CC
    struct newID_info* newID_infos = rebuildGraph(_csr);

    #pragma region BC
    //Traverse
    for(int sourceNewID = 0 ; sourceNewID <= _csr->newEndID ; sourceNewID ++){
        int oldID = _csr->mapNodeID_New_to_Old[sourceNewID];
        int sourceType = _csr->nodesType[oldID];

        if(sourceType & ClonedAP){
            // printf("newID %d, oldID %d, type %x\n", sourceNewID, oldID, sourceType);
            continue;
        }
        
        //變數值使用NewID
        for (int i = 0; i <= _csr->newEndID; i++) {
            sigma[i] =  0;
            dist[i]  = -1;
            //照該node的reach點數來初始，代表其他點看到這點至少看過reach-1個點在這個node之後。
            delta[i] = (float)newID_infos[i].w - 1.0f;
        }

        //initial value
        sigma[sourceNewID] = 1;
        dist[sourceNewID]  = 0;
        f1_indicator       = 0;
        f2_indicator       = 0;
        S_indicator        = 0;
        f1[f1_indicator++] = sourceNewID;
        int level = 0;

        //forward traverse
        while (f1_indicator>0){ 
        // printf("level: %d\n",level);
            // Allocate new memory for next_queue in each iteration
            int* currentQueue;
            int* nextQueue;
            if(level% 2 == 0){
                currentQueue = f1;
                nextQueue = f2;
            }
            else{
                currentQueue = f2;
                nextQueue = f1;
            }
            
            for(int v=0; v<f1_indicator; v++) {
                int curNewID = currentQueue[v];
                S[S_indicator++] = curNewID;  // Put node u into its level
                // Traverse the adjacent nodes in CSR format
                for(int new_nidx = _csr->orderedCsrV[curNewID] ; new_nidx < _csr->orderedCsrV[curNewID + 1] ; new_nidx ++) {
                    int new_nid = _csr->orderedCsrE[new_nidx]; //new_nid為curNewID的鄰居
                    // If w has not been visited, update distance and add to next_queue
                    if (dist[new_nid] < 0) {
                        dist[new_nid] = dist[curNewID] + 1;
                        nextQueue[f2_indicator++] = new_nid;
                    }
                    // When a shortest path is found
                    if (dist[new_nid] == dist[curNewID] + 1) {
                        sigma[new_nid] += sigma[curNewID];
                    }
                }
            }
            // Free current_queue and set it to next_queue for the next iteration
            f1_indicator = f2_indicator;
            f2_indicator = 0;
            level++;
        }

        for (int d = S_indicator - 1; d > 0; --d) {  // Start from the furthest level
            int w = S[d];
            oldID = _csr->mapNodeID_New_to_Old[w];
            // for(int v: predecessors[w]){
            //     delta[v] += (sigma[v] / (float)sigma[w]) * (1.0 + delta[w]);
            // }
            for(int new_nidx = _csr->orderedCsrV[w] ; new_nidx < _csr->orderedCsrV[w + 1] ; new_nidx ++) {
                int v = _csr->orderedCsrE[new_nidx];
                if (dist[v] == dist[w] - 1) {
                    delta[v] += (sigma[v] / (float)sigma[w]) * (1.0 + delta[w]);
                }
            }
            //BC紀錄值到舊的ID的位置
            _BCs[oldID] += delta[w] * newID_infos[sourceNewID].w; //_csr->representNode[sourceNewID]; 
            
        }

        // while(!(Q_front > Q_rear)){
        //     curNewID = nodeQ[Q_front++];
            
        //     for(new_nidx = _csr->orderedCsrV[curNewID] ; new_nidx < _csr->orderedCsrV[curNewID + 1] ; new_nidx ++){
        //         new_nid = _csr->orderedCsrE[new_nidx];

        //         if(dist_arr[new_nid] == -1){
        //             dist_arr[new_nid] = dist_arr[curNewID] + 1;
        //             nodeQ[++Q_rear] = new_nid;

        //             allDist += newID_infos[new_nid].ff + dist_arr[new_nid] * newID_infos[new_nid].w;
        //         }
        //     }
        // }
        // _csr->CCs[oldID] = allDist + _csr->ff[oldID];
        // printf("CC[%d] = %d\n", oldID, _csr->CCs[oldID]);
    }
    
    #pragma endregion 
    

    #pragma region ptintValue
    for(int sourceNewID = 0 ; sourceNewID <= _csr->newEndID ; sourceNewID ++){
        int oldID = _csr->mapNodeID_New_to_Old[sourceNewID];
        int sourceType = _csr->nodesType[oldID];

        if(sourceType & ClonedAP){
            printf("[ClonedAP] ");
            // printf("newID %d, oldID %d, type %x\n", sourceNewID, oldID, sourceType);
        }else{
            printf("[normal] ");
        }
        printf("newID %d, oldID %d, type %x, R_old:%d, ff_old:%d, R_new:%d, ff_new:%d\nneighbor{", sourceNewID, oldID, sourceType,_csr->representNode[oldID],_csr->ff[oldID],newID_infos[sourceNewID].w,newID_infos[sourceNewID].ff);

        for(int new_nidx = _csr->orderedCsrV[sourceNewID] ; new_nidx < _csr->orderedCsrV[sourceNewID + 1] ; new_nidx ++) {
            int new_nid = _csr->orderedCsrE[new_nidx]; //w為u的鄰居
            printf("%d ", new_nid);
        }
        printf("}\n");
    }
    #pragma endregion 

    #pragma region d1Node_Dist_And_CC_Recovery
    // // printf("_csr->totalNodeNumber = %2d\n", _csr->totalNodeNumber);
    // int d1NodeID        = -1;
    // int d1NodeParentID  = -1;
    // for(int d1NodeIndex = _csr->degreeOneNodesQ->rear ; d1NodeIndex >= 0 ; d1NodeIndex --){
    //     d1NodeID        = _csr->degreeOneNodesQ->dataArr[d1NodeIndex];
    //     d1NodeParentID  = _csr->D1Parent[d1NodeID];
    //     _BCs[d1NodeID]  = _BCs[d1NodeParentID] + _csr->totalNodeNumber - 2 * _csr->representNode[d1NodeID];
    //     // printf("d1NodeID = %2d, _CCs[%2d] = %2d, ParentID = %2d, _CCs[%2d] = %2d\n", d1NodeID, d1NodeID, _CCs[d1NodeID], d1NodeParentID, d1NodeParentID, _CCs[d1NodeParentID]);
    // }
    // printf("_csr->totalNodeNumber = %2d\n", _csr->totalNodeNumber);

    // int d1NodeID        = -1;
    // int d1NodeParentID  = -1;
    // for(int d1NodeIndex = _csr->degreeOneNodesQ->rear ; d1NodeIndex >= 0 ; d1NodeIndex --){
    //     d1NodeID        = _csr->degreeOneNodesQ->dataArr[d1NodeIndex];
    //     d1NodeParentID  = _csr->D1Parent[d1NodeID];
    //     _BCs[d1NodeID]  = (_csr->representNode[d1NodeID]-1) * (V-1-_csr->representNode[d1NodeID]);
    //     _BCs[d1NodeParentID]  += (V-_csr->representNode[d1NodeID]-2) * (_csr->representNode[d1NodeID]);
    //     // printf("d1NodeID = %2d, _CCs[%2d] = %2d, ParentID = %2d, _CCs[%2d] = %2d\n", d1NodeID, d1NodeID, _CCs[d1NodeID], d1NodeParentID, d1NodeParentID, _CCs[d1NodeParentID]);
    // }
    #pragma endregion //d1Node_Dist_And_CC_Recovery

    int oriEndNodeID = _csr->endNodeID - _csr->apCloneCount;
    printf("oriEndNodeID = %d\n", oriEndNodeID);
    for(int ID = _csr->startNodeID ; ID <= oriEndNodeID ; ID ++){
        printf("BC[%d] = %.2f\n", ID, _BCs[ID]);
    }
}



//D1-folding,DMF BC

//D1-folding,AP,DMF BC


//************************************************ */
//         循序_brandes + sharedBased 測試
//************************************************ */

void brandes_with_predecessors(CSR& csr, int V, float* BC) {
    double time_phase1=0.0;
    double time_phase2=0.0;
    double start_time=0.0;
    double end_time=0.0;
    
    // Allocate memory for BFS data structures
    vector<vector<int>> S(V);               // S is a 2D stack
    vector<int> sigma(V, 0);               // Sigma array
    vector<int> dist(V, -1);               // Distance array
    vector<float> delta(V, 0.0);           // Delta array
    vector<int> S_size(V, 0);              // Stack size for each level
    queue<int> f1, f2;                     // Current and Next frontier
    vector<vector<int>> predecessors(V);   // Predecessor list

    long long total_predecessor_count = 0; // To accumulate total predecessors

    

    for (int s = csr.startNodeID; s <= csr.endNodeID; ++s) {
        start_time=seconds();
        // Initialize arrays for each source node
        // sigma.assign(V, 0);   // Reset sigma to size V with all values 0
        // dist.assign(V, -1);   // Reset dist to size V with all values -1
        // delta.assign(V, 0.0); // Reset delta to size V with all values 0.0
        fill(sigma.begin(), sigma.end(), 0);
        fill(dist.begin(), dist.end(), -1);
        fill(delta.begin(), delta.end(), 0.0);
        S.assign(V, vector<int>());  // Reset S with empty vectors
        predecessors.assign(V, vector<int>());  // Reset Successors with empty vectors
        // for (auto& level : S) {
        //     level.clear();
        // }
        // for (auto& preds : predecessors) {
        //     preds.clear();
        // }
        

        sigma[s] = 1;
        dist[s] = 0;
        f1.push(s);

        int level = 0;

        // BFS forward phase
        while (!f1.empty()) {
            while (!f1.empty()) {
                int u = f1.front();
                f1.pop();
                S[level].push_back(u);

                // Traverse neighbors in CSR
                for (int i = csr.csrV[u]; i < csr.csrV[u + 1]; ++i) {
                    int w = csr.csrE[i];

                    if (dist[w] < 0) {
                        dist[w] = dist[u] + 1;
                        f2.push(w);
                    }

                    if (dist[w] == dist[u] + 1) {
                        sigma[w] += sigma[u];
                        predecessors[w].push_back(u);
                    }
                }
            }
            swap(f1, f2);
            level++;
        }
        end_time=seconds();
        time_phase1 += end_time - start_time;
        start_time=seconds();
        // Backward phase
        for (int d = level - 1; d >= 0; --d) {
            for (int w : S[d]) {
                for (int v : predecessors[w]) {
                    delta[v] += (sigma[v] / (float)sigma[w]) * (1.0 + delta[w]);
                }
                if (w != s) {
                    BC[w] += delta[w];
                }
            }
        }

        end_time=seconds();
        time_phase2 += end_time - start_time;
        // Accumulate total predecessors
        // for (const auto& preds : predecessors) {
        //     total_predecessor_count += preds.size();
        // }
    }
    // multi_time2 += multi_time1+time_phase1+time_phase2;
    printf("phase1 time: %0.6f\n", time_phase1);
    printf("phase2 time: %0.6f\n", time_phase2);
    // for(int i=csr.startNodeID;i<= csr.endNodeID; i++){
    //     printf("BC_ans[%d]: %0.2f\n",i,BC[i]);
    // }
    // Calculate and print average predecessors
    // double average_predecessors = (double)total_predecessor_count / V; //每個點當source時平均的pred edge數量。
    // cout << "avg  pred: " << average_predecessors << endl;
    // cout << "edge size: " << csr.csrESize << endl;
    // cout << "\n---------------------\n" << endl;
    // cout << "percentag edge: " << (float)average_predecessors / csr.csrESize *100<< endl; //pred edge/比例。
    // cout << "percentag node: " << (float)csr.csrVSize / csr.csrESize *100<< endl;
    // cout << "avg pred edge of node: " << (float)csr.csrESize / average_predecessors << endl;
    // cout << "avg edge of node: " << (float)csr.csrESize / csr.csrVSize << endl;
    // cout << "\n---------------------\n" << endl;
}

void brandes_with_predecessors_dynamic_check_ans(CSR csr, int V,int sourceID_test, vector<float> BC_ckeck) {
    // Allocate memory for BFS data structures
    vector<vector<int>> S(V);               // S is a 2D stack
    vector<int> sigma(V, 0);               // Sigma array
    vector<int> dist(V, -1);               // Distance array
    vector<float> delta(V, 0.0);           // Delta array
    vector<int> S_size(V, 0);              // Stack size for each level
    queue<int> f1, f2;                     // Current and Next frontier
    vector<vector<int>> predecessors(V);   // Predecessor list

    int s = sourceID_test;
    vector<float> BC(V, 0);
        // Initialize arrays for each source node
        fill(sigma.begin(), sigma.end(), 0);
        fill(dist.begin(), dist.end(), -1);
        fill(delta.begin(), delta.end(), 0.0);
        for (auto& level : S) {
            level.clear();
        }
        for (auto& preds : predecessors) {
            preds.clear();
        }

        sigma[s] = 1;
        dist[s] = 0;
        f1.push(s);

        int level = 0;

        // BFS forward phase
        while (!f1.empty()) {
            while (!f1.empty()) {
                int u = f1.front();
                f1.pop();
                S[level].push_back(u);

                // Traverse neighbors in CSR
                for (int i = csr.csrV[u]; i < csr.csrV[u + 1]; ++i) {
                    int w = csr.csrE[i];

                    if (dist[w] < 0) {
                        dist[w] = dist[u] + 1;
                        f2.push(w);
                    }

                    if (dist[w] == dist[u] + 1) {
                        sigma[w] += sigma[u];
                        predecessors[w].push_back(u);
                    }
                }
            }
            swap(f1, f2);
            level++;
        }

        // Backward phase
        for (int d = level - 1; d >= 0; --d) {
            for (int w : S[d]) {
                for (int v : predecessors[w]) {
                    delta[v] += (sigma[v] / (float)sigma[w]) * (1.0 + delta[w]);
                }
                if (w != s) {
                    BC[w] += delta[w];
                }
            }
        }

    bool flag=true;
    for(int i=0;i<delta.size();i++){
        if(delta[i]!=BC_ckeck[i]){
            printf("[ERROR] ans[%d]: %0.2f my_ans[%d]: %0.2f \n",i,delta[i],i,BC_ckeck[i]);
            flag=false;
        }
    }  
    if(flag)
        printf("[COORRECT] dynamic ans\n");

}


void computeCC_ans(struct CSR* _csr, int* _CCs){
    // showCSR(_csr);
    int* dist_arr       = (int*)calloc(sizeof(int), _csr->csrVSize);

    struct qQueue* Q    = InitqQueue();
    qInitResize(Q, _csr->csrVSize);

    int sourceID;

    #ifdef CheckDistAns
    sourceID = tempSourceID;
    int CC_ans = 0;
    #else
    sourceID = _csr->startNodeID;
    // sourceID = 1;
    #endif

    for(; sourceID <= _csr->endNodeID ; sourceID ++){
        memset(dist_arr, -1, sizeof(int) * _csr->csrVSize);
        resetQueue(Q);

        qPushBack(Q, sourceID);
        dist_arr[sourceID]  = 0;

        // #ifdef DEBUG
        // printf("\nSourceID = %2d ...\n", sourceID);
        // #endif      

        int currentNodeID   = -1;
        int neighborNodeID  = -1;


        while(!qIsEmpty(Q)){
            currentNodeID = qPopFront(Q);
            
            // #ifdef DEBUG
            // printf("%2d ===\n", currentNodeID);
            // #endif

            for(int neighborIndex = _csr->csrV[currentNodeID] ; neighborIndex < _csr->oriCsrV[currentNodeID + 1] ; neighborIndex ++){
                neighborNodeID = _csr->csrE[neighborIndex];

                // #ifdef DEBUG
                // printf("\t%2d meet %2d, dist_arr[%2d] = %2d\n", currentNodeID, neighborNodeID, neighborNodeID, dist_arr[neighborNodeID]);
                // #endif

                if(dist_arr[neighborNodeID] == -1){
                    qPushBack(Q, neighborNodeID);
                    dist_arr[neighborNodeID] = dist_arr[currentNodeID] + 1;

                    // #ifdef DEBUG
                    // printf("\tpush %2d to Q, dist_arr[%2d] = %2d\n", neighborNodeID, neighborNodeID, dist_arr[neighborNodeID]);
                    // #endif
                    
                    #ifdef CheckDistAns
                    CC_ans += dist_arr[neighborNodeID];
                    #else
                    _CCs[sourceID] += dist_arr[neighborNodeID];
                    #endif

                }
            }
        }
        
        // break;
        #ifdef CheckDistAns
        printf("CC[%d] = %d\n", tempSourceID, CC_ans);
        break;
        #endif
        
    }

    free(Q->dataArr);
    free(Q);

    #ifndef CheckDistAns
    free(dist_arr);
    #endif

    return;
}

void compute_D1_CC(struct CSR* _csr, int* _CCs){
    
    int* dist_arr = (int*)calloc(sizeof(int), _csr->csrVSize);
    struct qQueue* Q = InitqQueue();
    qInitResize(Q, _csr->csrVSize);

    D1Folding(_csr);

    #pragma region SourceTraverse_With_ff_And_represent
    //In this block, we get the CC of each remaining node in the component
    int sourceID = -1;
    for(int notD1NodeIndex = 0 ; notD1NodeIndex < _csr->ordinaryNodeCount ; notD1NodeIndex ++){
        sourceID = _csr->notD1Node[notD1NodeIndex];
        
        #ifdef DEBUG
        printf("sourceID = %2d, ff[%2d] = %2d, represent[%2d] = %2d, CC[%2d] = %d\n", sourceID, sourceID, _csr->ff[sourceID], sourceID, _csr->representNode[sourceID],sourceID,_CCs[sourceID]);
        #endif

        memset(dist_arr, -1, sizeof(int) * _csr->csrVSize);
        resetQueue(Q);
        
        qPushBack(Q, sourceID);
        dist_arr[sourceID] = 0;

        int currentNodeID   = -1;
        int neighborNodeID  = -1;

        while(!qIsEmpty(Q)){
            currentNodeID = qPopFront(Q);

            #ifdef DEBUG
            printf("currentNodeID = %2d, dist_arr[%2d] = %2d ===\n", currentNodeID, currentNodeID, dist_arr[currentNodeID]);
            #endif

            for(int neighborIndex = _csr->csrV[currentNodeID] ; neighborIndex < _csr->oriCsrV[currentNodeID + 1] ; neighborIndex ++){
                neighborNodeID = _csr->csrE[neighborIndex];

                #ifdef DEBUG
                printf("\t%2d meet %2d, dist_arr[%2d] = %2d\n", currentNodeID, neighborNodeID, neighborNodeID, dist_arr[neighborNodeID]);
                #endif

                if(dist_arr[neighborNodeID] == -1){
                    qPushBack(Q, neighborNodeID);
                    dist_arr[neighborNodeID] = dist_arr[currentNodeID] + 1;

                    //update CC (push-based) _CCs是論文中的far
                    _CCs[neighborNodeID] += _csr->ff[sourceID] + dist_arr[neighborNodeID] * _csr->representNode[sourceID];

                    #ifdef DEBUG
                    printf("\t\tpush %2d to Q, dist_arr[%2d] = %2d, _CCs[%2d] = %2d\n", neighborNodeID, neighborNodeID, dist_arr[neighborNodeID], neighborNodeID, _CCs[neighborNodeID]);
                    #endif
                }
            }
        }

        //each sourceNode update its CC with self.ff
        //因為每個點所壓縮node的距離總和(ff) 也要加上
        _CCs[sourceID] += _csr->ff[sourceID];

        // break;
    }
    #pragma endregion //SourceTraverse_With_ff_And_represent


    #pragma region d1Node_Dist_And_CC_Recovery
    // printf("_csr->totalNodeNumber = %2d\n", _csr->totalNodeNumber);
    int d1NodeID        = -1;
    int d1NodeParentID  = -1;
    for(int d1NodeIndex = _csr->degreeOneNodesQ->rear ; d1NodeIndex >= 0 ; d1NodeIndex --){
        d1NodeID        = _csr->degreeOneNodesQ->dataArr[d1NodeIndex];
        d1NodeParentID  = _csr->D1Parent[d1NodeID];
        //formula(16)的公式總結或精神
        //_CCs是論文中的far
        _CCs[d1NodeID]  = _CCs[d1NodeParentID] + _csr->totalNodeNumber - 2 * _csr->representNode[d1NodeID];
        printf("d1NodeID = %2d, _CCs[%2d] = %2d, ParentID = %2d, _CCs[%2d] = %2d\n", d1NodeID, d1NodeID, _CCs[d1NodeID], d1NodeParentID, d1NodeParentID, _CCs[d1NodeParentID]);
    }
    #pragma endregion //d1Node_Dist_And_CC_Recovery


    // #pragma region WriteCC_To_txt
    // for(int nodeID = _csr->startNodeID ; nodeID <= _csr->endNodeID ; nodeID ++){
    //     printf("CC[%d]: %d\n", nodeID, _CCs[nodeID]);
    // }
    // #pragma endregion
}




void compute_diameter(CSR* _csr) {
    int V = _csr->csrVSize;
    vector<int> dist(V, -1); // Distance array for BFS
    int diameter = 0;        // To store the graph diameter
    pair<int, int> farthestNodes; // To store the two nodes that are farthest apart

    for (int sourceID = _csr->startNodeID; sourceID <= _csr->endNodeID; sourceID++) {
        fill(dist.begin(), dist.end(), -1); // Reset distances
        queue<int> q;

        // BFS initialization
        q.push(sourceID);
        dist[sourceID] = 0;

        int maxDist = 0;
        int farthestNode = sourceID;

        // BFS to calculate shortest paths
        while (!q.empty()) {
            int currentNode = q.front();
            q.pop();

            for (int neighborIndex = _csr->csrV[currentNode]; neighborIndex < _csr->csrV[currentNode + 1]; neighborIndex++) {
                int neighborNode = _csr->csrE[neighborIndex];
                if (dist[neighborNode] == -1) { // Unvisited node
                    dist[neighborNode] = dist[currentNode] + 1;
                    q.push(neighborNode);

                    if (dist[neighborNode] > maxDist) {
                        maxDist = dist[neighborNode];
                        farthestNode = neighborNode;
                    }
                }
            }
        }

        // Update diameter and record farthest nodes
        if (maxDist > diameter) {
            diameter = maxDist;
            farthestNodes = {sourceID, farthestNode};
        }
    }

    // Output the result
    cout << "Diameter of the graph: " << diameter << endl;
    // cout << "Farthest nodes: (" << farthestNodes.first << ", " << farthestNodes.second << ")" << endl;
}

//用來分析Vw U1 U2的分布
void computeCC_shareBased_oneTraverse(struct CSR* _csr, int* _CCs){
    // showCSR(_csr);
    
    int* dist_arr           = (int*)malloc(sizeof(int) * _csr->csrVSize);
    int* neighbor_dist_ans  = (int*)malloc(sizeof(int) * _csr->csrVSize);

    struct qQueue* Q        = InitqQueue();
    qInitResize(Q, _csr->csrVSize);

    //record that nodes which haven't been source yet
    int* nodeDone = (int*)calloc(sizeof(int), _csr->csrVSize);
    
    long long Vw_count=0,U1_count=0,U2_count=0; //計算VW數量
    long long margin_count=0;
    long long shared_source_count=0;
    long long update_pred_degree=0;
    //record nodes belongs to which neighbor of source
    int* mapping_SI                 = (int*)malloc(sizeof(int) * 32);
    unsigned int* sharedBitIndex    = (unsigned int*)calloc(sizeof(unsigned int), _csr->csrVSize); //for recording blue edge bitIndex
    unsigned int* relation          = (unsigned int*)calloc(sizeof(unsigned int), _csr->csrVSize); //for recording red edge bitIndex
    
    //order ID by degree
    _csr->orderedCsrV  = (int*)calloc(sizeof(int), (_csr->csrVSize) *2);
    for(int i=_csr->startNodeID;i<=_csr->endNodeID;i++){
            _csr->orderedCsrV[i]=i;
    }
    quicksort_nodeID_with_degree(_csr->orderedCsrV, _csr->csrNodesDegree, _csr->startNodeID, _csr->endNodeID);

    for(int sourceID = _csr->startNodeID ; sourceID <= _csr->endNodeID ; sourceID ++){
    // for(int sourceIDIndex = _csr->startNodeID ; sourceIDIndex <= _csr->startNodeID ; sourceIDIndex ++){
    //     int sourceID = _csr->orderedCsrV[sourceIDIndex];
        if(nodeDone[sourceID] == 1){
            continue;
        }
        nodeDone[sourceID] = 1;

        // printf("SourceID = %2d\n", sourceID);

        memset(dist_arr, -1, sizeof(int) * _csr->csrVSize);
        
        resetQueue(Q);
        
        dist_arr[sourceID] = 0;
        qPushBack(Q, sourceID);

        int currentNodeID  = -1;
        int neighborNodeID = -1;
        int neighborIndex  = -1;
        
        //each neighbor of sourceID mapping to bit_SI, if it haven't been source yet
        int mappingCount = 0;
        for(neighborIndex = _csr->csrV[sourceID] ; neighborIndex < _csr->csrV[sourceID + 1] ; neighborIndex ++){
            neighborNodeID = _csr->csrE[neighborIndex];

            if(nodeDone[neighborNodeID] == 0){
                shared_source_count++;
                sharedBitIndex[neighborNodeID] = 1 << mappingCount;
                mapping_SI[mappingCount] = neighborNodeID;

                // printf("sharedBitIndex[%6d] = %8x,\tmapping_SI[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], mappingCount, mapping_SI[mappingCount]);
                #ifdef DEBUG
                printf("sharedBitIndex[%2d] = %8x,\tmapping_SI[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], mappingCount, mapping_SI[mappingCount]);
                #endif
                
                mappingCount ++;

                //Record to 32 bit only
                if(mappingCount == 1){
                    break;
                }

            }
        }
        
        // if(mappingCount < 3){
        //     //把sharedBitIndex重設。
        //     for(int mappingIndex = 0 ; mappingIndex < mappingCount ; mappingIndex ++){
        //         int nodeID = mapping_SI[mappingIndex];
        //         sharedBitIndex[nodeID] = 0;
        //     }
        //     memset(mapping_SI, 0, sizeof(int) * 32);

        //     #pragma region Ordinary_BFS_Forward_Traverse

        //     #ifdef DEBUG
        //     printf("\n####      Source %2d Ordinary BFS Traverse      ####\n\n", sourceID);
        //     #endif

        //     while(!qIsEmpty(Q)){
        //         currentNodeID = qPopFront(Q);

        //         #ifdef DEBUG
        //         printf("\tcurrentNodeID = %2d ... dist = %2d\n", currentNodeID, dist_arr[currentNodeID]);
        //         #endif

        //         for(neighborIndex = _csr->csrV[currentNodeID] ; neighborIndex < _csr->csrV[currentNodeID + 1] ; neighborIndex ++){
        //             neighborNodeID = _csr->csrE[neighborIndex];

        //             if(dist_arr[neighborNodeID] == -1){
        //                 qPushBack(Q, neighborNodeID);
        //                 dist_arr[neighborNodeID] = dist_arr[currentNodeID] + 1;

        //                 #ifdef DEBUG
        //                 printf("\t\t[1]dist[%2d] = %2d\n", neighborNodeID, dist_arr[neighborNodeID]);
        //                 #endif
        //             }
        //         }
        //     }
        //     #pragma endregion //Ordinary_BFS_Forward_Traverse



        //     #pragma region distAccumulation_pushBased
        //     //Update CC in the way of pushing is better for parallelism because of the it will not need to wait atomic operation on single address,
        //     //it can update all value in each CC address in O(1) time.
        //     for(int nodeID = _csr->startNodeID ; nodeID <= _csr->endNodeID ; nodeID ++){
        //         _CCs[nodeID] += dist_arr[nodeID];
        //     }
        //     #pragma endregion //distAccumulation_pushBased



        //     #pragma region checkingDistAns
        //     #ifdef CheckDistAns
        //     // CC_CheckDistAns(_csr, _CCs, sourceID, dist_arr);
        //     #endif

        //     #ifdef CheckCC_Ans
        //     dynamic_CC_trace_Ans(_csr, _CCs, sourceID);
        //     #endif

        //     #pragma endregion //checkingDistAns

        // }
        // else{

            #pragma region SourceTraverse
            //main source traversal : for getting the dist of each node from source
            // #ifdef DEBUG
            // printf("\n####      Source %2d First traverse...      ####\n\n", sourceID);
            // #endif

            while(!qIsEmpty(Q)){
                currentNodeID = qPopFront(Q);

                #ifdef DEBUG
                printf("currentNodeID = %2d ... dist = %2d\n", currentNodeID, dist_arr[currentNodeID]);
                #endif

                

                for(neighborIndex = _csr->csrV[currentNodeID] ; neighborIndex < _csr->csrV[currentNodeID + 1] ; neighborIndex ++){
                    neighborNodeID = _csr->csrE[neighborIndex];

                    if(dist_arr[neighborNodeID] == -1){//traverse new succesor and record its SI
                        qPushBack(Q, neighborNodeID);
                        dist_arr[neighborNodeID] = dist_arr[currentNodeID] + 1;
                        sharedBitIndex[neighborNodeID] |= sharedBitIndex[currentNodeID];

                        #ifdef DEBUG
                        printf("\t[1]unvisited_SI[%2d] => %2x, dist[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], neighborNodeID, dist_arr[neighborNodeID]);
                        #endif
                        
                        // if(sourceID == 5 && (neighborNodeID == 4 || neighborNodeID == 6)){
                        //     printf("\t[1]currentNodeID = %2d(dist %2d, SI %2x), neighborNodeID = %d(dist %2d, SI %2x)\n", currentNodeID, dist_arr[currentNodeID], sharedBitIndex[currentNodeID], neighborNodeID, dist_arr[neighborNodeID], sharedBitIndex[neighborNodeID]);
                        // }
                    }
                    else if(dist_arr[neighborNodeID] == dist_arr[currentNodeID] + 1){ //traverse to discovered succesor and record its SI
                        sharedBitIndex[neighborNodeID] |= sharedBitIndex[currentNodeID];    
                        
                        #ifdef DEBUG
                        printf("\t[2]visited_SI[%2d] => %2x, dist[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], neighborNodeID, dist_arr[neighborNodeID]);
                        #endif

                        // if(sourceID == 5 && (neighborNodeID == 4 || neighborNodeID == 6)){
                        //     printf("\t[2]currentNodeID = %2d(dist %2d, SI %2x), neighborNodeID = %d(dist %2d, SI %2x)\n", currentNodeID, dist_arr[currentNodeID], sharedBitIndex[currentNodeID], neighborNodeID, dist_arr[neighborNodeID], sharedBitIndex[neighborNodeID]);
                        // }
                    }
                    else if(dist_arr[neighborNodeID] == dist_arr[currentNodeID] && currentNodeID < neighborNodeID){ //traverse to discovered neighbor which is at same level as currentNodeID
                        relation[currentNodeID]     |= sharedBitIndex[neighborNodeID] & (~sharedBitIndex[currentNodeID]);
                        relation[neighborNodeID]    |= sharedBitIndex[currentNodeID]  & (~sharedBitIndex[neighborNodeID]);

                        #ifdef DEBUG
                        printf("\t[3]Red edge found(%2d, %2d), ", currentNodeID, neighborNodeID);
                        printf("relation[%2d] = %2x, relation[%2d] = %2x\n", currentNodeID, relation[currentNodeID], neighborNodeID, relation[neighborNodeID]);
                        #endif

                        // if(sourceID == 5 && (neighborNodeID == 4 || neighborNodeID == 6)){
                        //     printf("\t[3]currentNodeID = %2d(dist %2d, re %2x), neighborNodeID = %d(dist %2d, re %2x)\n", currentNodeID, dist_arr[currentNodeID], relation[currentNodeID], neighborNodeID, dist_arr[neighborNodeID], relation[neighborNodeID]);
                        // }
                    }//&& relation[neighborNodeID]
                    else if(dist_arr[neighborNodeID] == dist_arr[currentNodeID] - 1){ //traverse to discovered neighbor which is at same level as currentNodeID
                        // relation[currentNodeID]     |= relation[neighborNodeID];
                        //這一步是讓SI 跟R只會顯示一個 SI:1 R:0 幫助最終判鄰居是否相同
                        relation[currentNodeID]     = (relation[currentNodeID]|relation[neighborNodeID]) & (~sharedBitIndex[currentNodeID] );
                        #ifdef DEBUG
                        printf("\t[3]Red edge found(%2d, %2d), ", currentNodeID, neighborNodeID);
                        printf("relation[%2d] = %2x, relation[%2d] = %2x\n", currentNodeID, relation[currentNodeID], neighborNodeID, relation[neighborNodeID]);
                        #endif

                        // if(sourceID == 5 && (neighborNodeID == 4 || neighborNodeID == 6)){
                        //     printf("\t[3]currentNodeID = %2d(dist %2d, re %2x), neighborNodeID = %d(dist %2d, re %2x)\n", currentNodeID, dist_arr[currentNodeID], relation[currentNodeID], neighborNodeID, dist_arr[neighborNodeID], relation[neighborNodeID]);
                        // }
                    }
                }
            }

            //second source traversal : for handle the red edge
            // #ifdef DEBUG
            // printf("\n####      Source %2d Second traverse...      ####\n\n", sourceID);
            // #endif

            // Q->front = 0;
            // while(!qIsEmpty(Q)){
            //     currentNodeID = qPopFront(Q);

            //     #ifdef DEBUG
            //     printf("currentNodeID = %2d ... dist = %2d ... relation = %x\n", currentNodeID, dist_arr[currentNodeID], relation[currentNodeID]);
            //     #endif

            //     for(neighborIndex = _csr->csrV[currentNodeID] ; neighborIndex < _csr->csrV[currentNodeID + 1] ; neighborIndex ++){
            //         neighborNodeID = _csr->csrE[neighborIndex];

            //         if(dist_arr[neighborNodeID] == dist_arr[currentNodeID] + 1){
            //             relation[neighborNodeID] |= relation[currentNodeID];
                        
            //             #ifdef DEBUG
            //             printf("\t[4]relation[%2d] = %2x\n", neighborNodeID, relation[neighborNodeID]);
            //             #endif

            //             // if(sourceID == 5 && (neighborNodeID == 4 || neighborNodeID == 6)){
            //             //     printf("\t[4]currentNodeID = %2d(dist %2d, re %2x), neighborNodeID = %d(dist %2d, re %2x)\n", currentNodeID, dist_arr[currentNodeID], relation[currentNodeID], neighborNodeID, dist_arr[neighborNodeID], relation[neighborNodeID]);
            //             // }
            //         }
            //     }
            // }
            #pragma endregion //SourceTraverse

            #pragma region sourceDistAccumulation_pushBased
            for(int nodeID = _csr->startNodeID ; nodeID <= _csr->endNodeID ; nodeID ++){
                _CCs[nodeID] += dist_arr[nodeID];
            }

            #ifdef CheckCC_Ans
            dynamic_CC_trace_Ans(_csr, _CCs, sourceID);
            #endif

            #pragma endregion //distAccumulation_pushBased


            #pragma region neighborOfSource_GetDist
            //recover the data from source to neighbor of source
            for(int sourceNeighborIndex = 0 ; sourceNeighborIndex < mappingCount ; sourceNeighborIndex ++){
                memset(neighbor_dist_ans, 0, sizeof(int));

                int sourceNeighborID = mapping_SI[sourceNeighborIndex];
                unsigned int bit_SI = 1 << sourceNeighborIndex;

                nodeDone[sourceNeighborID] = 1;

                #ifdef DEBUG
                printf("\nnextBFS = %2d, bit_SI = %x\n", sourceNeighborID, bit_SI);
                #endif

                for(int nodeID = _csr->startNodeID ; nodeID <= _csr->endNodeID ; nodeID ++){
                    int nodeID_type=-1; //1: Vw -- 2:U1 -- 3:U2 
                    if((sharedBitIndex[nodeID] & bit_SI) > 0){ //要括號，因為"比大小優先於邏輯運算"
                        neighbor_dist_ans[nodeID] = dist_arr[nodeID] - 1;
                        Vw_count++;
                        nodeID_type=1;
                        // printf("\t[5]neighbor_dist_ans[%2d] = %2d, SI[%2d] = %x\n", nodeID, neighbor_dist_ans[nodeID], nodeID, sharedBitIndex[nodeID]);
                    }
                    else{
                        neighbor_dist_ans[nodeID] = dist_arr[nodeID] + 1;
                        // printf("\t[6]neighbor_dist_ans[%2d] = %2d, SI[%2d] = %x\n", nodeID, neighbor_dist_ans[nodeID], nodeID, sharedBitIndex[nodeID]);
                        if((relation[nodeID] & bit_SI) > 0){
                            U1_count++;
                            neighbor_dist_ans[nodeID] --;
                            nodeID_type=2;
                            // printf("\t[7]neighbor_dist_ans[%2d] = %2d, relation[%2d] = %x\n", nodeID, neighbor_dist_ans[nodeID], nodeID, relation[nodeID]);
                        }else{
                            nodeID_type=3;
                            U2_count++;
                        }
                    }

                    for(neighborIndex = _csr->csrV[nodeID] ; neighborIndex < _csr->csrV[nodeID + 1] ; neighborIndex ++){
                        neighborNodeID = _csr->csrE[neighborIndex];
                        int neighborID_type=-1;
                        if((sharedBitIndex[neighborNodeID] & bit_SI) > 0){ //要括號，因為"比大小優先於邏輯運算"
                            neighborID_type=1;
                            // printf("\t[5]neighbor_dist_ans[%2d] = %2d, SI[%2d] = %x\n", nodeID, neighbor_dist_ans[nodeID], nodeID, sharedBitIndex[nodeID]);
                        }
                        else{
                            // printf("\t[6]neighbor_dist_ans[%2d] = %2d, SI[%2d] = %x\n", nodeID, neighbor_dist_ans[nodeID], nodeID, sharedBitIndex[nodeID]);
                            if((relation[nodeID] & bit_SI) > 0){
                                neighborID_type=2;
                                // printf("\t[7]neighbor_dist_ans[%2d] = %2d, relation[%2d] = %x\n", nodeID, neighbor_dist_ans[nodeID], nodeID, relation[nodeID]);
                            }else{
                                neighborID_type=3;
                            }
                        }
                        if(nodeID_type!=neighborID_type){
                            update_pred_degree += _csr->csrNodesDegree[nodeID];
                            margin_count++;
                            break;
                        }
                    }
                }

                #pragma region neighborDistAccumulation_pushBased
                for(int nodeID = _csr->startNodeID ; nodeID <= _csr->endNodeID ; nodeID ++){
                    _CCs[nodeID] += neighbor_dist_ans[nodeID];
                }
                #pragma endregion //neighborDistAccumulation_pushBased

            }


            
            #pragma endregion //neighborOfSource_GetDist

            //reset the SI & relation arrays
            memset(relation, 0, sizeof(unsigned int) * _csr->csrVSize);
            memset(sharedBitIndex, 0, sizeof(unsigned int) * _csr->csrVSize);
        // }
    }
    float avg_margin_node = margin_count/shared_source_count;
    float avg_pred_update_degree = update_pred_degree/shared_source_count;
    float avg_pred_update_degree_ratio = avg_pred_update_degree/_csr->csrESize;

    float avg_Vw_node = Vw_count/shared_source_count;
    float avg_U1_node = U1_count/shared_source_count;
    float avg_U2_node = U2_count/shared_source_count;
    // cout<<"avg_margin_node: "<<avg_margin_node<<" marg_node_ratio: "<<avg_margin_node/_csr->csrVSize*100<<endl;
    // cout<<"avg_marg_edge% : "<<avg_pred_update_degree<<" marg_edge_ratio: "<<avg_pred_update_degree_ratio*100<<endl;
    // cout<<"avg_Vw_node:     "<<avg_Vw_node<<" Vw_node_ratio: "<<avg_Vw_node/_csr->csrVSize*100<<endl;
    // cout<<"avg_U1_node:     "<<avg_U1_node<<" U1_node_ratio: "<<avg_U1_node/_csr->csrVSize*100<<endl;
    // cout<<"avg_U2_node:     "<<avg_U2_node<<" U2_node_ratio: "<<avg_U2_node/_csr->csrVSize*100<<endl;

    // printf("\n\n[CC_sharedBased] Done!\n");
}

//使用shared 和successor來BC (循序版本，平行模板)
void computeBC_shareBased_Successor_MS( CSR* _csr, float* _BCs){
    // showCSR(_csr);
    int V =  _csr->csrVSize;

    struct qQueue* Q        = InitqQueue();
    qInitResize(Q, _csr->csrVSize);

    //record that nodes which haven't been source yet
    int* nodeDone = (int*)calloc(sizeof(int), _csr->csrVSize);
    
    //record nodes belongs to which neighbor of source
    int  mappingCount_max           = 32; //最大可以設64
    int* mapping_SI                 = (int*)malloc(sizeof(int) * 32);
    vector<unsigned int> sharedBitIndex(_csr->csrVSize, 0); // for recording blue edge bitIndex
    vector<unsigned int> relation(_csr->csrVSize, 0);       // for recording red edge bitIndex
    vector<unsigned long long> sameIndex(_csr->csrVSize, 0);// for recording blue edge BC_sameIndex

    // Allocate memory for BFS data structures
    vector<vector<int>> S(V);              // S is a 2D stack
    vector<int> sigma(V, 0);               // Sigma array
    vector<int> dist(V, -1);               // Distance array
    vector<float> delta(V, 0.0);           // Delta array
    vector<vector<int>> Successors(V);     // Successors list 

    //可重複使用的空間
    vector<int> S_size(V, 0);              // Stack size for each level
    queue<int> f1, f2;                     // Current and Next frontier
    queue<Q_struct>  Q_f1,Q_f2;
    vector<Q_struct> Q_f2_temp(V);
    //N(s)鄰居的traverse 路徑數量、距離、Suc、Sigma
    vector<vector<int>>   Ns_dist  (mappingCount_max, vector<int>(V,-1));               // Distance array
    vector<vector<int>>   Ns_sigma (mappingCount_max, vector<int>(V,0));
    vector<vector<float>> Ns_delta (mappingCount_max, vector<float>(V,0.0));            // Delta array
    vector<vector<vector<int>>> Ns_Successors(mappingCount_max,vector<vector<int>>(V)); // Successors [][][]: [source的Suc] [V] [V的Suc] 
    vector<vector<Q_struct>> Ns_S(V);               // S is a 2D stack [level][node]...


    //Suc與edge數據
    unsigned long Suc_times=0;
    unsigned long edge_times=0;
    //用degree做排序 大->小
    _csr->orderedCsrV  = (int*)calloc(sizeof(int), (_csr->csrVSize) *2);
    for(int i=_csr->startNodeID;i<=_csr->endNodeID;i++){
            _csr->orderedCsrV[i]=i;
    }
    quicksort_nodeID_with_degree(_csr->orderedCsrV, _csr->csrNodesDegree, _csr->startNodeID, _csr->endNodeID);

    // for(int sourceID = _csr->startNodeID ; sourceID <= _csr->endNodeID ; sourceID ++){
    


    for(int sourceIDIndex = _csr->startNodeID ; sourceIDIndex <= _csr->endNodeID ; sourceIDIndex ++){
        int sourceID = _csr->orderedCsrV[sourceIDIndex];
        if(nodeDone[sourceID] == 1){
            continue;
        }
        nodeDone[sourceID] = 1;

        // printf("SourceID = %2d\n", sourceID);

        int currentNodeID  = -1;
        int neighborNodeID = -1;
        int neighborIndex  = -1;
        
        //each neighbor of sourceID mapping to bit_SI, if it haven't been source yet
        //挑選鄰居
        int mappingCount = 0;
        for(neighborIndex = _csr->csrV[sourceID] ; neighborIndex < _csr->csrV[sourceID + 1] ; neighborIndex ++){
            neighborNodeID = _csr->csrE[neighborIndex];

            if(nodeDone[neighborNodeID] == 0){
                //Record to 32 bit only
                if(mappingCount == mappingCount_max){
                    break;
                }

                sharedBitIndex[neighborNodeID] = 1 << mappingCount;
                mapping_SI[mappingCount] = neighborNodeID;
                nodeDone[neighborNodeID] = 1;

                // printf("sharedBitIndex[%6d] = %8x,\tmapping_SI[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], mappingCount, mapping_SI[mappingCount]);
                #ifdef DEBUG
                printf("sharedBitIndex[%2d] = %8x,\tmapping_SI[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], mappingCount, mapping_SI[mappingCount]);
                #endif
                
                mappingCount ++;
            }
        }

        //**********************************/
        //   Source forward traverse
        //**********************************/
        #pragma region SourceTraverse //forward traverse
        
        // Initialize arrays for each source node
        sigma.assign(V, 0);   // Reset sigma to size V with all values 0
        dist.assign(V, -1);   // Reset dist to size V with all values -1
        delta.assign(V, 0.0); // Reset delta to size V with all values 0.0
        S.assign(V, vector<int>());  // Reset S with empty vectors
        Successors.assign(V, vector<int>());  // Reset Successors with empty vectors

        sigma[sourceID] = 1;
        dist[sourceID] = 0;
        f1.push(sourceID);
        int level = 0;
        // cout<<"source ID: "<<sourceID<<endl;
        // BFS forward phase
        
        while (!f1.empty()){
            while (!f1.empty()) {
                int currentNodeID = f1.front();
                f1.pop();
                S[level].push_back(currentNodeID);
                #ifdef DEBUG
                printf("currentNodeID = %2d ... dist = %2d\n", currentNodeID, dist[currentNodeID]);
                #endif
                // Traverse neighbors in CSR
                for(neighborIndex = _csr->csrV[currentNodeID] ; neighborIndex < _csr->csrV[currentNodeID + 1] ; neighborIndex ++) {
                    int neighborNodeID =  _csr->csrE[neighborIndex];

                    if (dist[neighborNodeID] == -1) {
                        dist[neighborNodeID] = dist[currentNodeID] + 1;
                        sigma[neighborNodeID] += sigma[currentNodeID];
                        sharedBitIndex[neighborNodeID] |= sharedBitIndex[currentNodeID];
                        Successors[currentNodeID].push_back(neighborNodeID);
                        f2.push(neighborNodeID);
                        #ifdef DEBUG
                        printf("\t[1]unvisited_SI[%2d] => %2x, dist[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], neighborNodeID, dist[neighborNodeID]);
                        #endif
                    }
                    else if (dist[neighborNodeID] == dist[currentNodeID] + 1) {
                        sharedBitIndex[neighborNodeID] |= sharedBitIndex[currentNodeID];
                        Successors[currentNodeID].push_back(neighborNodeID);
                        sigma[neighborNodeID] += sigma[currentNodeID];
                        #ifdef DEBUG
                        printf("\t[2]unvisited_SI[%2d] => %2x, dist[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], neighborNodeID, dist[neighborNodeID]);
                        #endif
                    }
                    else if(dist[neighborNodeID] == dist[currentNodeID] && currentNodeID < neighborNodeID){ //traverse to discovered neighbor which is at same level as currentNodeID
                        relation[currentNodeID]     |= sharedBitIndex[neighborNodeID] & (~sharedBitIndex[currentNodeID]);
                        relation[neighborNodeID]    |= sharedBitIndex[currentNodeID]  & (~sharedBitIndex[neighborNodeID]);
                        #ifdef DEBUG
                        printf("\t[3]Red edge found(%2d, %2d), ", currentNodeID, neighborNodeID);
                        printf("relation[%2d] = %2x, relation[%2d] = %2x\n", currentNodeID, relation[currentNodeID], neighborNodeID, relation[neighborNodeID]);
                        #endif

                    }//&& relation[neighborNodeID]
                    else if(dist[neighborNodeID] == dist[currentNodeID] - 1){
                        //這一步是讓SI 跟R只會顯示一個 SI:1 R:0 幫助最終判鄰居是否相同 
                        relation[currentNodeID]     = (relation[currentNodeID]|relation[neighborNodeID]) & (~sharedBitIndex[currentNodeID] );
                        #ifdef DEBUG
                        printf("\t[4]Red edge found(%2d, %2d), ", currentNodeID, neighborNodeID);
                        printf("relation[%2d] = %2x, relation[%2d] = %2x\n", currentNodeID, relation[currentNodeID], neighborNodeID, relation[neighborNodeID]);
                        #endif
                    }
                }

            }
            swap(f1, f2);
            level++;
        }



        #ifdef DEBUG
            //printf successor
            printf("\n");
            for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
                printf("Successor(%d) sigma(%d) ",i,sigma[i] );
                for (int j=0;j<Successors[i].size();j++){
                    printf("%d ",Successors[i][j]);
                }
                printf("\n");
            }

            for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
                printf("%d: SI(%d) R(%d)\n",i,sharedBitIndex[i],relation[i] );
            }
        #endif

        #pragma endregion
        //**********************************/
        //   Source backward traverse
        //**********************************/
        #pragma region sourceDistAccumulation_pushBased
            // Backward phase
            // for (int d = level - 1; d > 0; --d) {
            //     for (int w : S[d]) {
            //         for (int v : Successors[w]) {
            //             delta[w] += (sigma[w] / (float)sigma[v]) * (1.0 + delta[v]);
            //         }
            //         _BCs[w] += delta[w];
            //     }
            // }
            for (int d = level - 1; d >= 0; --d) {
                for (int node : S[d]) { 
                    for(neighborIndex = _csr->csrV[node] ; neighborIndex < _csr->csrV[node + 1] ; neighborIndex ++) {
                        int neighborNodeID =  _csr->csrE[neighborIndex];

                        //1:代表我有不一樣的鄰居(edge)  0:代表node的鄰居都是一樣的顏色(Suc)
                        sameIndex[node] |= (sharedBitIndex[node]^sharedBitIndex[neighborNodeID])|(relation[node]^relation[neighborNodeID]);
                        
                        //normal backward
                        if(dist[neighborNodeID] == dist[node] + 1 )
                            delta[node] += (sigma[node] / (float)sigma[neighborNodeID]) * (1.0 + delta[neighborNodeID]);
                        
                    }
                    ///normal BC accumulation
                    if (node != sourceID) {
                        // printf("delta[%d]: (%.2f)\n",node,delta[node]);
                        _BCs[node] += delta[node];
                    }
                }
            }
        #pragma endregion //distAccumulation_pushBased
        
        // for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
        //     printf("_BC_first[%d]: (%.2f)\n",i,_BCs[i]);
        // }
        
        #ifdef DEBUG //print SI、R、sameIndex
            for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
                printf("_BC[%d]: (%.2f)\n",i,_BCs[i]);
            }
            for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
                printf(" sameIndex[%d]: ",i);
                printbinary(sameIndex[i],mappingCount);
                printf("\tSI[%d]: ",i);
                printbinary(sharedBitIndex[i],mappingCount);
                printf("\t R[%d]: ",i);
                printbinary(relation[i],mappingCount);
            }
        #endif

        //****************************************/
        //   N(s) forward traverse(鄰居當Source)
        //****************************************/
        //初始Ns的資訊
        if(mappingCount){
            Ns_sigma.assign(mappingCount, vector<int>(V, 0));  // Reset Ns_sigma with zeros
            Ns_dist.assign(mappingCount, vector<int>(V, -1));  // Reset Ns_dist with -1 (unvisited)
            Ns_delta.assign(mappingCount, vector<float>(V, 0.0)); // Reset Ns_delta with 0.0
            Ns_Successors.assign(mappingCount, vector<vector<int>>(V)); // Clear and reset Ns_Successors
            Q_f2_temp.assign(V, Q_struct{0, -1}); // Reset Q_f2_temp using assign
            Ns_S.assign(V, vector<Q_struct>());  // Reset Ns_S with empty vectors
        }
        

        //這部分使用 multi-source實作
        #pragma region third_part

        // printf("---------initial Queue-----------\n");
        for (int i = 0; i < mappingCount; i++) {
            int sourceNode = mapping_SI[i];
            Ns_sigma[i][sourceNode] = 1.0;
            Ns_dist[i][sourceNode]  = 0;
            Q_f1.push({(1ULL << i),sourceNode});

            //printf
            // printf("f1[%d]:[%d,",i,sourceNode);
            // printbinary((1ULL << i),mappingCount);
        }
        // printf("----------------0-----------------\n");
        
        level=0;
        // BFS forward phase
        while (!Q_f1.empty()){
            while (!Q_f1.empty()) {
                int currentNodeID   = Q_f1.front().nodeID;
                uint64_t traverse_S = Q_f1.front().traverse_S;
                Ns_S[level].push_back(Q_f1.front());
                Q_f1.pop();
                //mapping
                // Suc/edge neighbor
                for (auto multi_node = 0; multi_node < mappingCount; multi_node++) {
                    // uint64_t bit_mask=(1ULL << multi_node);
                    if (traverse_S & (1ULL << multi_node)){ //該node在這層traverse到
                        if( (sameIndex[currentNodeID] & (1ULL << multi_node)) == 0 ){ //Suc_traverse
                            Suc_times++;
                            // printf("[%d] Suc_traverse\n",  currentNodeID);
                            for (auto SucNodeInDex = 0; SucNodeInDex < Successors[currentNodeID].size(); SucNodeInDex++) {
                                int SucNodeID = Successors[currentNodeID][SucNodeInDex];

                                Ns_dist [multi_node][SucNodeID]  = Ns_dist[multi_node][currentNodeID] +1;
                                Ns_sigma[multi_node][SucNodeID] += Ns_sigma[multi_node][currentNodeID];
                                Ns_Successors[multi_node][currentNodeID].push_back(SucNodeID);
                                //push node into Q_f2_temp
                                Q_f2_temp[SucNodeID].nodeID=SucNodeID;
                                Q_f2_temp[SucNodeID].traverse_S|= (traverse_S & (1ULL << multi_node));
                                // printf("[%d] push in Suc\n",  SucNodeID);
                            }
                        }else{//edge_traverse
                            edge_times++;
                            // printf("[%d] edge_traverse\n",  currentNodeID);
                            for (auto neighborIndex = _csr->csrV[currentNodeID]; neighborIndex < _csr->csrV[currentNodeID + 1]; neighborIndex++) {
                                int neighborNodeID = _csr->csrE[neighborIndex];
                                //累加路徑數量、dist
                                if (Ns_dist[multi_node][neighborNodeID] == -1) {
                                    Ns_Successors[multi_node][currentNodeID].push_back(neighborNodeID);
                                    
                                    Ns_dist [multi_node][neighborNodeID]  = Ns_dist[multi_node][currentNodeID] + 1;
                                    Ns_sigma[multi_node][neighborNodeID] += Ns_sigma[multi_node][currentNodeID];

                                    // printf("[%d] push in edge\n",  neighborNodeID);
                                    Q_f2_temp[neighborNodeID].nodeID=neighborNodeID;
                                    Q_f2_temp[neighborNodeID].traverse_S|= (traverse_S & (1ULL << multi_node));
                                }
                                else if (Ns_dist[multi_node][neighborNodeID] == Ns_dist[multi_node][currentNodeID] + 1) {
                                    Ns_Successors[multi_node][currentNodeID].push_back(neighborNodeID);
                                    Ns_sigma[multi_node][neighborNodeID] += Ns_sigma[multi_node][currentNodeID];
                                }
                            }
                        }

                    }
                }
                
            }
            
            //在這定義Q_f2_temp(vector)搜尋V個node，把對應index的node放進Q_f2
            for(int insert_q = _csr->startNodeID ; insert_q <=_csr->endNodeID ; insert_q++ ){
                if(Q_f2_temp[insert_q].nodeID!=-1){
                    Q_f2.push(Q_f2_temp[insert_q]);
                    //print DeBUG
                    // printf("Q_f2[%d]:[%d,",insert_q, Q_f2_temp[insert_q].nodeID);
                    // printbinary(Q_f2_temp[insert_q].traverse_S,mappingCount);
                }
            }
            // printf("----------------%d----------------\n",(level+1));
            
            //做Q_f1(空的), Q_f2交換
            swap(Q_f1, Q_f2);
            Q_f2_temp.assign(V, Q_struct{0, -1}); // Reset Q_f2_temp using assign
            level++;
        }

        //print Ns dist path Suc
        // for(int i=_csr->startNodeID; i<=_csr->endNodeID;i++){
        //     printf("dist{%d}: ",i);
        //     for(int j=0; j<Ns_dist[i].size();j++){
        //         printf("%d ",Ns_dist[i][j]);
        //     }
        //     printf("\n");
        // }
        
        // for(int i=_csr->startNodeID; i<=_csr->endNodeID;i++){
        //     printf("sigma{%d}: ",i);
        //     for(int j=0; j<Ns_sigma[i].size();j++){
        //         printf("%d ",Ns_sigma[i][j]);
        //     }
        //     printf("\n");
        // }

        // for(int i=0; i<mappingCount;i++){
        //     printf("Successors{%d}: \n",mapping_SI[i]);
        //     for(int j=_csr->startNodeID; j<=_csr->endNodeID;j++){
        //         printf("{%d}: ",j);
        //         for(auto& x:Ns_Successors[i][j]){
        //             printf("%d ",x);
        //         }
        //         printf("\n");
        //     }
        //     printf("\n");
        // }


        #pragma endregion //neighborOfSource_GetDist

        //****************************************/
        //   N(s) backward traverse
        //****************************************/
        #pragma region forth_part
        for (int d = level - 1; d > 0; --d) {
            // printf("--------back level: %d------------\n",d);
            for (auto node : Ns_S[d]) {
                for(auto multi_node = 0; multi_node < mappingCount; multi_node++){
                    uint64_t bit_mask = (1ULL<<multi_node);
                    if(node.traverse_S & bit_mask){
                        for (auto SucNodeInDex = 0; SucNodeInDex < Ns_Successors[multi_node][node.nodeID].size(); SucNodeInDex++) {
                            int SucNodeID = Ns_Successors[multi_node][node.nodeID][SucNodeInDex];
                            Ns_delta[multi_node][node.nodeID] += (Ns_sigma[multi_node][node.nodeID] / (float)Ns_sigma[multi_node][SucNodeID]) * (1.0 + Ns_delta[multi_node][SucNodeID]);
                        }
                        
                        // if(node.nodeID != mapping_SI[multi_node]){
                            // printf("Ns_delta[%d][%d]: (%.2f)\n",node.nodeID,mapping_SI[multi_node],Ns_delta[node.nodeID][multi_node]);
                            _BCs[node.nodeID] += Ns_delta[multi_node][node.nodeID];
                        // }
                        
                    }
                }
                
            }
        }

       
        #pragma endregion //neighborOfSource_GetDist

        //reset the SI & relation arrays
        relation.assign(_csr->csrVSize, 0);       // Reset relation to size _csr->csrVSize with all values 0
        sharedBitIndex.assign(_csr->csrVSize, 0); // Reset sharedBitIndex to size _csr->csrVSize with all values 0
        sameIndex.assign(_csr->csrVSize, 0);      // Reset sameIndex to size _csr->csrVSize with all values 0

    }

    unsigned long total_times=edge_times+Suc_times;
    printf("suc  traverse: %0.2f\n", Suc_times/(float)total_times);
    printf("edge traverse: %0.2f\n", edge_times/(float)total_times);
    
    // for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
    //     printf("_BC[%d]: (%.2f)\n",i,_BCs[i]);
    // }

    // printf("\n[BC_sharedBased] Done!\n");
}

 #pragma region brandes
//使用shared 和successor來BC (循序版本，Ns分別traverse)
void computeBC_shareBased_Successor_SS( CSR* _csr, float* _BCs){
    //時間量測
    double time_sort=0.0;
    double time_phase1=0.0;
    double time_phase2=0.0;
    double time_phase3=0.0;
    double time_phase4=0.0;
    double time_phase4_1=0.0;
    double start_time=0.0;
    double end_time=0.0;


    // multi_time1 = seconds();
    start_time=seconds();
    // showCSR(_csr);
    int V =  _csr->csrVSize;


    //record that nodes which haven't been source yet
    int* nodeDone = (int*)calloc(sizeof(int), _csr->csrVSize);
    
    //record nodes belongs to which neighbor of source
    int  mappingCount_max           = 32; //最大可以設32
    int* mapping_SI                 = (int*)malloc(sizeof(int) * mappingCount_max);
    vector<unsigned int> sharedBitIndex(_csr->csrVSize, 0); // for recording blue edge bitIndex
    vector<unsigned int> relation(_csr->csrVSize, 0);       // for recording red edge bitIndex
    vector<unsigned int> sameIndex(_csr->csrVSize, 0);// for recording blue edge BC_sameIndex

    // Allocate memory for BFS data structures 
    vector<int> sigma(V, 0);               // Sigma array
    vector<int> dist(V, -1);               // Distance array
    vector<float> delta(V, 0.0);           // Delta array
    vector<vector<int>> Successors(V);     // Successors list 
    // vector<float> _BCs_check(V, 0.0);           // Delta array
    //可重複使用的空間
    vector<vector<int>> S(V);              // S is a 2D stack
    queue<int> f1, f2;                     // Current and Next frontier
    //N(s)鄰居的traverse 路徑數量、距離、Suc、Sigma
    vector<int>  Ns_dist  (V, -1);            // Distance array
    vector<int>  Ns_sigma (V, 0);             // Sigma array
    vector<float> Ns_delta(V, 0.0);           // Delta array
    vector<vector<int>> Ns_Successors(V);     // Successors [][]: [V] [V的Suc] 
    
    //Suc與edge數據
    unsigned long Suc_times=0;
    unsigned long edge_times=0;
    long S_node = 0;
    long Ns_node = 0;


    
    
    //用degree做排序 大->小
    _csr->orderedCsrV  = (int*)calloc(sizeof(int), (_csr->csrVSize) *2);
    for(int i=_csr->startNodeID;i<=_csr->endNodeID;i++){
            _csr->orderedCsrV[i]=i;
    }
    quicksort_nodeID_with_degree(_csr->orderedCsrV, _csr->csrNodesDegree, _csr->startNodeID, _csr->endNodeID);
    
    // printf("orderedCsrV: ");
    // for(int i=_csr->startNodeID;i<=_csr->endNodeID;i++){
    //         printf("[%d][%d] ",i,_csr->orderedCsrV[i]);
    // }
    // printf("\n");

    end_time = seconds();
    time_sort += end_time - start_time;
    // for(int sourceID = _csr->startNodeID ; sourceID <= _csr->endNodeID ; sourceID ++){
    


    for(int sourceIDIndex = _csr->startNodeID ; sourceIDIndex <= _csr->endNodeID ; sourceIDIndex ++){
        
        int sourceID = _csr->orderedCsrV[sourceIDIndex];
        
        if(nodeDone[sourceID] == 1){
            continue;
        }
        nodeDone[sourceID] = 1;
        
        start_time=seconds();
        // printf("SourceID = %2d\n", sourceID);
        S_node++;

        int currentNodeID=-1;
        int neighborNodeID=-1;
        int neighborIndex =-1;
        //each neighbor of sourceID mapping to bit_SI, if it haven't been source yet
        //挑選鄰居
        int mappingCount = 0;
        for(neighborIndex = _csr->csrV[sourceID] ; neighborIndex < _csr->csrV[sourceID + 1] ; neighborIndex ++){
            neighborNodeID = _csr->csrE[neighborIndex];

            if(nodeDone[neighborNodeID] == 0){
                //Record to 32 bit only
                if(mappingCount == mappingCount_max){
                    break;
                }

                sharedBitIndex[neighborNodeID] = 1 << mappingCount;
                mapping_SI[mappingCount] = neighborNodeID;
                nodeDone[neighborNodeID] = 1;

                // printf("sharedBitIndex[%6d] = %8x,\tmapping_SI[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], mappingCount, mapping_SI[mappingCount]);
                #ifdef DEBUG
                printf("sharedBitIndex[%2d] = %8x,\tmapping_SI[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], mappingCount, mapping_SI[mappingCount]);
                #endif
                
                mappingCount ++;
            }
        }
        Ns_node+= (mappingCount);
        //**********************************/
        //   Source forward traverse
        //**********************************/
        #pragma region SourceTraverse //forward traverse
        
        // Initialize arrays for each source node
        // sigma.assign(V, 0);   // Reset sigma to size V with all values 0
        // dist.assign(V, -1);   // Reset dist to size V with all values -1
        // delta.assign(V, 0.0); // Reset delta to size V with all values 0.0
        for (int i = 0; i < V; ++i) {
            sigma[i] = 0;   // 每個 sigma 初始化為 0
        }
        for (int i = 0; i < V; ++i) {
            dist[i] = -1;   // 每個距離初始化為 -1
        }
        for (int i = 0; i < V; ++i) {
            delta[i] = 0.0; // 每個 delta 初始化為 0.0
        }
        
        // fill(sigma.begin(), sigma.end(), 0);
        // fill(dist.begin(), dist.end(), -1);
        // fill(delta.begin(), delta.end(), 0.0);
        // S.assign(V, vector<int>());  // Reset S with empty vectors
        // Successors.assign(V, vector<int>());  // Reset Successors with empty vectors
        for (auto& level : S) {
            level.clear();
        }
        for (auto& preds : Successors) {
            preds.clear();
        }

        sigma[sourceID] = 1;
        dist[sourceID] = 0;
        f1.push(sourceID);
        int level = 0;

        // cout<<"source ID: "<<sourceID<<endl;
        // BFS forward phase
        
        while (!f1.empty()){
            while (!f1.empty()) {
                currentNodeID = f1.front();
                f1.pop();
                S[level].push_back(currentNodeID);
                #ifdef DEBUG
                printf("currentNodeID = %2d ... dist = %2d\n", currentNodeID, dist[currentNodeID]);
                #endif
                // Traverse neighbors in CSR
                for(neighborIndex = _csr->csrV[currentNodeID] ; neighborIndex < _csr->csrV[currentNodeID + 1] ; neighborIndex ++) {
                    neighborNodeID =  _csr->csrE[neighborIndex];

                    if (dist[neighborNodeID] < 0 ) {
                        dist[neighborNodeID] = dist[currentNodeID] + 1;    
                        // sharedBitIndex[neighborNodeID] |= sharedBitIndex[currentNodeID];
                        f2.push(neighborNodeID);
                        #ifdef DEBUG
                        printf("\t[1]unvisited_SI[%2d] => %2x, dist[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], neighborNodeID, dist[neighborNodeID]);
                        #endif
                    }
                    
                    if (dist[neighborNodeID] == dist[currentNodeID] + 1) {
                        sharedBitIndex[neighborNodeID] |= sharedBitIndex[currentNodeID];
                        Successors[currentNodeID].push_back(neighborNodeID);
                        sigma[neighborNodeID] += sigma[currentNodeID];
                        #ifdef DEBUG
                        printf("\t[2]unvisited_SI[%2d] => %2x, dist[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], neighborNodeID, dist[neighborNodeID]);
                        #endif
                    } //&& currentNodeID < neighborNodeID
                    else if(dist[neighborNodeID] == dist[currentNodeID] ){ //traverse to discovered neighbor which is at same level as currentNodeID
                        relation[currentNodeID]     |= sharedBitIndex[neighborNodeID] & (~sharedBitIndex[currentNodeID]);
                        // relation[neighborNodeID]    |= sharedBitIndex[currentNodeID]  & (~sharedBitIndex[neighborNodeID]);
                        #ifdef DEBUG
                        printf("\t[3]Red edge found(%2d, %2d), ", currentNodeID, neighborNodeID);
                        printf("relation[%2d] = %2x, relation[%2d] = %2x\n", currentNodeID, relation[currentNodeID], neighborNodeID, relation[neighborNodeID]);
                        #endif

                    }//&& relation[neighborNodeID]
                    else if(dist[neighborNodeID] == dist[currentNodeID] - 1){
                        //這一步是讓SI 跟R只會顯示一個 SI:1 R:0 幫助最終判鄰居是否相同 
                        relation[currentNodeID]     = (relation[currentNodeID]|relation[neighborNodeID]) & (~sharedBitIndex[currentNodeID] );
                        #ifdef DEBUG
                        printf("\t[4]Red edge found(%2d, %2d), ", currentNodeID, neighborNodeID);
                        printf("relation[%2d] = %2x, relation[%2d] = %2x\n", currentNodeID, relation[currentNodeID], neighborNodeID, relation[neighborNodeID]);
                        #endif
                    }

                }

            }
            swap(f1, f2);
            level++;
        }



        #ifdef DEBUG
            //printf successor
            printf("\n");
            for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
                printf("Successor(%d) sigma(%d) ",i,sigma[i] );
                for (int j=0;j<Successors[i].size();j++){
                    printf("%d ",Successors[i][j]);
                }
                printf("\n");
            }

            for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
                printf("%d: SI(%d) R(%d)\n",i,sharedBitIndex[i],relation[i] );
            }
        #endif
        //  printf("------------Source %d--------------\n",sourceID);
        #pragma endregion

        end_time = seconds();
        time_phase1 += end_time - start_time;

        start_time = seconds();
        //**********************************/
        //   Source backward traverse
        //**********************************/
        #pragma region sourceDistAccumulation_pushBased
            // Backward phase
            // for (int d = level - 1; d > 0; --d) {
            //     for (int w : S[d]) {
            //         for (int v : Successors[w]) {
            //             delta[w] += (sigma[w] / (float)sigma[v]) * (1.0 + delta[v]);
            //         }
            //         _BCs[w] += delta[w];
            //     }
            // }
            //d >= 0 這裡一定要有 =0 為了記錄sameindex
            float ratio = 0.0f;
            for (int d = level - 1; d >= 0; --d) {
                for (int node : S[d]) {
                    // for (auto SucNodeInDex = 0; SucNodeInDex < Successors[node].size(); SucNodeInDex++) {
                    //     int SucNodeID = Successors[node][SucNodeInDex];
                    //     delta[node] += (sigma[node] / (float)sigma[SucNodeID]) * (1.0 + delta[SucNodeID]);
                    // } 
                    for(neighborIndex = _csr->csrV[node] ; neighborIndex < _csr->csrV[node + 1] ; neighborIndex ++) {
                        neighborNodeID =  _csr->csrE[neighborIndex];

                        //1:代表我有不一樣的鄰居(edge)  0:代表node的鄰居都是一樣的顏色(Suc)
                        // sameIndex[node] |= (sharedBitIndex[node] | relation[node]) ^ (sharedBitIndex[neighborNodeID] | relation[neighborNodeID]);
                        sameIndex[node] |= (sharedBitIndex[node]^sharedBitIndex[neighborNodeID])|(relation[node]^relation[neighborNodeID]);
                        
                        //normal backward
                        if(dist[neighborNodeID] == dist[node] + 1 ){
                            ratio = (float)sigma[node] / sigma[neighborNodeID];
                            delta[node] += ratio * (1.0f + Ns_delta[neighborNodeID]);
                        }

                    }
                    ///normal BC accumulation
                    if (node != sourceID) {
                        // printf("delta[%d]: (%.2f)\n",node,delta[node]);
                        _BCs[node] += delta[node];
                        // _BCs_check[node] += delta[node];
                    }
                }
            }
        #pragma endregion //distAccumulation_pushBased
        
        // printf("-------------sourceID %d check-------------\n",sourceID);
        // brandes_with_predecessors_dynamic_check_ans(*_csr,V,sourceID,delta);

        // printf("------------sourceID %d delta-------------\n",sourceID);
        // for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
        //     printf("my_delta[%d]: (%.2f)\n",i,delta[i]);
        // }

        // printf("------------sourceID %d BC-------------\n",sourceID);
        //     for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
        //         printf("my_BC[%d]: (%.2f)\n",i,_BCs[i]);
        //     }

        // for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
        //     printf("_BC_first[%d]: (%.2f)\n",i,_BCs[i]);
        // }
        
        #ifdef DEBUG //print SI、R、sameIndex
            for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
                printf("_BC[%d]: (%.2f)\n",i,_BCs[i]);
            }
            for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
                printf(" sameIndex[%d]: ",i);
                printbinary(sameIndex[i],mappingCount);
                printf("\tSI[%d]: ",i);
                printbinary(sharedBitIndex[i],mappingCount);
                printf("\t R[%d]: ",i);
                printbinary(relation[i],mappingCount);
            }
        #endif

        end_time = seconds();
        time_phase2 += end_time - start_time;
        
        //****************************************/
        //   N(s) forward & backward traverse(鄰居當Source)
        //****************************************/

        //這部分使用 一個N(s)做完forward以及backward 再換下一個N(s)
        #pragma region third_and_forth_part
        //每個Ns做一次
        int sourceNode = -1;
        int SucNodeID  = -1;
        int SucNodeInDex = -1;
        for(auto Ns_index = 0 ; Ns_index < mappingCount ; Ns_index++ ){
            // start_time = seconds();
            start_time = seconds();
            sourceNode = mapping_SI[Ns_index];
            uint32_t Ns_mask = (1<<Ns_index);
            // printf("------------Source N(s) %d--------------\n",sourceNode);
            //****************************************/
            //               initial Ns陣列值
            //****************************************/
            //初始Ns的值
            // Ns_sigma.assign(V, 0);   // Reset sigma to size V with all values 0
            // Ns_dist.assign(V, -1);   // Reset dist to size V with all values -1
            // Ns_delta.assign(V, 0.0); // Reset delta to size V with all values 0.0
            for (int i = 0; i < V; ++i) {
                Ns_sigma[i] = 0;   // 每個 sigma 初始化為 0
            }
            for (int i = 0; i < V; ++i) {
                Ns_dist[i] = -1;   // 每個距離初始化為 -1
            }
            for (int i = 0; i < V; ++i) {
                Ns_delta[i] = 0.0; // 每個 delta 初始化為 0.0
            }
            
            // fill(Ns_sigma.begin(), Ns_sigma.end(), 0);
            // fill(Ns_dist.begin(), Ns_dist.end(), -1);
            // fill(Ns_delta.begin(), Ns_delta.end(), 0.0);
            // S.assign(V, vector<int>());  // Reset S with empty vectors
            // Ns_Successors.assign(V, vector<int>());  // Reset Successors with empty vectors
            for (auto& level : S) {
                level.clear();
            }
            for (auto& Suc : Ns_Successors) {
                Suc.clear();
            }

            
            //****************************************/
            //               initial Source
            //****************************************/
            Ns_sigma[sourceNode] = 1;
            Ns_dist [sourceNode] = 0;
            f1.push(sourceNode);


            //****************************************/
            //         N(s) forward traverse
            //****************************************/
            level=0;
            while (!f1.empty()){
                while (!f1.empty()) {
                    int currentNodeID = f1.front();
                    S[level].push_back(currentNodeID);
                    f1.pop();

                    if( (sameIndex[currentNodeID] & Ns_mask) == 0 ){ //edge_traverse

                        Suc_times++;
                        for (  SucNodeInDex = 0 ; SucNodeInDex<Successors[currentNodeID].size(); SucNodeInDex++) {  
                            SucNodeID = Successors[currentNodeID][SucNodeInDex];
                            if(Ns_dist[SucNodeID] < 0){
                                Ns_dist [SucNodeID]  = Ns_dist[currentNodeID] + 1;
                                //push node into Q_f2_temp
                                f2.push(SucNodeID);
                            }
                            Ns_Successors[currentNodeID].push_back(SucNodeID);
                            Ns_sigma[SucNodeID] += Ns_sigma[currentNodeID];
                        }
                    }else{//Suc_traverse

                        edge_times++;
                        for (neighborIndex = _csr->csrV[currentNodeID]; neighborIndex < _csr->csrV[currentNodeID + 1]; neighborIndex++) {
                            
                            neighborNodeID = _csr->csrE[neighborIndex];
                            //累加路徑數量、dist
                            if (Ns_dist[neighborNodeID] < 0) {
                                // Ns_Successors[currentNodeID].push_back(neighborNodeID);
                                Ns_dist [neighborNodeID]  = Ns_dist[currentNodeID] + 1;
                                // Ns_sigma[neighborNodeID] += Ns_sigma[currentNodeID];
                                // printf("[%d] push in edge\n",  neighborNodeID);
                                f2.push(neighborNodeID);
                            }
                            
                            if (Ns_dist[neighborNodeID] == Ns_dist[currentNodeID] + 1) {
                                Ns_Successors[currentNodeID].push_back(neighborNodeID);
                                Ns_sigma[neighborNodeID] += Ns_sigma[currentNodeID];
                            }
                        }
                    }

                }
                swap(f1,f2);
                level++;
            }

            
            end_time = seconds();
            time_phase3 += end_time - start_time;
            start_time = seconds();
            //****************************************/
            //         N(s) backward traverse
            //****************************************/

            float ratio = 0.0f;
            for (int d = level - 1; d > 0; d--) {
                
                for (int node : S[d]) {
                    // printf("back node: %d\n",node);
                    // if( (sameIndex[node] & Ns_mask) == 0 ) { //Suc_traverse
                    //     for (int SucNodeID : Successors[node]) {
                    //         Ns_delta[node] += (Ns_sigma[node] / static_cast<float>(Ns_sigma[SucNodeID])) * (1.0 + Ns_delta[SucNodeID]);
                    //     }
                    // }else{
                        for (int SucNodeID : Ns_Successors[node]) {
                            ratio = (float)Ns_sigma[node] / Ns_sigma[SucNodeID];
                            Ns_delta[node] += ratio * (1.0f + Ns_delta[SucNodeID]);
                        }
                    // }
                    
                    
                    // if(node != sourceNode){ 
                        _BCs[node] += Ns_delta[node];
                    // } 
                }
                
            }

            // printf("-------------Ns sourceID %d check-------------\n",sourceNode);
            // brandes_with_predecessors_dynamic_check_ans(*_csr,V,sourceNode,Ns_delta);


            // printf("------------Ns sourceID %d delta-------------\n",sourceNode);
            // for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
            //     printf("my_delta[%d]: (%.2f)\n",i,Ns_delta[i]);
            // }

            // printf("-------------Ns sourceID %d BC-------------\n",sourceNode);
            // for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
            //     printf("my_BC[%d]: (%.2f)\n",i,_BCs[i]);
            // }
            end_time = seconds();
            time_phase4 += end_time - start_time;
           
            #ifdef DEBUG
            for(int i= _csr->startNodeID ; i<= _csr->endNodeID ;i++){
                printf("%d of Ns_delta[%d]: (%.2f)\n",sourceNode,i,Ns_delta[i]);
            }
            #endif


        }

        #pragma endregion //neighborOfSource_GetDist


        //reset the SI & relation arrays
        relation.assign(V, 0);       // Reset relation to size _csr->csrVSize with all values 0
        sharedBitIndex.assign(V, 0); // Reset sharedBitIndex to size _csr->csrVSize with all values 0
        sameIndex.assign(V, 0);      // Reset sameIndex to size _csr->csrVSize with all values 0


    }
    mymethod_time2 += mymethod_time1 + time_phase1 +time_phase2+time_phase3+time_phase4;
    // multi_time2 = seconds();
    unsigned long total_times=edge_times+Suc_times;
    printf("suc  traverse: %0.2f\n", Suc_times/(float)total_times);
    printf("edge traverse: %0.2f\n", edge_times/(float)total_times);
    printf("S_node       : %ld\n", S_node);
    printf("Ns_node      : %ld\n", Ns_node);
    printf("sort   time: %0.6f\n", time_sort);
    printf("phase1 time: %0.6f\n", time_phase1);
    printf("phase2 time: %0.6f\n", time_phase2);
    printf("phase3 time: %0.6f\n", time_phase3);
    printf("phase4 time: %0.6f\n", time_phase4);
    // for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
    //     printf("my_BC[%d]: (%.2f)\n",i,_BCs[i]);
    // }

    // printf("\n[BC_sharedBased] Done!\n");
}

 #pragma endregion
void computeBC_shareBased_Successor_SS_edge_update( CSR* _csr, float* _BCs){
    //時間量測
    double time_sort=0.0;
    double time_phase1=0.0;
    double time_phase2=0.0;
    double time_phase3=0.0;
    double time_phase4=0.0;
    double start_time=0.0;
    double end_time=0.0;


    // multi_time1 = seconds();
    start_time=seconds();
    // showCSR(_csr);
    int V =  _csr->csrVSize;


    //record that nodes which haven't been source yet
    int* nodeDone = (int*)calloc(sizeof(int), _csr->csrVSize);
    
    //record nodes belongs to which neighbor of source
    int  mappingCount_max           = 32; //最大可以設32
    int* mapping_SI                 = (int*)malloc(sizeof(int) * 32);
    vector<unsigned int> sharedBitIndex(_csr->csrVSize, 0); // for recording blue edge bitIndex
    vector<unsigned int> relation(_csr->csrVSize, 0);       // for recording red edge bitIndex
    vector<unsigned int> sameIndex(_csr->csrVSize, 0);// for recording blue edge BC_sameIndex

    // Allocate memory for BFS data structures
    vector<vector<int>> S(V);              // S is a 2D stack
    vector<int> sigma(V, 0);               // Sigma array
    vector<int> dist(V, -1);               // Distance array
    vector<float> delta(V, 0.0);           // Delta array
    vector<vector<int>> Successors(V);     // Successors list 
    vector<float> _BCs_check(V, 0.0);           // Delta array
    //可重複使用的空間
    queue<int> f1, f2;                     // Current and Next frontier
    //N(s)鄰居的traverse 路徑數量、距離、Suc、Sigma
    vector<vector<int>> N_S(V);              // S is a 2D stack
    vector<int>  Ns_dist  (V, -1);            // Distance array
    vector<int>  Ns_sigma (V, 0);             // Sigma array
    vector<float> Ns_delta(V, 0.0);           // Delta array
    vector<vector<int>> Ns_Successors(V);     // Successors [][]: [V] [V的Suc] 
    
    //Suc與edge數據
    unsigned long Suc_times=0;
    unsigned long edge_times=0;
    long S_node = 0;
    long Ns_node = 0;


    
    
    //用degree做排序 大->小
    _csr->orderedCsrV  = (int*)calloc(sizeof(int), (_csr->csrVSize) *2);
    for(int i=_csr->startNodeID;i<=_csr->endNodeID;i++){
            _csr->orderedCsrV[i]=i;
    }
    quicksort_nodeID_with_degree(_csr->orderedCsrV, _csr->csrNodesDegree, _csr->startNodeID, _csr->endNodeID);
    
    // printf("orderedCsrV: ");
    // for(int i=_csr->startNodeID;i<=_csr->endNodeID;i++){
    //         printf("[%d][%d] ",i,_csr->orderedCsrV[i]);
    // }
    // printf("\n");

    end_time = seconds();
    time_sort += end_time - start_time;
    // for(int sourceID = _csr->startNodeID ; sourceID <= _csr->endNodeID ; sourceID ++){
    


    for(int sourceIDIndex = _csr->startNodeID ; sourceIDIndex <= _csr->endNodeID ; sourceIDIndex ++){
        
        int sourceID = _csr->orderedCsrV[sourceIDIndex];
        
        if(nodeDone[sourceID] == 1){
            continue;
        }
        nodeDone[sourceID] = 1;
        
        start_time=seconds();
        // printf("SourceID = %2d\n", sourceID);
        // S_node++;

        
        //each neighbor of sourceID mapping to bit_SI, if it haven't been source yet
        //挑選鄰居
        int mappingCount = 0;
        for(int neighborIndex = _csr->csrV[sourceID] ; neighborIndex < _csr->csrV[sourceID + 1] ; neighborIndex ++){
            const int neighborNodeID = _csr->csrE[neighborIndex];

            if(nodeDone[neighborNodeID] == 0){
                //Record to 32 bit only
                if(mappingCount == mappingCount_max){
                    break;
                }

                sharedBitIndex[neighborNodeID] = 1 << mappingCount;
                mapping_SI[mappingCount] = neighborNodeID;
                nodeDone[neighborNodeID] = 1;

                // printf("sharedBitIndex[%6d] = %8x,\tmapping_SI[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], mappingCount, mapping_SI[mappingCount]);
                #ifdef DEBUG
                printf("sharedBitIndex[%2d] = %8x,\tmapping_SI[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], mappingCount, mapping_SI[mappingCount]);
                #endif
                
                mappingCount ++;
            }
        }
        // Ns_node+= (mappingCount);
        //**********************************/
        //   Source forward traverse
        //**********************************/
        #pragma region SourceTraverse //forward traverse
        
        // Initialize arrays for each source node
        sigma.assign(V, 0);   // Reset sigma to size V with all values 0
        dist.assign(V, -1);   // Reset dist to size V with all values -1
        delta.assign(V, 0.0); // Reset delta to size V with all values 0.0
        // fill(sigma.begin(), sigma.end(), 0);
        // fill(dist.begin(), dist.end(), -1);
        // fill(delta.begin(), delta.end(), 0.0);
        // S.assign(V, vector<int>());  // Reset S with empty vectors
        // Successors.assign(V, vector<int>());  // Reset Successors with empty vectors
        for (auto& level : S) {
            level.clear();
        }
        for (auto& preds : Successors) {
            preds.clear();
        }

        sigma[sourceID] = 1;
        dist[sourceID] = 0;
        f1.push(sourceID);
        int level = 0;

        // cout<<"source ID: "<<sourceID<<endl;
        // BFS forward phase
        
        while (!f1.empty()){
            while (!f1.empty()) {
                int currentNodeID = f1.front();
                f1.pop();
                S[level].push_back(currentNodeID);
                #ifdef DEBUG
                printf("currentNodeID = %2d ... dist = %2d\n", currentNodeID, dist[currentNodeID]);
                #endif
                // Traverse neighbors in CSR
                for(int neighborIndex = _csr->csrV[currentNodeID] ; neighborIndex < _csr->csrV[currentNodeID + 1] ; neighborIndex ++) {
                    int neighborNodeID =  _csr->csrE[neighborIndex];

                    if (dist[neighborNodeID] < 0 ) {
                        dist[neighborNodeID] = dist[currentNodeID] + 1;    
                        // sharedBitIndex[neighborNodeID] |= sharedBitIndex[currentNodeID];
                        f2.push(neighborNodeID);
                        #ifdef DEBUG
                        printf("\t[1]unvisited_SI[%2d] => %2x, dist[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], neighborNodeID, dist[neighborNodeID]);
                        #endif
                    }
                    
                    if (dist[neighborNodeID] == dist[currentNodeID] + 1) {
                        sharedBitIndex[neighborNodeID] |= sharedBitIndex[currentNodeID];
                        Successors[currentNodeID].push_back(neighborNodeID);
                        sigma[neighborNodeID] += sigma[currentNodeID];
                        #ifdef DEBUG
                        printf("\t[2]unvisited_SI[%2d] => %2x, dist[%2d] = %2d\n", neighborNodeID, sharedBitIndex[neighborNodeID], neighborNodeID, dist[neighborNodeID]);
                        #endif
                    } //&& currentNodeID < neighborNodeID
                    else if(dist[neighborNodeID] == dist[currentNodeID] ){ //traverse to discovered neighbor which is at same level as currentNodeID
                        relation[currentNodeID]     |= sharedBitIndex[neighborNodeID] & (~sharedBitIndex[currentNodeID]);
                        // relation[neighborNodeID]    |= sharedBitIndex[currentNodeID]  & (~sharedBitIndex[neighborNodeID]);
                        #ifdef DEBUG
                        printf("\t[3]Red edge found(%2d, %2d), ", currentNodeID, neighborNodeID);
                        printf("relation[%2d] = %2x, relation[%2d] = %2x\n", currentNodeID, relation[currentNodeID], neighborNodeID, relation[neighborNodeID]);
                        #endif

                    }//&& relation[neighborNodeID]
                    else if(dist[neighborNodeID] == dist[currentNodeID] - 1){
                        //這一步是讓SI 跟R只會顯示一個 SI:1 R:0 幫助最終判鄰居是否相同 
                        relation[currentNodeID]     = (relation[currentNodeID]|relation[neighborNodeID]) & (~sharedBitIndex[currentNodeID] );
                        #ifdef DEBUG
                        printf("\t[4]Red edge found(%2d, %2d), ", currentNodeID, neighborNodeID);
                        printf("relation[%2d] = %2x, relation[%2d] = %2x\n", currentNodeID, relation[currentNodeID], neighborNodeID, relation[neighborNodeID]);
                        #endif
                    }

                }

            }
            swap(f1, f2);
            level++;
        }



        #ifdef DEBUG
            //printf successor
            printf("\n");
            for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
                printf("Successor(%d) sigma(%d) ",i,sigma[i] );
                for (int j=0;j<Successors[i].size();j++){
                    printf("%d ",Successors[i][j]);
                }
                printf("\n");
            }

            for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
                printf("%d: SI(%d) R(%d)\n",i,sharedBitIndex[i],relation[i] );
            }
        #endif
        //  printf("------------Source %d--------------\n",sourceID);
        #pragma endregion

        end_time = seconds();
        time_phase1 += end_time - start_time;

        start_time = seconds();
        //**********************************/
        //   Source backward traverse
        //**********************************/
        #pragma region sourceDistAccumulation_pushBased
            // Backward phase
            // for (int d = level - 1; d > 0; --d) {
            //     for (int w : S[d]) {
            //         for (int v : Successors[w]) {
            //             delta[w] += (sigma[w] / (float)sigma[v]) * (1.0 + delta[v]);
            //         }
            //         _BCs[w] += delta[w];
            //     }
            // }
            //d >= 0 這裡一定要有 =0 為了記錄sameindex
            float ratio = 0.0f;
            for (int d = level - 1; d >= 0; --d) {
                for (int node : S[d]) {
                    // for (auto SucNodeInDex = 0; SucNodeInDex < Successors[node].size(); SucNodeInDex++) {
                    //     int SucNodeID = Successors[node][SucNodeInDex];
                    //     delta[node] += (sigma[node] / (float)sigma[SucNodeID]) * (1.0 + delta[SucNodeID]);
                    // } 
                    for(int neighborIndex = _csr->csrV[node] ; neighborIndex < _csr->csrV[node + 1] ; neighborIndex ++) {
                        int neighborNodeID =  _csr->csrE[neighborIndex];

                        //1:代表我有不一樣的鄰居(edge)  0:代表node的鄰居都是一樣的顏色(Suc)
                        // sameIndex[node] |= (sharedBitIndex[node] | relation[node]) ^ (sharedBitIndex[neighborNodeID] | relation[neighborNodeID]);
                        sameIndex[node] |= (sharedBitIndex[node]^sharedBitIndex[neighborNodeID])|(relation[node]^relation[neighborNodeID]);
                        
                        //normal backward
                        if(dist[neighborNodeID] == dist[node] + 1 ){
                            ratio = (float)sigma[node] / sigma[neighborNodeID];
                            delta[node] += ratio * (1.0f + Ns_delta[neighborNodeID]);
                        }

                    }
                    ///normal BC accumulation
                    if (node != sourceID) {
                        // printf("delta[%d]: (%.2f)\n",node,delta[node]);
                        _BCs[node] += delta[node];
                        // _BCs_check[node] += delta[node];
                    }
                }
            }
        #pragma endregion //distAccumulation_pushBased
        
        // printf("-------------sourceID %d check-------------\n",sourceID);
        // brandes_with_predecessors_dynamic_check_ans(*_csr,V,sourceID,delta);

        // printf("------------sourceID %d delta-------------\n",sourceID);
        // for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
        //     printf("my_delta[%d]: (%.2f)\n",i,delta[i]);
        // }

        // printf("------------sourceID %d BC-------------\n",sourceID);
        //     for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
        //         printf("my_BC[%d]: (%.2f)\n",i,_BCs[i]);
        //     }

        // for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
        //     printf("_BC_first[%d]: (%.2f)\n",i,_BCs[i]);
        // }
        
        #ifdef DEBUG //print SI、R、sameIndex
            for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
                printf("_BC[%d]: (%.2f)\n",i,_BCs[i]);
            }
            for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
                printf(" sameIndex[%d]: ",i);
                printbinary(sameIndex[i],mappingCount);
                printf("\tSI[%d]: ",i);
                printbinary(sharedBitIndex[i],mappingCount);
                printf("\t R[%d]: ",i);
                printbinary(relation[i],mappingCount);
            }
        #endif

        end_time = seconds();
        time_phase2 += end_time - start_time;
        
        //****************************************/
        //   N(s) forward & backward traverse(鄰居當Source)
        //****************************************/

        //這部分使用 一個N(s)做完forward以及backward 再換下一個N(s)
        #pragma region third_and_forth_part
        //每個Ns做一次
        int sourceNode =0;
        for(auto Ns_index = 0 ; Ns_index < mappingCount ; Ns_index++ ){
            // start_time = seconds();
           start_time = seconds();
            sourceNode   = mapping_SI[Ns_index];
            uint32_t Ns_mask = (1<<Ns_index);
            // printf("------------Source N(s) %d--------------\n",sourceNode);
            //****************************************/
            //               initial Ns陣列值
            //****************************************/
            //初始Ns的值
            Ns_sigma.assign(V, 0);   // Reset sigma to size V with all values 0
            Ns_dist.assign(V, -1);   // Reset dist to size V with all values -1
            Ns_delta.assign(V, 0.0); // Reset delta to size V with all values 0.0
            // fill(Ns_sigma.begin(), Ns_sigma.end(), 0);
            // fill(Ns_dist.begin(), Ns_dist.end(), -1);
            // fill(Ns_delta.begin(), Ns_delta.end(), 0.0);
            // S.assign(V, vector<int>());  // Reset S with empty vectors
            // Ns_Successors.assign(V, vector<int>());  // Reset Successors with empty vectors
            for (auto& level : S) {
                level.clear();
            }
            for (auto& Suc : Ns_Successors) {
                Suc.clear();
            }

            
            //****************************************/
            //               initial Source
            //****************************************/
            Ns_sigma[sourceNode] = 1;
            Ns_dist [sourceNode] = 0;
            f1.push(sourceNode);


            //****************************************/
            //         N(s) forward traverse
            //****************************************/
            level=0;
            while (!f1.empty()){
                while (!f1.empty()) {
                    int currentNodeID = f1.front();
                    S[level].push_back(currentNodeID);
                    f1.pop();

                    if( (sameIndex[currentNodeID] & Ns_mask) == 0 ){ //Suc_traverse
                        // Suc_times++;
                        for (auto SucNodeInDex = 0; SucNodeInDex < Successors[currentNodeID].size(); SucNodeInDex++) {
                            int SucNodeID = Successors[currentNodeID][SucNodeInDex];
                            if(Ns_dist[SucNodeID] < 0){
                                Ns_dist [SucNodeID]  = Ns_dist[currentNodeID] + 1;
                                //push node into Q_f2_temp
                                f2.push(SucNodeID);
                            }
                            Ns_Successors[currentNodeID].push_back(SucNodeID);
                            Ns_sigma[SucNodeID] += Ns_sigma[currentNodeID];
                        }
                    }else{//edge_traverse
                        // edge_times++;
                        for (auto neighborIndex = _csr->csrV[currentNodeID]; neighborIndex < _csr->csrV[currentNodeID + 1]; neighborIndex++) {
                            const int neighborNodeID = _csr->csrE[neighborIndex];
                            //累加路徑數量、dist
                            if (Ns_dist[neighborNodeID] < 0) {
                                // Ns_Successors[currentNodeID].push_back(neighborNodeID);
                                Ns_dist [neighborNodeID]  = Ns_dist[currentNodeID] + 1;
                                // Ns_sigma[neighborNodeID] += Ns_sigma[currentNodeID];
                                // printf("[%d] push in edge\n",  neighborNodeID);
                                f2.push(neighborNodeID);
                            }
                            
                            if (Ns_dist[neighborNodeID] == Ns_dist[currentNodeID] + 1) {
                                Ns_Successors[currentNodeID].push_back(neighborNodeID);
                                Ns_sigma[neighborNodeID] += Ns_sigma[currentNodeID];
                            }
                        }
                    }

                }
                swap(f1,f2);
                level++;
            }

            
            end_time = seconds();
            time_phase3 += end_time - start_time;
            start_time = seconds();
            //****************************************/
            //         N(s) backward traverse
            //****************************************/

            float ratio = 0.0;
            for (int d = level - 1; d > 0; d--) {
                
                for (int node : S[d]) {
                    // printf("back node: %d\n",node);
                    // if( (sameIndex[node] & Ns_mask) == 0 ) { //Suc_traverse
                    //     for (int SucNodeID : Successors[node]) {
                    //         Ns_delta[node] += (Ns_sigma[node] / static_cast<float>(Ns_sigma[SucNodeID])) * (1.0 + Ns_delta[SucNodeID]);
                    //     }
                    // }else{
                        for (int SucNodeID : Ns_Successors[node]) {
                            ratio = (float)Ns_sigma[node] / Ns_sigma[SucNodeID];
                            Ns_delta[node] += ratio * (1.0f + Ns_delta[SucNodeID]);
                        }
                    // }
                    
                    
                    // if(node != sourceNode){
                        // if(node == 5)
                        //     printf("brfore BC: (%0.2f) Ns_delta: (%.2f)\n",_BCs[node],Ns_delta[node]);
                        _BCs[node] += Ns_delta[node];
                        // if(node == 5)
                        //     printf("after BC: (%0.2f) Ns_delta: (%.2f)\n",_BCs[node],Ns_delta[node]);
                    // } 
                }
                
            }

            // printf("-------------Ns sourceID %d check-------------\n",sourceNode);
            // brandes_with_predecessors_dynamic_check_ans(*_csr,V,sourceNode,Ns_delta);


            // printf("------------Ns sourceID %d delta-------------\n",sourceNode);
            // for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
            //     printf("my_delta[%d]: (%.2f)\n",i,Ns_delta[i]);
            // }

            // printf("-------------Ns sourceID %d BC-------------\n",sourceNode);
            // for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
            //     printf("my_BC[%d]: (%.2f)\n",i,_BCs[i]);
            // }
            end_time = seconds();
            time_phase4 += end_time - start_time;
           
            #ifdef DEBUG
            for(int i= _csr->startNodeID ; i<= _csr->endNodeID ;i++){
                printf("%d of Ns_delta[%d]: (%.2f)\n",sourceNode,i,Ns_delta[i]);
            }
            #endif


        }

        #pragma endregion //neighborOfSource_GetDist


        //reset the SI & relation arrays
        relation.assign(V, 0);       // Reset relation to size _csr->csrVSize with all values 0
        sharedBitIndex.assign(V, 0); // Reset sharedBitIndex to size _csr->csrVSize with all values 0
        sameIndex.assign(V, 0);      // Reset sameIndex to size _csr->csrVSize with all values 0


    }

    // multi_time2 = seconds();
    unsigned long total_times=edge_times+Suc_times;
    printf("suc  traverse: %0.2f\n", Suc_times/(float)total_times);
    printf("edge traverse: %0.2f\n", edge_times/(float)total_times);
    printf("S_node       : %ld\n", S_node);
    printf("Ns_node      : %ld\n", Ns_node);
    printf("sort   time: %0.6f\n", time_sort);
    printf("phase1 time: %0.6f\n", time_phase1);
    printf("phase2 time: %0.6f\n", time_phase2);
    printf("phase3 time: %0.6f\n", time_phase3);
    printf("phase4 time: %0.6f\n", time_phase4);
    // for (int i= _csr->startNodeID;i<=_csr->endNodeID;i++) {
    //     printf("my_BC[%d]: (%.2f)\n",i,_BCs[i]);
    // }

    // printf("\n[BC_sharedBased] Done!\n");
}



void computeBC_shareBased_Successor_SS_test( CSR* csr, float* _BCs){
    // Allocate memory for BFS data structures
    // multi_time1 = seconds();
    int V = csr->csrVSize;
    vector<vector<int>> S(V);               // S is a 2D stack
    vector<int> sigma(V, 0);               // Sigma array
    vector<int> dist(V, -1);               // Distance array
    vector<float> delta(V, 0.0);           // Delta array
    vector<int> S_size(V, 0);              // Stack size for each level
    queue<int> f1, f2;                     // Current and Next frontier
    vector<vector<int>> Sucecessors(V);   // Predecessor list

    // long long total_predecessor_count = 0; // To accumulate total predecessors

    double time_sort=0.0;
    double time_phase1=0.0;
    double time_phase2=0.0;
    double time_phase3=0.0;
    double time_phase4=0.0;
    double start_time=0.0;
    double end_time=0.0;

    for (int s = csr->startNodeID; s <= csr->endNodeID; ++s) {
        start_time=seconds();
        // Initialize arrays for each source node
        sigma.assign(V, 0);
        dist.assign(V, -1);
        delta.assign(V, 0);
        for (auto& level : S) {
            level.clear();
        }
        for (auto& preds : Sucecessors) {
            preds.clear();
        }

        sigma[s] = 1;
        dist[s] = 0;
        f1.push(s);

        int level = 0;

        // BFS forward phase
        while (!f1.empty()) {
            while (!f1.empty()) {
                int u = f1.front();
                f1.pop();
                S[level].push_back(u);

                // Traverse neighbors in CSR
                for (int i = csr->csrV[u]; i < csr->csrV[u + 1]; ++i) {
                    int w = csr->csrE[i];

                    if (dist[w] < 0) {
                        dist[w] = dist[u] + 1;
                        sigma[w] += sigma[u];
                        Sucecessors[u].push_back(w);
                        f2.push(w);
                    }
                    else if (dist[w] == dist[u] + 1) {
                        sigma[w] += sigma[u];
                        Sucecessors[u].push_back(w);
                    }
                }
            }
            swap(f1, f2);
            level++;
        }

        end_time=seconds();
        time_phase1 += end_time - start_time;
        start_time=seconds();
        // Backward phase
        for (int d = level - 1; d >= 0; --d) {
            for (int w : S[d]) {
                for (int v : Sucecessors[w]) {
                    delta[w] += (sigma[w] / (float)sigma[v]) * (1.0 + delta[v]);
                }
                if (w != s) {
                    _BCs[w] += delta[w];
                }
            }
        }

        end_time=seconds();
        time_phase2 += end_time - start_time;
       
    }
    // multi_time2 = seconds();
    printf("phase1 time: %0.6f\n", time_phase1);
    printf("phase2 time: %0.6f\n", time_phase2);

}

//************************************************ */
//                   循序_brandes MS原版
//************************************************ */
void Seq_multi_source_brandes( CSR& csr, int max_multi, vector<float> &BC) {
    // Start timing
    // multi_time_start = seconds();

    int v_size = csr.csrVSize;
    int* map_S = (int*)malloc(sizeof(int) * max_multi); // Multiple sources
    bool* nodeDone = (bool*)calloc(v_size, sizeof(bool));

    size_t multi_size = v_size * max_multi;

    int* s_size = (int*)malloc(sizeof(int) * v_size);
    float* dist_MULTI = (float*)malloc(sizeof(float) * multi_size);
    float* sigma_MULTI = (float*)malloc(sizeof(float) * multi_size);
    float* delta_MULTI = (float*)malloc(sizeof(float) * multi_size);

    Q_struct** s = (Q_struct**)malloc(v_size * sizeof(Q_struct*));
    Q_struct* f1 = (Q_struct*)malloc(v_size * sizeof(Q_struct));
    Q_struct* f2 = (Q_struct*)malloc(v_size * sizeof(Q_struct));

    // Pre-initialize dist_MULTI to INFINITE
    float* dist_INIT = (float*)malloc(sizeof(float) * multi_size);
    
    for (size_t i = 0; i < multi_size; i++) {
        dist_INIT[i] = INFINITE;
    }

    for (int sourceID = csr.startNodeID; sourceID <= csr.endNodeID; ++sourceID) {
        if (nodeDone[sourceID]) continue;

        // multi_time1 = seconds();

        nodeDone[sourceID] = true;
        int mappingCount = 0;
        map_S[mappingCount++] = sourceID;

        // Find other sources
        for (int neighborIndex = csr.csrV[sourceID]; neighborIndex < csr.csrV[sourceID + 1] && mappingCount < max_multi; neighborIndex++) {
            int neighborNodeID = csr.csrE[neighborIndex];
            if (!nodeDone[neighborNodeID]) {
                map_S[mappingCount++] = neighborNodeID;
                nodeDone[neighborNodeID] = true;
            }
        }

        // Initialize dist_MULTI, sigma_MULTI, delta_MULTI
        memcpy(dist_MULTI, dist_INIT, sizeof(float) * multi_size);
        memset(sigma_MULTI, 0, sizeof(float) * multi_size);
        memset(delta_MULTI, 0, sizeof(float) * multi_size);
        memset(s_size, 0, sizeof(int) * v_size);

        int f1_indicator = 0;
        int f2_indicator = 0;
        int s_indicator = 0;

        // Initialize currentQueue
        for (int i = 0; i < mappingCount; i++) {
            int sourceNode = map_S[i];
            int position = mappingCount * sourceNode + i;
            sigma_MULTI[position] = 1.0;
            dist_MULTI[position] = 0.0;

            f1[f1_indicator].nodeID = sourceNode;
            f1[f1_indicator].traverse_S = (1ULL << i);
            f1_indicator++;
        }

        // Initialize nextQueue traverse_S to zero
        for (int i = 0; i < v_size; i++) {
            f2[i].traverse_S = 0;
        }

        int level=0;
        while (f1_indicator > 0) {

            Q_struct* currentQueue;
            Q_struct* nextQueue;
            if(level% 2 == 0){
                currentQueue = f1;
                nextQueue = f2;
            }
            else{
                currentQueue = f2;
                nextQueue = f1;
            }
            // Store currentQueue into s[s_indicator]
            s[s_indicator] = (Q_struct*)malloc(f1_indicator * sizeof(Q_struct));
            memcpy(s[s_indicator], currentQueue, f1_indicator * sizeof(Q_struct));
            s_size[s_indicator++] = f1_indicator;

            // Process currentQueue
            for (auto i = 0; i < f1_indicator; i++) {
                int v = currentQueue[i].nodeID;
                uint64_t traverse_S = currentQueue[i].traverse_S;

                for (auto neighborIndex = csr.csrV[v]; neighborIndex < csr.csrV[v + 1]; neighborIndex++) {
                    int neighborNodeID = csr.csrE[neighborIndex];

                    for (auto multi_node = 0; multi_node < mappingCount; multi_node++) {
                        if (traverse_S & (1ULL << multi_node)) {
                            int position_v = mappingCount * v + multi_node;
                            int position_n = mappingCount * neighborNodeID + multi_node;

                            if (dist_MULTI[position_n] == dist_MULTI[position_v] + 1) {
                                sigma_MULTI[position_n] += sigma_MULTI[position_v];
                            } else if (dist_MULTI[position_n] > dist_MULTI[position_v] + 1) {
                                dist_MULTI[position_n] = dist_MULTI[position_v] + 1;
                                sigma_MULTI[position_n] = sigma_MULTI[position_v];

                                // Check if neighborNodeID is already in nextQueue
                                int found = -1;
                                for (auto find = 0; find < f2_indicator; find++) {
                                    if (nextQueue[find].nodeID == neighborNodeID) {
                                        found = find;
                                        break;
                                    }
                                }
                                if (found >= 0) {
                                    nextQueue[found].traverse_S |= (1ULL << multi_node);
                                } else {
                                    nextQueue[f2_indicator].nodeID = neighborNodeID;
                                    nextQueue[f2_indicator].traverse_S = (1ULL << multi_node);
                                    f2_indicator++;
                                }
                            }
                        }
                    }
                }
            }

            // Swap currentQueue and nextQueue
            f1_indicator = f2_indicator;
            f2_indicator = 0;
            level++;
            // Reset nextQueue traverse_S for next iteration
            // for (int i = 0; i < f1_indicator; i++) {
            //     nextQueue[i].traverse_S = 0;
            // }
        }

        // multi_time2 = seconds();
        // multi_forward_Time += (multi_time2 - multi_time1);
        // multi_time1 = seconds();

        // Back-propagation
        for (int layer = s_indicator - 1; layer >= 0; layer--) {
            for (int i = 0; i < s_size[layer]; i++) {
                int v = s[layer][i].nodeID;
                uint64_t traverse_S = s[layer][i].traverse_S;

                for (int multi_node = 0; multi_node < mappingCount; multi_node++) {
                    if (traverse_S & (1ULL << multi_node)) {
                        int position_v = mappingCount * v + multi_node;

                        float coeff = 0.0;

                        // For each neighbor w of v
                        for (int neighborIndex = csr.csrV[v]; neighborIndex < csr.csrV[v + 1]; neighborIndex++) {
                            int w = csr.csrE[neighborIndex];
                            int position_w = mappingCount * w + multi_node;

                            if (dist_MULTI[position_w] == dist_MULTI[position_v] + 1) {
                                coeff += (sigma_MULTI[position_v] / sigma_MULTI[position_w]) * (1.0 + delta_MULTI[position_w]);
                            }
                        }

                        delta_MULTI[position_v] += coeff;

                        // If v is not the source node, accumulate delta into BC[v]
                        if (v != map_S[multi_node]) {
                            BC[v] += delta_MULTI[position_v];
                        }
                    }
                }
            }
            // Free s[layer]
            free(s[layer]);
        }

        // multi_time2 = seconds();
        // multi_backward_Time += (multi_time2 - multi_time1);
    }

    // multi_time_end = seconds();
    // multi_total_time = (multi_time_end - multi_time_start);

    // Free memory
    free(s_size);
    free(dist_MULTI);
    free(sigma_MULTI);
    free(delta_MULTI);
    free(map_S);
    free(nodeDone);
    free(s);
    free(f1);
    free(f2);
    free(dist_INIT);
}

void Seq_multi_source_brandes_ordered( CSR& csr, int max_multi, vector<float> &BC) {
    // Start timing
    // multi_time_start = seconds();

    int v_size = csr.csrVSize;
    int* map_S = (int*)malloc(sizeof(int) * max_multi); // Multiple sources
    bool* nodeDone = (bool*)calloc(v_size, sizeof(bool));

    size_t multi_size = v_size * max_multi;

    int* s_size = (int*)malloc(sizeof(int) * v_size);
    float* dist_MULTI = (float*)malloc(sizeof(float) * multi_size);
    float* sigma_MULTI = (float*)malloc(sizeof(float) * multi_size);
    float* delta_MULTI = (float*)malloc(sizeof(float) * multi_size);

    Q_struct** s = (Q_struct**)malloc(v_size * sizeof(Q_struct*));
    Q_struct* f1 = (Q_struct*)malloc(v_size * sizeof(Q_struct));
    Q_struct* f2 = (Q_struct*)malloc(v_size * sizeof(Q_struct));

    // Pre-initialize dist_MULTI to INFINITE
    float* dist_INIT = (float*)malloc(sizeof(float) * multi_size);
    
    for (size_t i = 0; i < multi_size; i++) {
        dist_INIT[i] = INFINITE;
    }

    //order ID by degree
    csr.orderedCsrV  = (int*)calloc(sizeof(int), (csr.csrVSize) *2);
    for(int i=csr.startNodeID;i<=csr.endNodeID;i++){
            csr.orderedCsrV[i]=i;
    }
    quicksort_nodeID_with_degree(csr.orderedCsrV, csr.csrNodesDegree, csr.startNodeID, csr.endNodeID);
    // cout<<"after sort\n";
    // for(int i=csr.startNodeID;i<=csr.endNodeID;i++){
    //     cout<<csr.orderedCsrV[i]<<"[ "<<csr.csrNodesDegree[csr.orderedCsrV[i]]<<" ]>";
    // }
    // cout<<endl;

    for (int sourceIndex = csr.startNodeID; sourceIndex <= csr.endNodeID; ++sourceIndex) {
        int sourceID =csr.orderedCsrV[sourceIndex];
        if (nodeDone[sourceID]) continue;

        // multi_time1 = seconds();

        nodeDone[sourceID] = true;
        int mappingCount = 0;
        map_S[mappingCount++] = sourceID;

        // Find other sources
        for (int neighborIndex = csr.csrV[sourceID]; neighborIndex < csr.csrV[sourceID + 1] && mappingCount < max_multi; neighborIndex++) {
            int neighborNodeID = csr.csrE[neighborIndex];
            if (!nodeDone[neighborNodeID]) {
                map_S[mappingCount++] = neighborNodeID;
                nodeDone[neighborNodeID] = true;
            }
        }

        // Initialize dist_MULTI, sigma_MULTI, delta_MULTI
        memcpy(dist_MULTI, dist_INIT, sizeof(float) * multi_size);
        memset(sigma_MULTI, 0, sizeof(float) * multi_size);
        memset(delta_MULTI, 0, sizeof(float) * multi_size);
        memset(s_size, 0, sizeof(int) * v_size);

        int f1_indicator = 0;
        int f2_indicator = 0;
        int s_indicator = 0;

        // Initialize currentQueue
        for (int i = 0; i < mappingCount; i++) {
            int sourceNode = map_S[i];
            int position = mappingCount * sourceNode + i;
            sigma_MULTI[position] = 1.0;
            dist_MULTI[position] = 0.0;

            f1[f1_indicator].nodeID = sourceNode;
            f1[f1_indicator].traverse_S = (1ULL << i);
            f1_indicator++;
        }

        // Initialize nextQueue traverse_S to zero
        for (int i = 0; i < v_size; i++) {
            f2[i].traverse_S = 0;
        }

        int level=0;
        while (f1_indicator > 0) {

            Q_struct* currentQueue;
            Q_struct* nextQueue;
            if(level% 2 == 0){
                currentQueue = f1;
                nextQueue = f2;
            }
            else{
                currentQueue = f2;
                nextQueue = f1;
            }
            // Store currentQueue into s[s_indicator]
            s[s_indicator] = (Q_struct*)malloc(f1_indicator * sizeof(Q_struct));
            memcpy(s[s_indicator], currentQueue, f1_indicator * sizeof(Q_struct));
            s_size[s_indicator++] = f1_indicator;

            // Process currentQueue
            for (auto i = 0; i < f1_indicator; i++) {
                int v = currentQueue[i].nodeID;
                uint64_t traverse_S = currentQueue[i].traverse_S;

                for (auto neighborIndex = csr.csrV[v]; neighborIndex < csr.csrV[v + 1]; neighborIndex++) {
                    int neighborNodeID = csr.csrE[neighborIndex];

                    for (auto multi_node = 0; multi_node < mappingCount; multi_node++) {
                        if (traverse_S & (1ULL << multi_node)) {
                            int position_v = mappingCount * v + multi_node;
                            int position_n = mappingCount * neighborNodeID + multi_node;

                            if (dist_MULTI[position_n] == dist_MULTI[position_v] + 1) {
                                sigma_MULTI[position_n] += sigma_MULTI[position_v];
                            } else if (dist_MULTI[position_n] > dist_MULTI[position_v] + 1) {
                                dist_MULTI[position_n] = dist_MULTI[position_v] + 1;
                                sigma_MULTI[position_n] = sigma_MULTI[position_v];

                                // Check if neighborNodeID is already in nextQueue
                                int found = -1;
                                for (auto find = 0; find < f2_indicator; find++) {
                                    if (nextQueue[find].nodeID == neighborNodeID) {
                                        found = find;
                                        break;
                                    }
                                }
                                if (found >= 0) {
                                    nextQueue[found].traverse_S |= (1ULL << multi_node);
                                } else {
                                    nextQueue[f2_indicator].nodeID = neighborNodeID;
                                    nextQueue[f2_indicator].traverse_S = (1ULL << multi_node);
                                    f2_indicator++;
                                }
                            }
                        }
                    }
                }
            }

            // Swap currentQueue and nextQueue
            f1_indicator = f2_indicator;
            f2_indicator = 0;
            level++;
            // Reset nextQueue traverse_S for next iteration
            // for (int i = 0; i < f1_indicator; i++) {
            //     nextQueue[i].traverse_S = 0;
            // }
        }

        // multi_time2 = seconds();
        // multi_forward_Time += (multi_time2 - multi_time1);
        // multi_time1 = seconds();

        // Back-propagation
        for (int layer = s_indicator - 1; layer >= 0; layer--) {
            for (int i = 0; i < s_size[layer]; i++) {
                int v = s[layer][i].nodeID;
                uint64_t traverse_S = s[layer][i].traverse_S;

                for (int multi_node = 0; multi_node < mappingCount; multi_node++) {
                    if (traverse_S & (1ULL << multi_node)) {
                        int position_v = mappingCount * v + multi_node;

                        float coeff = 0.0;

                        // For each neighbor w of v
                        for (int neighborIndex = csr.csrV[v]; neighborIndex < csr.csrV[v + 1]; neighborIndex++) {
                            int w = csr.csrE[neighborIndex];
                            int position_w = mappingCount * w + multi_node;

                            if (dist_MULTI[position_w] == dist_MULTI[position_v] + 1) {
                                coeff += (sigma_MULTI[position_v] / sigma_MULTI[position_w]) * (1.0 + delta_MULTI[position_w]);
                            }
                        }

                        delta_MULTI[position_v] += coeff;

                        // If v is not the source node, accumulate delta into BC[v]
                        if (v != map_S[multi_node]) {
                            BC[v] += delta_MULTI[position_v];
                        }
                    }
                }
            }
            // Free s[layer]
            free(s[layer]);
        }

        // multi_time2 = seconds();
        // multi_backward_Time += (multi_time2 - multi_time1);
    }

    // multi_time_end = seconds();
    // multi_total_time = (multi_time_end - multi_time_start);

    // Free memory
    free(s_size);
    free(dist_MULTI);
    free(sigma_MULTI);
    free(delta_MULTI);
    free(map_S);
    free(nodeDone);
    free(s);
    free(f1);
    free(f2);
    free(dist_INIT);
}



//************************************************ */
//                   平行程式 SS
//************************************************ */

__global__ void resetBC_value(float* dist,int* f1,int* sigma,float* delta,int* stack,int* level,int target,int size){

    register const int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(idx < size){
        dist[idx] = 1<<20;
        sigma[idx] = 0;
        delta[idx] = 0;
        level[idx] = -1;
        f1[idx] = -1;
    }
    f1[0] = target;
    stack[0] = target;
    if(idx == target){
        dist[idx] = 0.0f;
        sigma[idx] = 1;
        level[idx] = 0;
    }
}

__global__ void printArray_int(int* array, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        printf("g_dist[%d] = %d\n", idx, array[idx]);
    }
}
__global__ void printArray_float(float* array, int size,int mappingcount) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        printf("[ ");
        for(int i=0;i<mappingcount;i++){
            printf("%f,", array[idx+i]);
        }
        printf("]>");
        
    }
}
__global__ void printstack_int(int* array, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        printf("stack[%d] = %d\n", idx, array[idx]);
    }
}

__device__ __forceinline__ float atomicMinFloat (float * addr, float value){
     float old;
     old = (value >= 0) ? __int_as_float(atomicMin((int *)addr, __float_as_int(value))) :
          __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));
     return old;
}


__device__  __forceinline__ void atomicOr64(uint64_t* address, uint64_t val) {
    uint32_t* address_as_ui = (uint32_t*)address;
    uint32_t old_lo, old_hi, new_lo, new_hi;

    // 循環直到成功進行原子操作
    do {
        old_lo = address_as_ui[0];
        old_hi = address_as_ui[1];
        new_lo = old_lo | (uint32_t)(val);
        new_hi = old_hi | (uint32_t)(val >> 32);
    } while ((atomicCAS(address_as_ui, old_lo, new_lo) != old_lo) || 
             (atomicCAS(address_as_ui + 1, old_hi, new_hi) != old_hi));
}

__global__ void allBC(int* g_csrV,int* g_csrE ,int* nextQueueSize,int* currentQueue,int* nextQueue,float* dist,int* sigma,int blocknum,int j,int size){

    register const int bid = blockIdx.x + j * blocknum; // 0 + 0 * INT_MAX

    if(bid > size || currentQueue[bid] == -1) return; //大於currentQueueSize

    register const int node = currentQueue[bid];
    register const int degree = g_csrV[node+1] - g_csrV[node];
    register const int threadOffset = (int)ceil(degree/(blockDim.x*1.0)); //需要看的鄰居，疊代次數
    register float     old;
    // printf("bid: %d,node: %d ,degree: %d, blockDim.x: %d\n",bid,node,degree,blockDim.x);
    for(int i=0;i<threadOffset;i++){
        register const int position = g_csrV[node] + threadIdx.x + i * blockDim.x;
        if(position < g_csrV[node+1] ){
            // printf("node: %d ,position: %d, dist: %d\n",node,g_csrE[position],dist[g_csrE[position]]);
            if(dist[node] + 1.0 < dist[g_csrE[position]]){
                //Unweighted
                // dist[g_csrE[position]] = dist[node] + 1;
                //Weighted
                old = atomicMinFloat(&dist[g_csrE[position]], (dist[node] + 1.0));
                // printf("old: %d, dist: %d\n",old,dist[g_csrE[position]]);
                if(old != dist[g_csrE[position]]){
                int next = atomicAdd(nextQueueSize,1);
                nextQueue[next] = __ldg(&g_csrE[position]);
                // printf("nextQueue[%d]: %d\n",next,nextQueue[next]);
                    // printf("%d(%d) %d(%d)\n",node,level[node],adjacencyList[position],level[adjacencyList[position]]);
                    // printf("A: %d(%.2f) --> %d(%.2f)\n",node,dist[node],adjacencyList[position],dist[adjacencyList[position]]);
                }
            }
            if(dist[node] + 1 == dist[g_csrE[position]]){
                atomicAdd(&sigma[g_csrE[position]],sigma[node]);
                //printf("B: %d(%f) --> %d(%f)\n",node,sigma[node],adjacencyList[position],sigma[adjacencyList[position]]);
            }
            // printf("node: %d ,dist: %d, sigma: %d \n",g_csrE[position],dist[g_csrE[position]],sigma[g_csrE[position]]);
        }
    }
}

__global__ void deltaCalculation(int* g_csrV,int* g_csrE,float* g_delta,int* sigma,int* stack,float* dist,int blocknum,int j,int startposition,int size){

    register const int bid = blockIdx.x + j*blocknum;
    register const int node = stack[startposition + bid];
    // printf("traverse node: %d\n",node);
    if(bid >= size || node == -1) return;

    register const int degree = g_csrV[node+1] - g_csrV[node];
    register const int threadOffset = (int)ceil(degree/(blockDim.x*1.0));

    for(int i=0;i<threadOffset;i++) {
        register const int position = g_csrV[node] + threadIdx.x + i * blockDim.x;
        if(position < g_csrV[node+1] && dist[node] - 1.0 == dist[g_csrE[position]]){
            // printf("traverse node: %d\n",node);
            atomicAdd(&g_delta[g_csrE[position]],((float)sigma[g_csrE[position]]/sigma[node])*(1.0+g_delta[node]));
            //printf("%d(%d,%.2f) %d(%d,%.2f)\n",node,level[node],sigma[node],adjacencyList[position],level[adjacencyList[position]],sigma[adjacencyList[position]]);
            // printf("g_delta[%d]: %f\n",g_csrE[position],g_delta[g_csrE[position]]);

        }
    }

}

__global__ void sum_BC_Result(float* result,float* delta,int size,int s){

    register const int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idx < size && idx!=s){
        result[idx] += delta[idx];
        // printf("BC[%d]: %f\n",idx,result[idx]);
    }
        

}



void brandes_SS_par( CSR& csr, int V, float *BC) {

    //CPU variable
    int    currentQueueSize;
    int*   stackOffset = (int*)calloc(V,sizeof(int));
    //GPU MALLOC　variable
    int*   g_stack;      
    int*   g_sigma;     
    float* g_dist;
    int*   g_level;     
    float* g_delta; 
    int*   g_S_size;
    int*   g_f1;
    int*   g_f2;
    int*   g_nextQueueSize; //用來回傳給CPU判別currentQueueSize，是否繼續traverse
    int*   g_csrE;
    int*   g_csrV;
    float* g_BC;

    // printf("start malloc\n");
    hipMalloc((void **)&g_stack,V * sizeof(int)); //用CPU的stack offset存每一層的位置
    hipMalloc((void **)&g_sigma,V * sizeof(int));
    hipMalloc((void **)&g_dist,V * sizeof(float));
    hipMalloc((void **)&g_level,V * sizeof(int));
    hipMalloc((void **)&g_delta,V * sizeof(float));
    hipMalloc((void **)&g_S_size,V*sizeof(int));
    
    size_t free_byte;
    size_t total_byte;
    hipMemGetInfo(&free_byte,&total_byte);
    hipMalloc((void **)&g_f1, free_byte * 0.3);
    hipMalloc((void **)&g_f2, free_byte * 0.3);
    hipMalloc((void **)&g_nextQueueSize,sizeof(int));
    hipMalloc((void **)&g_csrV, V * sizeof(int));
    hipMalloc((void **)&g_csrE, csr.csrESize * sizeof(int));
    hipMalloc((void **)&g_BC, V * sizeof(float));
    hipMemcpy(g_csrV,csr.csrV ,  V * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(g_csrE,csr.csrE ,  csr.csrESize * sizeof(int),hipMemcpyHostToDevice);
    hipMemset(g_BC, 0.0f, V * sizeof(float));
    // printf("end malloc\n");
    // std::cout << "Total GPU memory: " << total_byte / (1024.0 * 1024.0) << " MB" << std::endl;
    // std::cout << "Free GPU memory: " << free_byte / (1024.0 * 1024.0) << " MB" << std::endl;
    int threadnum = 32;
   
    for (int s = csr.startNodeID; s <= csr.endNodeID; ++s) {
        // Initialize variables for each source node
        
        //初始g_f1 queue
        resetBC_value<<<ceil(V/64.0),min(V,64)>>>(g_dist,g_f1,g_sigma,g_delta,g_stack,g_level,s,V);
        CHECK(hipDeviceSynchronize());
        hipMemset(g_nextQueueSize,0,sizeof(int));
        currentQueueSize = 1;
        
        int level =0;
        // BFS forward phase: frontier-based BFS with extra mallocs
        while (currentQueueSize>0) { //!qIsEmpty(current_queue)
            // printf(" forward level: %d\n",level);
            // printf("currentQueueSize: %d\n",currentQueueSize);
            // Allocate new memory for next_queue in each iteration
            int *g_currentQueue;
            int *g_nextQueue;
            if(level% 2 == 0){
                g_currentQueue = g_f1;
                g_nextQueue = g_f2;
            }
            else{
                g_currentQueue = g_f2;
                g_nextQueue = g_f1;
            }

            stackOffset[level+1] = currentQueueSize + stackOffset[level];
            int blocknum = (currentQueueSize < INT_MAX) ? currentQueueSize : INT_MAX;
            //平行跑BFS
            for(int i=0;i<(int)ceil(currentQueueSize/(float)INT_MAX);i++){
                allBC<<<blocknum,threadnum>>>(g_csrV,g_csrE,g_nextQueueSize,g_currentQueue,g_nextQueue,g_dist,g_sigma,INT_MAX,i,currentQueueSize);
                CHECK(hipDeviceSynchronize());
            }
                    

            
            hipMemcpy(&currentQueueSize,g_nextQueueSize,sizeof(int),hipMemcpyDeviceToHost);
            hipMemcpy(&g_stack[stackOffset[level+1]],g_nextQueue,currentQueueSize*sizeof(int),hipMemcpyDeviceToDevice);
            hipMemset(g_nextQueueSize,0,sizeof(int));
           
            level++;
            
        }
        // printstack_int<<<ceil(V/64.0),min(V,64)>>>(g_stack,V);
        // hipDeviceSynchronize();
        // printArray_int<<<ceil(V/64.0),min(V,64)>>>(g_dist,V);
        // Backward phase to compute BC values
        // for(int st=0;st<V;st++){
        //     printf("stackOffset[%d]: %d\n",st,stackOffset[st]);
        // }
        // printf("total level: %d\n",level);
        for (int d = level - 1; d >= 0; --d) {
            // std::cout << "backward level(" << d << "):\t" << stackOffset[d+1] - stackOffset[d] << std::endl;
            int degree =(stackOffset[d+1] - stackOffset[d]);
            int blocknum = (degree < INT_MAX) ? degree : INT_MAX;
            
            for(int i=0;i<(int)ceil(degree/(float)INT_MAX);i++)
                deltaCalculation<<<blocknum,threadnum>>>(g_csrV,g_csrE,g_delta,g_sigma,g_stack,g_dist,INT_MAX,i,stackOffset[d],degree);
            
            CHECK(hipDeviceSynchronize());
            // printArray_float<<<ceil(V/64.0),min(V,64)>>>(g_delta,V);
        }
        // printArray_float<<<ceil(V/64.0),min(V,64)>>>(g_delta,V);

        sum_BC_Result<<<ceil(V/64.0),min(V,64)>>>(g_BC,g_delta,V,s);
        CHECK(hipDeviceSynchronize());
    }
    hipMemcpy(BC,g_BC ,  V * sizeof(int),hipMemcpyDeviceToHost);
    // Free memory for S and its levels
    free(stackOffset);
    hipFree(g_sigma);
    hipFree(g_delta);
    hipFree(g_stack);
    hipFree(g_level);
    hipFree(g_dist);
    hipFree(g_f1);
    hipFree(g_f2);
    hipFree(g_nextQueueSize);
    hipFree(g_BC);
}


//************************************************ */
//                   平行程式 MS
//************************************************ */


__global__ void resetBC_value_MS(float* dist,Q_struct* f1,Q_struct* f2,int* sigma,float* delta,Q_struct* stack,int target,int size){

    register const int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(idx < size){
        dist[idx] = 1<<20;
        sigma[idx] = 0;
        delta[idx] = 0.0f;
        f1[idx].nodeID = -1;
        f1[idx].traverse_S=0;
        f2[idx].nodeID = -1;
        f2[idx].traverse_S=0;
    }
    // f1[0].nodeID = target;
    // stack[0].nodeID = target;
    // if(idx == target){
    //     dist[idx] = 0.0f;
    //     sigma[idx] = 1;
    // }
}

__global__ void INITIAL_value_MS(float* dist,Q_struct* f1,int* sigma,int* g_map_S,int size){

    register const int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(idx < size){
        register const int sourceID = g_map_S[idx];
        register const int position = size * sourceID + idx; //size:mapping count
        // printf("position: %d idx: %d\n",position,idx);
        sigma[position]=1;
        dist[position]=0.0f;
        f1[idx].nodeID = sourceID;
        f1[idx].traverse_S=1ULL << idx;
    }
    // f1[0].nodeID = target;
    // stack[0].nodeID = target;
    // if(idx == target){
    //     dist[idx] = 0.0f;
    //     sigma[idx] = 1;
    // }
}

__global__ void deltaCalculation_MS(int* g_csrV,int* g_csrE,float* g_delta,int* sigma,Q_struct* stack,float* dist,int blocknum,int j,int startposition,int size,int mappingcount){

    register const int bid = blockIdx.x + j*blocknum;
    register const int node = stack[startposition + bid].nodeID;
    register const uint64_t traverse_S = stack[startposition + bid].traverse_S;
    // printf("traverse node: %d\n",node);
    if(bid >= size || node == -1) return;

    register const int degree = g_csrV[node+1] - g_csrV[node];
    register const int threadOffset = (int)ceil(degree/(blockDim.x*1.0));

    for(int i=0;i<threadOffset;i++) {
        register const int position = g_csrV[node] + threadIdx.x + i * blockDim.x;
        if(position < g_csrV[node+1]){ //&& dist[node] - 1 == dist[g_csrE[position]]
            // printf("traverse node: %d\n",node);
            register const int neighborNodeID = g_csrE[position];
            for (int multi_node = 0; multi_node < mappingcount; multi_node++) {
                // printf("multi_node: %d \n",multi_node);
                if (traverse_S & (1ULL << multi_node)) {
                    register const int position_v = mappingcount * node           + multi_node;
                    register const int position_n = mappingcount * neighborNodeID + multi_node;
                    if(dist[position_v] - 1.0 == dist[position_n]){
                        atomicAdd(&g_delta[position_n],((float)sigma[position_n]/sigma[position_v])*(1.0+g_delta[position_v]));
                    }
                }
            }  
            //printf("%d(%d,%.2f) %d(%d,%.2f)\n",node,level[node],sigma[node],adjacencyList[position],level[adjacencyList[position]],sigma[adjacencyList[position]]);
            // printf("g_delta[%d]: %f\n",g_csrE[position],g_delta[g_csrE[position]]);

        }
    }

}

__global__ void INITIAL_Qtruct(Q_struct* f1,int size){

    register const int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < size){
        f1[idx].nodeID = -1;
        f1[idx].traverse_S=0;
    }
  
}

__global__ void sum_BC_Result_MS(float* result,float* delta,int size,int* map_S,int mappingCount){
    extern __shared__ int shared_map_S[];
    register const int idx = threadIdx.x + blockIdx.x * blockDim.x;

     // Load map_S into shared memory
    if (threadIdx.x < mappingCount) {
        shared_map_S[threadIdx.x] = map_S[threadIdx.x];
    }
    __syncthreads();

    if(idx < size){
        // #pragma unroll
        for(int i=0;i<mappingCount;i++){
            if(shared_map_S[i]==idx)
                continue;
            
            // atomicAdd(&result[idx], delta[idx*mappingCount+i]);
            result[idx] += delta[idx*mappingCount+i];
        }
        // printf("BC[%d]: %f\n",idx,result[idx]);
    }
        

}

__global__ void allBC_MS(int* g_csrV,int* g_csrE ,int* nextQueueSize,Q_struct* currentQueue,Q_struct* nextQueue,float* dist,int* sigma,int blocknum,int j,int size, int mappingcount){

    register const int bid = blockIdx.x + j * blocknum; // 0 + 0 * INT_MAX

    if(bid > size || currentQueue[bid].nodeID == -1) return; //大於currentQueueSize
    
    register const int node = currentQueue[bid].nodeID;
    register const uint64_t traverse_S = currentQueue[bid].traverse_S; //我改這裡
    register const int degree = g_csrV[node+1] - g_csrV[node];
    register const int threadOffset = (int)ceil(degree/(blockDim.x*1.0)); //需要看的鄰居，疊代次數
    register float     old;
    register int next;
    // printf("bid: %d,node: %d ,degree: %d, threadOffset: %d\n",bid,node,degree,threadOffset);
    for(int i=0;i<threadOffset;i++){
        register const int position = g_csrV[node] + threadIdx.x + i * blockDim.x; //該點node的鄰居位置
        if(position < g_csrV[node+1] ){
            register const int neighborNodeID = g_csrE[position];
            // printf("node: %d ,neighbor: %d threadOffset: %d\n",node,neighborNodeID,threadOffset);
            
            for (int multi_node = 0; multi_node < mappingcount; multi_node++) {
                // printf("multi_node: %d \n",multi_node);
                if (traverse_S & (1ULL << multi_node)) {
                    register const int position_v = mappingcount * node           + multi_node;
                    register const int position_n = mappingcount * neighborNodeID + multi_node;
                    // printf("node: %d ,neighbor: %d,pos_v: %d,pos_n: %d ,multi_node:%d\n",node,neighborNodeID,position_v,position_n,multi_node);
                    // 更新 dist_MULTI 和 sigma_MULTI
                    
                    // printf("ok1\n");
                    if (dist[position_n] > dist[position_v] + 1.0f) {
                        old = atomicMinFloat(&dist[position_n], (dist[position_v] + 1.0));
                        
                        if(old != dist[position_n]){
                            sigma[position_n] = 0;
                            // 检查 neighborNodeID 是否已在 nextQueue 中
                            // printf("node: %d ,neighbor: %d\n",node,neighborNodeID);
                            // printf("node: %d ,neighbor: %d\n",node,neighborNodeID);
                            next = atomicAdd(nextQueueSize,1);
                            nextQueue[next].nodeID = __ldg(&g_csrE[position]);
                            // nextQueue[next].nodeID     = neighborNodeID;
                            nextQueue[next].traverse_S = (1ULL << multi_node);
                            // bool found = false;
                            // for (int find = 0; find < next ; find++) {
                            //     // printf("enxtQueue[find].nodeID: %d\t neighborNodeID: %d\n",nextQueue[find].nodeID,neighborNodeID);
                            //     if (nextQueue[find].nodeID == neighborNodeID) {
                                    
                                    
                            //         // printf("node: %d ,neighbor: %d, nextQueueSize: %d\n",node,neighborNodeID,next);
                            //         nextQueue[find].traverse_S|=(1ULL << multi_node);
                            //         found = true;
                            //         atomicAdd(nextQueueSize,-1);
                            //         // break;
                            //     }
                            // }
                            
                            // if(found ==false){
                            //     nextQueue[next].nodeID = __ldg(&g_csrE[position]);
                            //     // printf("node: %d ,neighbor: %d, traverse_S: %d\n",node,neighborNodeID,(1ULL << multi_node));
                            //     // nextQueue[next].nodeID = neighborNodeID;
                            //     nextQueue[next].traverse_S = (1ULL << multi_node);
                            // }
                           
                        }
                        
                        // dist[position_n] = dist[position_v] + 1;
                        // sigma[position_n] = sigma[position_v];
                    }

                    if (dist[position_n] == dist[position_v] + 1.0f) {
                        atomicAdd(&sigma[position_n], sigma[position_v]);
                    }

                    // break;
                }
            }    
        }
    }

}

__global__ void allBC_MS_VnextQ(int* g_csrV,int* g_csrE ,Q_struct* currentQueue,Q_struct* nextQueue,float* dist,int* sigma,int blocknum,int j,int size, int mappingcount){

    register const int bid = blockIdx.x + j * blocknum; // 0 + 0 * INT_MAX

    if(bid > size || currentQueue[bid].nodeID == -1) return; //大於currentQueueSize
    
    register const int node = currentQueue[bid].nodeID;
    register const uint64_t traverse_S = currentQueue[bid].traverse_S; //我改這裡
    register const int degree = g_csrV[node+1] - g_csrV[node];
    register const int threadOffset = (int)ceil(degree/(blockDim.x*1.0)); //需要看的鄰居，疊代次數
    register float     old;
    // printf("bid: %d,node: %d ,degree: %d, threadOffset: %d\n",bid,node,degree,threadOffset);
    for(int i=0;i<threadOffset;i++){
        register const int position = g_csrV[node] + threadIdx.x + i * blockDim.x; //該點node的鄰居位置
        if(position < g_csrV[node+1] ){
            register const int neighborNodeID = g_csrE[position];
            // printf("node: %d ,neighbor: %d threadOffset: %d\n",node,neighborNodeID,threadOffset);
            
            for (int multi_node = 0; multi_node < mappingcount; multi_node++) {
                // printf("multi_node: %d \n",multi_node);
                if (traverse_S & (1ULL << multi_node)) {
                    register const int position_v = mappingcount * node           + multi_node;
                    register const int position_n = mappingcount * neighborNodeID + multi_node;
                    // printf("node: %d ,neighbor: %d,pos_v: %d,pos_n: %d ,multi_node:%d\n",node,neighborNodeID,position_v,position_n,multi_node);
                    // 更新 dist_MULTI 和 sigma_MULTI
                    
                    // printf("ok1\n");
                    if (dist[position_n] > dist[position_v] + 1.0f) {
                        old = atomicMinFloat(&dist[position_n], (dist[position_v] + 1.0));
                        
                        if(old != dist[position_n]){
                            sigma[position_n] = 0;
                            // 检查 neighborNodeID 是否已在 nextQueue 中
                            // printf("node: %d ,neighbor: %d\n",node,neighborNodeID);
                            // printf("node: %d ,neighbor: %d\n",node,neighborNodeID);
                            nextQueue[g_csrE[position]].nodeID = __ldg(&g_csrE[position]);
                            // nextQueue[g_csrE[position]].traverse_S |= (1ULL << multi_node);
                            atomicOr64(&nextQueue[g_csrE[position]].traverse_S,(1ULL << multi_node));
                            
                            
                            // bool found = false;
                            // for (int find = 0; find < next ; find++) {
                            //     // printf("enxtQueue[find].nodeID: %d\t neighborNodeID: %d\n",nextQueue[find].nodeID,neighborNodeID);
                            //     if (nextQueue[find].nodeID == neighborNodeID) {
                                    
                                    
                            //         // printf("node: %d ,neighbor: %d, nextQueueSize: %d\n",node,neighborNodeID,next);
                            //         nextQueue[find].traverse_S|=(1ULL << multi_node);
                            //         found = true;
                            //         atomicAdd(nextQueueSize,-1);
                            //         // break;
                            //     }
                            // }
                            
                            // if(found ==false){
                            //     nextQueue[next].nodeID = __ldg(&g_csrE[position]);
                            //     // printf("node: %d ,neighbor: %d, traverse_S: %d\n",node,neighborNodeID,(1ULL << multi_node));
                            //     // nextQueue[next].nodeID = neighborNodeID;
                            //     nextQueue[next].traverse_S = (1ULL << multi_node);
                            // }
                           
                        }
                        
                        // dist[position_n] = dist[position_v] + 1;
                        // sigma[position_n] = sigma[position_v];
                    }

                    if (dist[position_n] == dist[position_v] + 1.0f) {
                        atomicAdd(&sigma[position_n], sigma[position_v]);
                    }

                    // break;
                }
            }    
        }
    }

}


__global__ void rearrange_queue_MS(Q_struct* nextQueue,Q_struct* nextQueue_temp,int* nextQueueSize, const int V) { 
    register const int idx = threadIdx.x + blockIdx.x * blockDim.x;    
    register int next;
    if(idx < V) {
        if(nextQueue_temp[idx].nodeID!=-1){
            next = atomicAdd(nextQueueSize,1);
            nextQueue[next]=nextQueue_temp[idx];
        }
    }
}


//這個版本為1，相同source的分開不同block的，速度變慢
void brandes_MS_par( CSR& csr, int max_multi, float* BC) {
    // Start timing
    // multi_time_start = seconds();

    int V = csr.csrVSize;
    int multi_size = V * max_multi;
    //CPU variable
    int    currentQueueSize;
    int*   stackOffset = (int*)calloc(multi_size,sizeof(int));
    int*   map_S = (int*)malloc(sizeof(int) * max_multi); // Multiple sources
    bool*  nodeDone = (bool*)calloc(V, sizeof(bool));
    //CPU print 專用
    int*   sigma    = (int*)malloc(sizeof(int) * multi_size);
    float* dist     = (float*)malloc(sizeof(float) * multi_size);
    Q_struct*  f1   = (Q_struct*)malloc(sizeof(Q_struct) * V);
    Q_struct*  queue   = (Q_struct*)malloc(sizeof(Q_struct) * V);
    float*  delta   = (float*)malloc(sizeof(float) * multi_size);
    //GPU MALLOC　variable
    Q_struct*   g_stack;       
    int*   g_sigma;     
    float* g_dist;
    int*   g_level;     
    float* g_delta; 
    int*   g_S_size;
    Q_struct*   g_f1;
    Q_struct*   g_f2;
    Q_struct*   nextQueue_temp;
    int*   g_nextQueueSize; //用來回傳給CPU判別currentQueueSize，是否繼續traverse
    int*   g_csrE;
    int*   g_csrV;
    int*   g_map_S;   //用來記錄Source對應的node array位置: 0 1 2 -> node 22 15 6
    float* g_BC;


    // printf("start malloc\n");
    hipMalloc((void **)&g_stack,multi_size * sizeof(Q_struct)); //用CPU的stack offset存每一層的位置 因為node可能在不同層重複出現，所以要開multi_size的大小
    hipMalloc((void **)&nextQueue_temp,V * sizeof(Q_struct));
    hipMalloc((void **)&g_sigma,multi_size * sizeof(int));
    hipMalloc((void **)&g_dist,multi_size * sizeof(float));
    hipMalloc((void **)&g_level,V * sizeof(int));
    hipMalloc((void **)&g_delta,multi_size * sizeof(float));
    hipMalloc((void **)&g_S_size,V* sizeof(int));
    
    size_t free_byte;
    size_t total_byte;
    hipMemGetInfo(&free_byte,&total_byte);
    hipMalloc((void **)&g_f1, multi_size * sizeof(Q_struct)); //free_byte * 0.3
    hipMalloc((void **)&g_f2, multi_size * sizeof(Q_struct)); //free_byte * 0.3
    hipMalloc((void **)&g_nextQueueSize,sizeof(int));
    hipMalloc((void **)&g_csrV, V * sizeof(int));
    hipMalloc((void **)&g_csrE, csr.csrESize * sizeof(int));
    hipMalloc((void **)&g_BC, V * sizeof(float));
    hipMalloc((void **)&g_map_S, max_multi * sizeof(int));
    hipMemcpy(g_csrV,csr.csrV ,  V * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(g_csrE,csr.csrE ,  csr.csrESize * sizeof(int),hipMemcpyHostToDevice);
    hipMemset(g_BC, 0.0f, V * sizeof(float));
    // printf("end malloc\n");
    // std::cout << "Total GPU memory: " << total_byte / (1024.0 * 1024.0) << " MB" << std::endl;
    // std::cout << "Free GPU memory: " << free_byte / (1024.0 * 1024.0) << " MB" << std::endl;
    int threadnum = 32;


    //origin

    for (int sourceID = csr.startNodeID; sourceID <= csr.endNodeID; ++sourceID) {
        if (nodeDone[sourceID]) continue;

        // multi_time1 = seconds();

        nodeDone[sourceID] = true;
        int mappingCount = 0;
        map_S[mappingCount++] = sourceID;

        // Find other sources
        for (int neighborIndex = csr.csrV[sourceID]; neighborIndex < csr.csrV[sourceID + 1] && mappingCount < max_multi; neighborIndex++) {
            int neighborNodeID = csr.csrE[neighborIndex];
            if (!nodeDone[neighborNodeID]) {
                map_S[mappingCount++] = neighborNodeID;
                nodeDone[neighborNodeID] = true;
            }
        }
    
        //亂挑multi-source的程式
        // for (int neighborIndex = csr.startNodeID; neighborIndex <= csr.endNodeID && mappingCount < max_multi; neighborIndex++) {
        //     int neighborNodeID = csr.csrE[neighborIndex];
        //     if (!nodeDone[neighborNodeID]) {
        //         map_S[mappingCount++] = neighborNodeID;
        //         nodeDone[neighborNodeID] = true;
        //     }
        // }


        // Initialize dist_MULTI, sigma_MULTI, delta_MULTI
        //初始g_f1 queue
        hipMemcpy(g_map_S,map_S, mappingCount * sizeof(int),hipMemcpyHostToDevice);
        resetBC_value_MS<<<ceil(multi_size/128.0),min(multi_size,128)>>>(g_dist,g_f1,g_f2,g_sigma,g_delta,g_stack,sourceID,multi_size);
        hipMemset(g_nextQueueSize,0,sizeof(int));
        currentQueueSize = mappingCount;
        memset(stackOffset, 0, sizeof(int) * multi_size);
        INITIAL_value_MS<<<ceil(mappingCount/128.0),min(mappingCount,128)>>>(g_dist,g_f1,g_sigma,g_map_S,mappingCount);
        hipDeviceSynchronize();
        #pragma  region print
        //檢查GPU資料
        // hipMemcpy(sigma,g_sigma, multi_size * sizeof(int),hipMemcpyHostToHost);
        // hipMemcpy(dist, g_dist,   multi_size * sizeof(float),hipMemcpyHostToHost);
        // hipMemcpy(f1, g_f1,   mappingCount * sizeof(Q_struct),hipMemcpyDeviceToHost);

        // printf("sigma: ");
        // for(int i=0;i<V;i++){
        //     printf("[");
        //     for(int j=0;j<mappingCount;j++){
        //         printf("%d,",sigma[mappingCount*i+j]);
        //     }
        //     printf("] ");
        // }
        // printf("\n");

        // printf("dist: ");
        // for(int i=0;i<V;i++){
        //     printf("[");
        //     for(int j=0;j<mappingCount;j++){
        //         printf("%.0f,",dist[mappingCount*i+j]);
        //     }
        //     printf("] ");
        // }
        // printf("\n");

        // printf("--------------------multi Source--------------------");
        // for(int i=0;i<max_multi;i++){
        //     printf("%d ",map_S[i]);
        // }
        // printf("\n");

        // printf("f1: ");
        // for(int i=0;i<multi_size;i++){
        //     printf("[%d, %d]> ",f1[i].nodeID,f1[i].traverse_S);
        // }
        // printf("\n");
        #pragma  endregion

        // int f1_indicator = 0;
        // int f2_indicator = 0;
        // int s_indicator = 0;
       

        int level=0;
        while (currentQueueSize > 0) { //currentQueueSize > 0
            // std::cout<<"currentQueueSize: "<<currentQueueSize<<std::endl;
            Q_struct* g_currentQueue;
            Q_struct* g_nextQueue;
            if(level% 2 == 0){
                g_currentQueue = g_f1;
                g_nextQueue = g_f2;
            }
            else{
                g_currentQueue = g_f2;
                g_nextQueue = g_f1;
            }
            
            stackOffset[level+1] = currentQueueSize + stackOffset[level];
            int blocknum = (currentQueueSize < INT_MAX) ? currentQueueSize : INT_MAX;
            //平行跑BFS
            // printf("currentQueueSize: %d\n",currentQueueSize);
            for(int i=0;i<(int)ceil(currentQueueSize/(float)INT_MAX);i++){
                allBC_MS<<<blocknum,threadnum>>>(g_csrV,g_csrE,g_nextQueueSize,g_currentQueue,g_nextQueue,g_dist,g_sigma,INT_MAX,i,currentQueueSize,mappingCount);
                // allBC_MS_VnextQ<<<blocknum,threadnum>>>(g_csrV,g_csrE,g_currentQueue,g_nextQueue,g_dist,g_sigma,INT_MAX,i,currentQueueSize,mappingCount);
                hipDeviceSynchronize();
                // CHECK(hipMemcpy(&nextQueueSize_temp,g_nextQueueSize,sizeof(int),hipMemcpyDeviceToHost));
                // int shared_mem_size = (*g_nextQueueSize) * sizeof(Q_struct);
                // rearrange_queue_MS<<<1,1>>>(nextQueue_temp,g_nextQueueSize,g_nextQueue);
                // hipDeviceSynchronize();
            }
                    

            // Swap currentQueue and nextQueue
            // CHECK(hipMemcpy(&currentQueueSize,g_nextQueueSize,sizeof(int),hipMemcpyDeviceToHost));
            // CHECK(hipMemcpy(&g_stack[stackOffset[level+1]],g_nextQueue,currentQueueSize*sizeof(Q_struct),hipMemcpyDeviceToDevice));
            // CHECK(hipMemset(g_nextQueueSize,0,sizeof(int)));
            hipMemcpy(&currentQueueSize,g_nextQueueSize,sizeof(int),hipMemcpyDeviceToHost);
            hipMemcpy(&g_stack[stackOffset[level+1]],g_nextQueue,currentQueueSize*sizeof(Q_struct),hipMemcpyDeviceToDevice);
            hipMemset(g_nextQueueSize,0,sizeof(int));
            level++;


            #pragma  region print
            // CHECK(hipMemcpy(&queue[0],g_nextQueue, currentQueueSize*sizeof(Q_struct),hipMemcpyDeviceToHost));
            // printf("f1: ");
            // for(int i=0;i<currentQueueSize;i++){
            //     printf("[%d, %lu]> ",queue[i].nodeID,queue[i].traverse_S);
            // }
            // printf("\n");
            #pragma  endregion


        }

        #pragma  region print
        //檢查GPU資料
        // hipMemcpy(sigma,g_sigma, multi_size * sizeof(int),hipMemcpyDeviceToHost);
        // hipMemcpy(dist, g_dist,   multi_size * sizeof(float),hipMemcpyDeviceToHost);
        // hipMemcpy(f1, g_f1,   mappingCount * sizeof(Q_struct),hipMemcpyDeviceToHost);

        // printf("sigma: ");
        // for(int i=0;i<V;i++){
        //     printf("[");
        //     for(int j=0;j<mappingCount;j++){
        //         printf("%d,",sigma[mappingCount*i+j]);
        //     }
        //     printf("] ");
        // }
        // printf("\n");

        // printf("dist: ");
        // for(int i=0;i<V;i++){
        //     printf("[");
        //     for(int j=0;j<mappingCount;j++){
        //         printf("%.0f,",dist[mappingCount*i+j]);
        //     }
        //     printf("] ");
        // }
        // printf("\n");


        // printf("f1: ");
        // for(int i=0;i<multi_size;i++){
        //     printf("[%d, %d]> ",f1[i].nodeID,f1[i].traverse_S);
        // }
        // printf("\n");
        #pragma  endregion



        // multi_time2 = seconds();
        // multi_forward_Time += (multi_time2 - multi_time1);
        // multi_time1 = seconds();

        // Back-propagation
        //  std::cout << "--------------backward--------------"<< std::endl;
        for (int d = level - 1; d > 0; d--) {
            int degree =(stackOffset[d+1] - stackOffset[d]);
            int blocknum = (degree < INT_MAX) ? degree : INT_MAX;
            // std::cout << "backward level(" << d << "):\t" << stackOffset[d+1] - stackOffset[d] << std::endl;
            for(int i=0;i<(int)ceil(degree/(float)INT_MAX);i++){
                deltaCalculation_MS<<<blocknum,threadnum>>>(g_csrV,g_csrE,g_delta,g_sigma,g_stack,g_dist,INT_MAX,i,stackOffset[d],degree,mappingCount);
                CHECK(hipDeviceSynchronize());
            }
                
            // hipDeviceSynchronize();
            #pragma  region print
            // CHECK(hipMemcpy(&delta[0], g_delta,   multi_size * sizeof(float),hipMemcpyDeviceToHost));
            // printf("delta: ");
            // for(int i=0;i<V;i++){
            //     printf("[");
            //     for(int j=0;j<mappingCount;j++){
            //         printf("%.3f,",delta[mappingCount*i+j]);
            //     }
            //     printf("] ");
            // }
            // printf("\n");
            #pragma  endregion
        }
        int shared_mem_size = (mappingCount) * sizeof(int);
        sum_BC_Result_MS<<<ceil(V/128.0),min(V,128),shared_mem_size>>>(g_BC,g_delta,V,g_map_S,mappingCount);
        CHECK(hipDeviceSynchronize());

        

        // multi_time2 = seconds();
        // multi_backward_Time += (multi_time2 - multi_time1);
    }
    CHECK(hipMemcpy(&BC[0],g_BC, V*sizeof(float),hipMemcpyDeviceToHost));
    // multi_time_end = seconds();
    // multi_total_time = (multi_time_end - multi_time_start);

   
}

//這個版本為1，相同source的擠在相同block的，速度快
void brandes_MS_par_VnextQ( CSR& csr, int max_multi, float* BC) {
    // Start timing
    // multi_time_start = seconds();

    int V = csr.csrVSize;
    int multi_size = V * max_multi;
    //CPU variable
    int    currentQueueSize;
    int*   stackOffset = (int*)calloc(multi_size,sizeof(int));
    int*   map_S = (int*)malloc(sizeof(int) * max_multi); // Multiple sources
    bool*  nodeDone = (bool*)calloc(V, sizeof(bool));
    //CPU print 專用
    int*   sigma    = (int*)malloc(sizeof(int) * multi_size);
    float* dist     = (float*)malloc(sizeof(float) * multi_size);
    Q_struct*  f1   = (Q_struct*)malloc(sizeof(Q_struct) * V);
    Q_struct*  queue   = (Q_struct*)malloc(sizeof(Q_struct) * multi_size);
    float*  delta   = (float*)malloc(sizeof(float) * multi_size);
    //GPU MALLOC　variable
    Q_struct*   g_stack;       
    int*   g_sigma;     
    float* g_dist;
    int*   g_level;     
    float* g_delta; 
    int*   g_S_size;
    Q_struct*   g_f1;
    Q_struct*   g_f2;
    Q_struct*   g_nextQueue_temp;
    int*   g_nextQueueSize; //用來回傳給CPU判別currentQueueSize，是否繼續traverse
    int*   g_csrE;
    int*   g_csrV;
    int*   g_map_S;   //用來記錄Source對應的node array位置: 0 1 2 -> node 22 15 6
    float* g_BC;


    // printf("start malloc\n");
    hipMalloc((void **)&g_stack,multi_size * sizeof(Q_struct)); //用CPU的stack offset存每一層的位置 因為node可能在不同層重複出現，所以要開multi_size的大小
    hipMalloc((void **)&g_sigma,multi_size * sizeof(int));
    hipMalloc((void **)&g_dist,multi_size * sizeof(float));
    hipMalloc((void **)&g_level,V * sizeof(int));
    hipMalloc((void **)&g_delta,multi_size * sizeof(float));
    hipMalloc((void **)&g_S_size,V* sizeof(int));
    
    size_t free_byte;
    size_t total_byte;
    hipMemGetInfo(&free_byte,&total_byte);
    hipMalloc((void **)&g_f1, multi_size * sizeof(Q_struct)); //free_byte * 0.3
    hipMalloc((void **)&g_f2, multi_size * sizeof(Q_struct)); //free_byte * 0.3
    hipMalloc((void **)&g_nextQueue_temp,V * sizeof(Q_struct));
    hipMalloc((void **)&g_nextQueueSize,sizeof(int));
    hipMalloc((void **)&g_csrV, V * sizeof(int));
    hipMalloc((void **)&g_csrE, csr.csrESize * sizeof(int));
    hipMalloc((void **)&g_BC, V * sizeof(float));
    hipMalloc((void **)&g_map_S, max_multi * sizeof(int));
    hipMemcpy(g_csrV,csr.csrV ,  V * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(g_csrE,csr.csrE ,  csr.csrESize * sizeof(int),hipMemcpyHostToDevice);
    hipMemset(g_BC, 0.0f, V * sizeof(float));
    // printf("end malloc\n");
    // std::cout << "Total GPU memory: " << total_byte / (1024.0 * 1024.0) << " MB" << std::endl;
    // std::cout << "Free GPU memory: " << free_byte / (1024.0 * 1024.0) << " MB" << std::endl;
    int threadnum = 32;

    //order ID by degree
    csr.orderedCsrV  = (int*)calloc(sizeof(int), (csr.csrVSize) *2);
    for(int i=csr.startNodeID;i<=csr.endNodeID;i++){
            csr.orderedCsrV[i]=i;
    }
    quicksort_nodeID_with_degree(csr.orderedCsrV, csr.csrNodesDegree, csr.startNodeID, csr.endNodeID);


    //origin
    for (int sourceIndex = csr.startNodeID; sourceIndex <= csr.endNodeID; ++sourceIndex) {
        int sourceID=csr.orderedCsrV[sourceIndex];
        if (nodeDone[sourceID]) continue;

        // multi_time1 = seconds();

        nodeDone[sourceID] = true;
        int mappingCount = 0;
        map_S[mappingCount++] = sourceID;

        // Find other sources
        // for (int neighborIndex = csr.csrV[sourceID]; neighborIndex < csr.csrV[sourceID + 1] && mappingCount < max_multi; neighborIndex++) {
        //     int neighborNodeID = csr.csrE[neighborIndex];
        //     if (!nodeDone[neighborNodeID]) {
        //         map_S[mappingCount++] = neighborNodeID;
        //         nodeDone[neighborNodeID] = true;
        //     }
        // }
        for (int neighborIndex = csr.startNodeID; neighborIndex <= csr.endNodeID && mappingCount < max_multi; neighborIndex++) {
            int neighborNodeID = csr.orderedCsrV[neighborIndex];
            if (!nodeDone[neighborNodeID]) {
                map_S[mappingCount++] = neighborNodeID;
                nodeDone[neighborNodeID] = true;
            }
        }


        // Initialize dist_MULTI, sigma_MULTI, delta_MULTI
        //初始g_f1 queue
        hipMemcpy(g_map_S,map_S, mappingCount * sizeof(int),hipMemcpyHostToDevice);
        resetBC_value_MS<<<ceil(multi_size/64.0),min(multi_size,64)>>>(g_dist,g_f1,g_f2,g_sigma,g_delta,g_stack,sourceID,multi_size);
        hipDeviceSynchronize();
        hipMemset(g_nextQueueSize,0,sizeof(int));
        currentQueueSize = mappingCount;
        memset(stackOffset, 0, sizeof(int) * multi_size);
        INITIAL_value_MS<<<ceil(mappingCount/64.0),min(mappingCount,64)>>>(g_dist,g_f1,g_sigma,g_map_S,mappingCount);
        hipDeviceSynchronize();
        #pragma  region print
        //檢查GPU資料
        // hipMemcpy(sigma,g_sigma, multi_size * sizeof(int),hipMemcpyHostToHost);
        // hipMemcpy(dist, g_dist,   multi_size * sizeof(float),hipMemcpyHostToHost);
        // hipMemcpy(f1, g_f1,   mappingCount * sizeof(Q_struct),hipMemcpyDeviceToHost);

        // printf("sigma: ");
        // for(int i=0;i<V;i++){
        //     printf("[");
        //     for(int j=0;j<mappingCount;j++){
        //         printf("%d,",sigma[mappingCount*i+j]);
        //     }
        //     printf("] ");
        // }
        // printf("\n");

        // printf("dist: ");
        // for(int i=0;i<V;i++){
        //     printf("[");
        //     for(int j=0;j<mappingCount;j++){
        //         printf("%.0f,",dist[mappingCount*i+j]);
        //     }
        //     printf("] ");
        // }
        // printf("\n");

        // printf("--------------------multi Source-------------------- ");
        // for(int i=0;i<mappingCount;i++){
        //     printf("%d ",map_S[i]);
        // }
        // printf("\n");

        // printf("f1: ");
        // for(int i=0;i<multi_size;i++){
        //     printf("[%d, %d]> ",f1[i].nodeID,f1[i].traverse_S);
        // }
        // printf("\n");
        #pragma  endregion

        // int f1_indicator = 0;
        // int f2_indicator = 0;
        // int s_indicator = 0;
       

        int level=0;
        while (currentQueueSize > 0) { //currentQueueSize > 0
            // std::cout<<"currentQueueSize: "<<currentQueueSize<<std::endl;
            Q_struct* g_currentQueue;
            Q_struct* g_nextQueue;
            INITIAL_Qtruct<<<ceil(V/64.0),min(V,64)>>>(g_nextQueue_temp,V);
            hipDeviceSynchronize();
            
            int index = level & 1;  // 等價於 level % 2
            g_currentQueue = (index == 0) ? g_f1 : g_f2;
            g_nextQueue    = (index == 0) ? g_f2 : g_f1;
            
            stackOffset[level+1] = currentQueueSize + stackOffset[level];
            int blocknum = (currentQueueSize < INT_MAX) ? currentQueueSize : INT_MAX;
            //平行跑BFS
            // printf("currentQueueSize: %d\n",currentQueueSize);
            for(int i=0;i<(int)ceil(currentQueueSize/(float)INT_MAX);i++){
                // allBC_MS<<<blocknum,threadnum>>>(g_csrV,g_csrE,g_nextQueueSize,g_currentQueue,g_nextQueue,g_dist,g_sigma,INT_MAX,i,currentQueueSize,mappingCount);
                allBC_MS_VnextQ<<<blocknum,threadnum>>>(g_csrV,g_csrE,g_currentQueue,g_nextQueue_temp,g_dist,g_sigma,INT_MAX,i,currentQueueSize,mappingCount);
                hipDeviceSynchronize();
                rearrange_queue_MS<<<ceil(V/64.0),min(V,64)>>>(g_nextQueue,g_nextQueue_temp,g_nextQueueSize,V);
                hipDeviceSynchronize();
            }
                    

            // Swap currentQueue and nextQueue
            // CHECK(hipMemcpy(&currentQueueSize,g_nextQueueSize,sizeof(int),hipMemcpyDeviceToHost));
            // CHECK(hipMemcpy(&g_stack[stackOffset[level+1]],g_nextQueue,currentQueueSize*sizeof(Q_struct),hipMemcpyDeviceToDevice));
            // CHECK(hipMemset(g_nextQueueSize,0,sizeof(int)));
            hipMemcpy(&currentQueueSize,g_nextQueueSize,sizeof(int),hipMemcpyDeviceToHost);
            hipMemcpy(&g_stack[stackOffset[level+1]],g_nextQueue,currentQueueSize*sizeof(Q_struct),hipMemcpyDeviceToDevice);
            hipMemset(g_nextQueueSize,0,sizeof(int));
            level++;


            #pragma  region print
            // CHECK(hipMemcpy(&queue[0],g_nextQueue, currentQueueSize*sizeof(Q_struct),hipMemcpyDeviceToHost));
            // printf("real_f1: ");
            // for(int i=0;i<currentQueueSize;i++){
            //     printf("[%d, %lu]> ",queue[i].nodeID,queue[i].traverse_S);
            // }
            // printf("\n");
            #pragma  endregion


        }

        #pragma  region print
        //檢查GPU資料
        // hipMemcpy(sigma,g_sigma, multi_size * sizeof(int),hipMemcpyDeviceToHost);
        // hipMemcpy(dist, g_dist,   multi_size * sizeof(float),hipMemcpyDeviceToHost);
        // hipMemcpy(f1, g_f1,   mappingCount * sizeof(Q_struct),hipMemcpyDeviceToHost);

        // printf("sigma: ");
        // for(int i=0;i<V;i++){
        //     printf("[");
        //     for(int j=0;j<mappingCount;j++){
        //         printf("%d,",sigma[mappingCount*i+j]);
        //     }
        //     printf("] ");
        // }
        // printf("\n");

        // printf("dist: ");
        // for(int i=0;i<V;i++){
        //     printf("[");
        //     for(int j=0;j<mappingCount;j++){
        //         printf("%.0f,",dist[mappingCount*i+j]);
        //     }
        //     printf("] ");
        // }
        // printf("\n");


        // printf("f1: ");
        // for(int i=0;i<multi_size;i++){
        //     printf("[%d, %d]> ",f1[i].nodeID,f1[i].traverse_S);
        // }
        // printf("\n");
        #pragma  endregion



        // multi_time2 = seconds();
        // multi_forward_Time += (multi_time2 - multi_time1);
        // multi_time1 = seconds();

        // Back-propagation
        //  std::cout << "--------------backward--------------"<< std::endl;
        for (int d = level - 1; d > 0; d--) {
            int degree =(stackOffset[d+1] - stackOffset[d]);
            int blocknum = (degree < INT_MAX) ? degree : INT_MAX;
            // std::cout << "backward level(" << d << "):\t" << stackOffset[d+1] - stackOffset[d] << std::endl;
            for(int i=0;i<(int)ceil(degree/(float)INT_MAX);i++){
                deltaCalculation_MS<<<blocknum,threadnum>>>(g_csrV,g_csrE,g_delta,g_sigma,g_stack,g_dist,INT_MAX,i,stackOffset[d],degree,mappingCount);
                CHECK(hipDeviceSynchronize());
            }
                
            // hipDeviceSynchronize();
            #pragma  region print
            // CHECK(hipMemcpy(&delta[0], g_delta,   multi_size * sizeof(float),hipMemcpyDeviceToHost));
            // printf("delta: ");
            // for(int i=0;i<V;i++){
            //     printf("[");
            //     for(int j=0;j<mappingCount;j++){
            //         printf("%.3f,",delta[mappingCount*i+j]);
            //     }
            //     printf("] ");
            // }
            // printf("\n");
            #pragma  endregion
        }
        int shared_mem_size = (mappingCount) * sizeof(int);
        sum_BC_Result_MS<<<ceil(V/128.0),min(V,128),shared_mem_size>>>(g_BC,g_delta,V,g_map_S,mappingCount);
        CHECK(hipDeviceSynchronize());

        

        // multi_time2 = seconds();
        // multi_backward_Time += (multi_time2 - multi_time1);
    }
    CHECK(hipMemcpy(&BC[0],g_BC, V*sizeof(float),hipMemcpyDeviceToHost));



    // multi_time_end = seconds();
    // multi_total_time = (multi_time_end - multi_time_start);
    // 釋放所有 GPU 資源
    hipFree(g_stack);
    hipFree(g_sigma);
    hipFree(g_dist);
    hipFree(g_level);
    hipFree(g_delta);
    hipFree(g_S_size);
    hipFree(g_f1);
    hipFree(g_f2);
    hipFree(g_nextQueue_temp);
    hipFree(g_nextQueueSize);
    hipFree(g_csrE);
    hipFree(g_csrV);
    hipFree(g_map_S);
    hipFree(g_BC);
   
}


